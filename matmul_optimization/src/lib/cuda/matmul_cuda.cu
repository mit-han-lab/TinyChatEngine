#include "hip/hip_runtime.h"
#include <assert.h>
#include <sys/time.h>
#include <stdlib.h>
#include <cstdlib>
#include <iostream>

#include "../matmul.h"

#include <torch/torch.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <hip/hip_fp16.h>
#include <c10/cuda/CUDAGuard.h>
#include "gemm_cuda.h"
#include "dequantize.cuh"

const int threadDim = 32;
const int TILE_SIZE = threadDim;

static bool first_run = true;

__global__ void matrixMul_blockC(float *A, float *B, float *C, int A_row, int A_column, int B_column){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	float acc = 0;
	for (int k = 0; k < A_column; k++)
		acc += A[j * A_column + k] * B[k * B_column + i];
	C[j * B_column +i] = acc;
}

__global__ void matrixMultiplyShared(const float *A, const float *B, float *C, int A_row, int A_column, int B_column) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float As[TILE_SIZE][TILE_SIZE];
	__shared__ float Bs[TILE_SIZE][TILE_SIZE];

	float value = 0;

	for (int i = 0; i < A_column / TILE_SIZE; i++){
		As[threadIdx.y][threadIdx.x] = A[(blockIdx.y * TILE_SIZE + threadIdx.y) * A_column + TILE_SIZE * i + threadIdx.x];
		Bs[threadIdx.y][threadIdx.x] = B[(i * TILE_SIZE + threadIdx.y) * B_column + blockIdx.x * TILE_SIZE + threadIdx.x];

		__syncthreads();

		for (int k = 0; k < TILE_SIZE; k++)
			value += As[threadIdx.y][k] * Bs[k][threadIdx.x];

		__syncthreads();
	}


	C[row * B_column + col] = value;
}


/* AWQ Implementation */

// Pack two half values.
static inline __device__ __host__ unsigned
__pack_half2(const half x, const half y) {
  unsigned v0 = *((unsigned short *)&x);
  unsigned v1 = *((unsigned short *)&y);
  return (v1 << 16) | v0;
}

__global__ void __launch_bounds__(64) gemm_forward_4bit_cuda_m16n128k32(int split_k_iters, half* __restrict__ A, int* __restrict__ B, half* __restrict__ scaling_factors, int* __restrict__ zeros, int M, int IC, int OC, half* __restrict__ C) 
{
  static constexpr uint32_t ZERO = 0x0;
  float C_warp[32];
  __shared__ half A_shared[16 * (32 + 8)];
  __shared__ half B_shared[32 * (128 + 8)];
  
  __shared__ half scaling_factors_shared[128];
  __shared__ half zeros_shared[128];

  int j_factors1 = ((OC + 128 - 1) / 128);

  int blockIdx_x = 0;
  int blockIdx_y = blockIdx.x % ((M + 16 - 1) / 16 * j_factors1);
  int blockIdx_z = blockIdx.x / ((M + 16 - 1) / 16 * j_factors1);

  half A_shared_warp[8];
  half B_shared_warp[32];
  for (int j_0_4_init = 0; j_0_4_init < 4; ++j_0_4_init) {
    for (int i = 0; i < 8; ++i) {
      C_warp[(j_0_4_init * 8) + i] = 0.0;
    }
  }

  static constexpr int row_stride_warp = 32 * 8 / 32;
  static constexpr int row_stride = 2 * 32 * 8 / 128;
  bool ld_zero_flag = (threadIdx.y * 32 + threadIdx.x) * 8 < 128;
  // TODO: Haotian: blockIdx_y / j_factors1 in A loading to support bsz > 16
  bool ld_A_flag = (blockIdx_y / j_factors1 * 16 + threadIdx.y * row_stride_warp + threadIdx.x * 8 / 32) < M;     // threadIdx.y is warp_id
  // bool wb_C_flag = (threadIdx.x / 4) < M;

  half* A_ptr = A 
                + (((int)blockIdx_y) / j_factors1 * 16 + (((int)threadIdx.y) * row_stride_warp) + ((int)threadIdx.x) / (32 / 8)) * IC
                + (((int)threadIdx.x) % (32 / 8)) * 8;
  
  int* B_ptr = B
            + ((int)threadIdx.y) * (OC / 8) * 2
            + (((int)threadIdx.x) / (128 / 8)) * (OC / 8)
            + (((int)blockIdx_y) % j_factors1) * (128 / 8)
            + (((int)threadIdx.x) % (128 / 8)) * 1;
                        
  half* A_shared_ptr = A_shared 
                    + ((int)threadIdx.y) * row_stride_warp * (32 + 8) 
                    + (((int)threadIdx.x) / (32 / 8)) * (32 + 8)
                    + (((int)threadIdx.x) % (32 / 8) ) * 8;

  half* B_shared_ptr = B_shared
                    + ((int)threadIdx.y) * (row_stride / 2) * (128 + 8)
                    + (((int)threadIdx.x) / (128 / 8)) * (128 + 8)
                    + (((int)threadIdx.x) % (128 / 8)) * 8;
  
  int* zeros_ptr = zeros
                + (((int)blockIdx_y) % j_factors1) * (128 / 8)
                + ((int)threadIdx.x) % (128 / 8);
  
  half* scaling_factors_ptr = scaling_factors
                            + (((int)blockIdx_y) % j_factors1) * (128) 
                            + (((int)threadIdx.x) % (128 / 8)) * 8;

  half* C_ptr = C 
              + blockIdx_z * M * OC        // blockIdz.x -> split_k dim
              + (((int)blockIdx_y) % j_factors1) * 128
              + ((int)threadIdx.y) * 64
              + (((int)threadIdx.x) % 4) * 2;

  // preload s.f. and zeros
  int k_bound = (IC / 32 + split_k_iters - 1) / split_k_iters;
  if ((k_bound - 1) * 32 + blockIdx_z >= IC) k_bound -= 1;
  for (int _k_0_0 = 0; _k_0_0 < k_bound; ++_k_0_0) {
    int k_0_0 = _k_0_0 * split_k_iters + blockIdx_z;
    __syncthreads();
    // TODO: Haotian: blockIdx_y / j_factors1 in A loading to support bsz > 16
    if (ld_A_flag)
    {
      *(uint4*)(A_shared_ptr) = *(uint4*)(A_ptr + (k_0_0 * 32));
    }
    else
    {
      *(uint4*)(A_shared_ptr) = make_uint4(0, 0, 0, 0);
    }

    // for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 2; ++ax0_ax1_fused_0) {
    uint32_t zeros_loaded = *(uint32_t*)(zeros_ptr + k_0_0 * 32 / 128 * (OC / 8));
    uint4 B_loaded_zero = dequantize_s4_to_fp16x2(zeros_loaded);
    uint4 B_loaded_scale = *(uint4*)(scaling_factors_ptr + k_0_0 * 32 / 128 * (OC));
    /*
    if (blockIdx_z == 0 && blockIdx_y == 0 && k_0_0 == 0 && threadIdx.x == 0 && threadIdx.y == 0){
      printf("%x %x %x %x %x %x %x %x\n", B_loaded_scale.x, B_loaded_scale.y, B_loaded_scale.z, B_loaded_scale.w, B_loaded_zero.x, B_loaded_zero.y, B_loaded_zero.z, B_loaded_zero.w);
    }
    */
    // uint4 B_loaded_scale = make_uint4(0, 0, 0, 0);
    int* B_ptr_local = B_ptr + k_0_0 * 32 * (OC / 8);

    for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 8; ++ax0_ax1_fused_0) {

      // B: 32 x 136 (128+8) float16
      // each warp: 32 x 4
      // each thr: read 32 bit -> convert to 8xFP16 (a UINT4) -> scale and minus zero -> WB UINT4
      // *(uint4*)(B_shared + ((((ax0_ax1_fused_0 * 544) + (((int)threadIdx.y) * 272)) + ((((int)threadIdx.x) >> 4) * 136)) + ((((int)threadIdx.x) & 15) * 8))) = *(uint4*)(B + ((((((k_0_0 * 163840) + (ax0_ax1_fused_0 * 20480)) + (((int)threadIdx.y) * 10240)) + ((((int)threadIdx.x) >> 4) * 5120)) + (((int)blockIdx_y) * 128)) + ((((int)threadIdx.x) & 15) * 8)));
      // row stride in shared memory: (NWARPS * 32 * 8 / cta_N) 
      uint32_t B_loaded = *(uint32_t*)(B_ptr_local + ax0_ax1_fused_0 * row_stride * (OC / 8));
      uint4 B_loaded_fp16 = dequantize_s4_to_fp16x2(B_loaded);
      //uint4 B_loaded_zero = *(uint4*)(zeros_shared + (threadIdx.x % (cta_N / 8)) * 8);

      // uint4 B_loaded_scale = *(uint4*)(scaling_factors_shared + (threadIdx.x % (cta_N / 8)) * 8);
      // - zero and * scale
      // TODO (Haotian): can save 4 assembly instructions if sormulate as deq = q * scale - zero * scale.
      asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.x) : "r"(B_loaded_fp16.x), "r"(B_loaded_zero.x));
      asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.x) : "r"(B_loaded_fp16.x), "r"(B_loaded_scale.x), "r"(ZERO));
      asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.y) : "r"(B_loaded_fp16.y), "r"(B_loaded_zero.y));
      asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.y) : "r"(B_loaded_fp16.y), "r"(B_loaded_scale.y), "r"(ZERO));
      asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.z) : "r"(B_loaded_fp16.z), "r"(B_loaded_zero.z));
      asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.z) : "r"(B_loaded_fp16.z), "r"(B_loaded_scale.z), "r"(ZERO));
      asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.w) : "r"(B_loaded_fp16.w), "r"(B_loaded_zero.w));
      asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.w) : "r"(B_loaded_fp16.w), "r"(B_loaded_scale.w), "r"(ZERO));
      /*
      if (ax0_ax1_fused_0 == 0 && blockIdx_z == 0 && blockIdx_y == 0 && k_0_0 == 0 && threadIdx.x == 17 && threadIdx.y == 0){
        printf("[x] %X %X %X %X\n", B_loaded_fp16.x, B_loaded_fp16.y, B_loaded_fp16.z, B_loaded_fp16.w);
      }
      */

      // write back
      *(uint4*)(B_shared_ptr + ax0_ax1_fused_0 * row_stride * (128 + 8)) = B_loaded_fp16;
    }
    __syncthreads();

    for (int k_0_1 = 0; k_0_1 < 2; ++k_0_1) {
      {
        unsigned int addr;
        __asm__ __volatile__(
          "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
          : "=r"(addr)
          : "l"((void *)((&(A_shared[(k_0_1 * 16)])) + (((((int)threadIdx.x) & 15) * 40) + ((((int)threadIdx.x) >> 4) * 8))))
        );


        __asm__ __volatile__(
          "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
          "{%0, %1, %2, %3}, [%4];\n"
          : "=r"(((unsigned *)(A_shared_warp + 0))[0]), "=r"(((unsigned *)(A_shared_warp + 0))[1]), "=r"(((unsigned *)(A_shared_warp + 0))[2]), "=r"(((unsigned *)(A_shared_warp + 0))[3])
          : "r"(addr)
        );
      }

      for (int ax1_0 = 0; ax1_0 < 4; ++ax1_0) {
        {
          unsigned int addr;
          __asm__ __volatile__(
            "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
            : "=r"(addr)
            : "l"((void *)((&(B_shared[(((k_0_1 * 2176) + (((int)threadIdx.y) * 64)) + (ax1_0 * 16))])) + (((((int)threadIdx.x) & 15) * 136) + ((((int)threadIdx.x) >> 4) * 8))))
          );
          __asm__ __volatile__(
            "ldmatrix.sync.aligned.m8n8.x4.trans.shared.b16"
            "{%0, %1, %2, %3}, [%4];\n"
            : "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[0]), "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[1]), "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[2]), "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[3])
            : "r"(addr)
          );
        }
      }
      for (int j_0_4 = 0; j_0_4 < 4; ++j_0_4) {
        {
          __asm__ __volatile__(
            "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
            "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
            :  "=f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[3])
            : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + (j_0_4 * 8)))[0]), "r"(((unsigned *)(B_shared_warp + (j_0_4 * 8)))[1]), "f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "f"(((float *)(C_warp + (j_0_4 * 8)))[3]));
        }

        {
          __asm__ __volatile__(
            "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
            "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
            :  "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3])
            : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + ((j_0_4 * 8) + 4)))[0]), "r"(((unsigned *)(B_shared_warp + ((j_0_4 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3]));
        }
      }
    }
  }

// TODO: Shang: Hoist loop invariance.
  for (int ax1_0_1 = 0; ax1_0_1 < 4; ++ax1_0_1) {
    for (int local_id = 0; local_id < 8; ++local_id) {
      int row_offset = (((int)blockIdx_y) / j_factors1) * 16 + ((int)threadIdx.x) / 4 + (local_id % 4) / 2 * 8;
      if (row_offset < M)
      {
        *(C_ptr + ax1_0_1 * 16 + row_offset * OC + (local_id / 4) * 8 + local_id % 2) = __float2half(C_warp[(ax1_0_1 * 8) + local_id]);
      }
    }
  }
}

// in_feats: M, IC [float16]
// kernel: IC, OC // 8 [int32] -> cast to IC, OC [uint4b]
// scaling_factors: IC // G, OC [float16]
// zeros: IC // G, OC // 8 [int32] -> cast to IC // G, OC [uint4b]
// assume that batch_size < 16 for now

torch::Tensor gemm_forward_cuda(
    torch::Tensor _in_feats,
    torch::Tensor _kernel,
    torch::Tensor _scaling_factors,
    torch::Tensor _zeros,
    int split_k_iters)
{
    // std::cout << "0000" << std::endl;
    int num_in_feats = _in_feats.size(0);
    // std::cout << "num_in_feats = " << num_in_feats << std::endl;
    int num_in_channels = _in_feats.size(1);
    // std::cout << "num_in_channels = " << num_in_channels << std::endl;
    // std::cout << "a" << std::endl;
    // if (first_run) {
    //   _in_feats = _in_feats.to(torch::kCUDA);
    //   // _kernel = _kernel.to(torch::kCUDA);
    //   // _scaling_factors = _scaling_factors.to(torch::kCUDA);
    //   // _zeros = _zeros.to(torch::kCUDA);
    //   first_run = false;
    // }
    // _in_feats = _in_feats.to(torch::kCUDA);
    // std::cout << "?" << std::endl;
    
    // const at::cuda::OptionalCUDAGuard device_guard(device_of(_in_feats));
    
    // std::cout << "b" << std::endl;

    auto options = torch::TensorOptions().dtype(_in_feats.dtype()).device(_in_feats.device());
    // std::cout << "c" << std::endl;
    at::Tensor _out_feats = torch::empty({split_k_iters, num_in_feats, _kernel.size(1) * 8}, options);
    // std::cout << "d" << std::endl;
    int num_out_feats = _out_feats.size(-2);
    int num_out_channels = _out_feats.size(-1);

    // std::cout << "1" << std::endl;
    auto in_feats = reinterpret_cast<half*>(_in_feats.data_ptr<float>());
    // std::cout << "2" << std::endl;
    // // std::cout << _kernel << std::endl;
    auto kernel = reinterpret_cast<int*>(_kernel.data_ptr<int>());
    // std::cout << "3" << std::endl;
    auto out_feats = reinterpret_cast<half*>(_out_feats.data_ptr<float>());
    // std::cout << "4" << std::endl;
    auto scaling_factors = reinterpret_cast<half*>(_scaling_factors.data_ptr<float>());
    // std::cout << "5" << std::endl;
    auto zeros = reinterpret_cast<int*>(_zeros.data_ptr<int>());

    // std::cout << "BBBBB" << std::endl;

    if (num_out_channels % 128 != 0)
        throw std::invalid_argument("OC is not multiple of cta_N = 128");
    if (num_out_channels % 8 != 0)
        throw std::invalid_argument("OC is not multiple of pack_num = 8");
    int j_factors1 = num_out_channels / 128 / 1;
    dim3 num_blocks((num_out_feats + 16 - 1) / 16 * j_factors1 * split_k_iters);
    
    // threadIdx.x: 32
    // threadIdx.y: i_factors[2] * j_factors[2]
    dim3 threads_per_block(32, 2);
    // std::cout << "CCCCC" << std::endl;
    gemm_forward_4bit_cuda_m16n128k32<<<num_blocks, threads_per_block>>>(
        split_k_iters, in_feats, kernel, scaling_factors, zeros, num_in_feats, num_in_channels, num_out_channels, out_feats);

    // std::cout << "DDDDD" << std::endl;

    return _out_feats.sum(0);
}


namespace matmul{
  void MatmulOperator::mat_mul_accelerator_int4_fast(const struct matmul_params *params) {
		const struct matrix *A = &params->A, *B = &params->B, *C = &params->C;

    // std::cout << "mat_mul_accelerator_int4_fast -- A->row: " << A->row << " A->column: " << A->column 
              // << "; B->row: " << B->row << " B->column: " << B->column 
              // << "; C->row: " << C->row << " C->column: " << C->column << std::endl;
    
    // torch::Tensor out_feats = gemm_forward_cuda(
    //     torch::from_blob(A->data_ptr, {A->row, A->column}, torch::kHalf),
    //     torch::from_blob(B->data_ptr, {B->row, B->column}, torch::kInt),
    //     torch::from_blob(params->scales, {B->row / 128, B->column * 8}, torch::kHalf),
    //     torch::from_blob(params->int32_zero_point, {B->row / 128, B->column}, torch::kInt),
    //     8);
    
    auto option_fp = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, 1);
    auto option_int = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA, 1);
    torch::Tensor _in_feats = torch::from_blob(A->data_ptr, {A->row, A->column}, option_fp);
    torch::Tensor _kernel = torch::from_blob(B->data_ptr, {B->row, B->column}, option_int);
    torch::Tensor _scaling_factors = torch::from_blob(params->scales, {B->row / 128, B->column * 8}, option_fp);
    torch::Tensor _zeros = torch::from_blob(params->int32_zero_point, {B->row / 128, B->column}, option_int);

    torch::Tensor out_feats = gemm_forward_cuda(
        _in_feats,
        _kernel,
        _scaling_factors,
        _zeros,
        8);
    
    hipMemcpy(C->data_ptr, out_feats.data_ptr(), C->column * C->row * sizeof(float), hipMemcpyDeviceToHost);
  };

  void MatmulOperator::mat_mul_accelerator_int4_fast_no_offset(const struct matmul_params *params) {
		const struct matrix *A = &params->A, *B = &params->B, *C = &params->C;

    // std::cout << "mat_mul_accelerator_int4_fast_no_offset -- A->row: " << A->row << " A->column: " << A->column 
              // << "; B->row: " << B->row << " B->column: " << B->column 
              // << "; C->row: " << C->row << " C->column: " << C->column << std::endl;
    
    // std::cout << "AAAAA" << std::endl;
    
    // torch::Tensor out_feats = gemm_forward_cuda(
    //     torch::from_blob(A->data_ptr, {A->row, A->column}, torch::kHalf),
    //     torch::from_blob(B->data_ptr, {B->row, B->column}, torch::kInt),
    //     torch::from_blob(params->scales, {B->row / 128, B->column * 8}, torch::kHalf),
    //     torch::from_blob(params->int32_zero_point, {B->row / 128, B->column}, torch::kInt),
    //     8);
    
    auto option_fp = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, 1);
    auto option_int = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA, 1);
    // std::cout << "jcuycu" << std::endl;
    //torch::Tensor _in_feats = torch::from_blob(A->data_ptr, {A->row, A->column}, option_fp);
    torch::Tensor _in_feats = torch::from_blob(A->data_ptr, {A->row, A->column}).to(torch::kCUDA);
    // std::cout << "kchy" << std::endl;
    // torch::Tensor _kernel = torch::from_blob(B->int32_data_ptr, {B->row, B->column}, option_int);
    torch::Tensor _kernel = torch::from_blob(B->int32_data_ptr, {B->row, B->column}, torch::kInt).to(torch::kCUDA);
    // std::cout << "qued" << std::endl;
    // torch::Tensor _scaling_factors = torch::from_blob(params->scales, {B->row / 128, B->column * 8}, option_fp);
    torch::Tensor _scaling_factors = torch::from_blob(params->scales, {B->row / 128, B->column * 8}).to(torch::kCUDA);
    // std::cout << "-cud" << std::endl;
    // torch::Tensor _zeros = torch::from_blob(params->int32_zero_point, {B->row / 128, B->column}, option_int);
    torch::Tensor _zeros = torch::from_blob(params->int32_zero_point, {B->row / 128, B->column}, torch::kInt).to(torch::kCUDA);
    // std::cout << "msadi" << std::endl;

    torch::Tensor out_feats = gemm_forward_cuda(
        _in_feats,
        _kernel,
        _scaling_factors,
        _zeros,
        8);
    
    hipMemcpy(C->data_ptr, out_feats.data_ptr(), C->column * C->row * sizeof(float), hipMemcpyDeviceToHost);
  };

	void MatmulOperator::mat_mul_cuda(const struct matmul_params *params){
		const struct matrix *A = &params->A, *B = &params->B, *C = &params->C;
		assert(A->column == B->row);
		assert(C->column == B->column);
		assert(C->row == A->row);

		float *d_A;
		float *d_B;
		float *d_C;

		// Initailize C
		/*for (int i = 0; i < C->row; i++)
		  for (int j = 0; j < C->column; j++)
		  C->data_ptr[j + C->column * i] = 0;*/

		// Allocate memory
		hipMalloc(&d_A, A->column*A->row*sizeof(float));
		hipMalloc(&d_B, B->column*B->row*sizeof(float));
		hipMalloc(&d_C, C->column*C->row*sizeof(float));

		// Copy data to GPU
		hipMemcpy(d_A, A->data_ptr, A->column*A->row*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_B, B->data_ptr, B->column*B->row*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_C, C->data_ptr, C->column*C->row*sizeof(float), hipMemcpyHostToDevice);

		// Make sure we can break the input matrix into blocks
		assert(A->column % threadDim == 0);
		assert(A->row % threadDim == 0);
		assert(B->column % threadDim == 0);
		const dim3 threadsPerBlock(threadDim, threadDim);
		const dim3 numBlocks(C->column / threadsPerBlock.x, C->row / threadsPerBlock.y);

		// Invoke the cuda imp.

		// struct timeval start, end;
		// gettimeofday(&start, NULL);
		//matrixMul_blockC<<< numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, A->row, A->column, B->column);
		matrixMultiplyShared<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, A->row, A->column, B->column);
		hipDeviceSynchronize();
		// gettimeofday(&end, NULL);
		// int us = interval_to_us(&start, &end);
		// // std::cout << "cuda kernel: " << us / 1000 << " ms" << std::endl;

		// Get the result back
		hipMemcpy(C->data_ptr, d_C, C->column*C->row*sizeof(float), hipMemcpyDeviceToHost);
	}
}  // namespace matmul
