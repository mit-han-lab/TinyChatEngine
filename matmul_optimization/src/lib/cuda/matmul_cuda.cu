#include "hip/hip_runtime.h"
#include <assert.h>
#include <sys/time.h>
#include <stdlib.h>
#include <cstdlib>
#include <iostream>

#include "../matmul.h"
#include "linear.h"

// #include <torch/torch.h>
#include <hip/hip_runtime.h>
// #include <torch/extension.h>
#include <hip/hip_fp16.h>
// #include <c10/cuda/CUDAGuard.h>
#include "gemm_cuda.h"
#include "dequantize.cuh"

__global__ void float2half(float* floatArray, half* halfArray, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) {
        halfArray[index] = __float2half(floatArray[index]);
    }
}

__global__ void half2float(half* halfArray, float* floatArray, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) {
        floatArray[index] = __half2float(halfArray[index]);
    }
}

__global__ void half2float_merge_k_iters(half *halfArray, float *floatArray, int N, int split_k_iters) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < N) {
        float sum = 0;
        for (int j = 0; j < split_k_iters; j++) {
            sum += __half2float(halfArray[index + j * N]);
        }
        floatArray[index] = sum;
    }
}

const int threadDim = 32;
const int TILE_SIZE = threadDim;

__global__ void matrixMul_blockC(float *A, float *B, float *C, int A_row, int A_column, int B_column){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	float acc = 0;
  if(i < B_column && j < A_row) {
    for (int k = 0; k < A_column; k++)
      acc += A[j * A_column + k] * B[k * B_column + i];
    C[j * B_column +i] = acc;
  }
}

__global__ void matrixMultiplyShared(const float *A, const float *B, float *C, int A_row, int A_column, int B_column) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float As[TILE_SIZE][TILE_SIZE];
	__shared__ float Bs[TILE_SIZE][TILE_SIZE];

	float value = 0;

	for (int i = 0; i < A_column / TILE_SIZE; i++){
		As[threadIdx.y][threadIdx.x] = A[(blockIdx.y * TILE_SIZE + threadIdx.y) * A_column + TILE_SIZE * i + threadIdx.x];
		Bs[threadIdx.y][threadIdx.x] = B[(i * TILE_SIZE + threadIdx.y) * B_column + blockIdx.x * TILE_SIZE + threadIdx.x];

		__syncthreads();

		for (int k = 0; k < TILE_SIZE; k++)
			value += As[threadIdx.y][k] * Bs[k][threadIdx.x];

		__syncthreads();
	}


	C[row * B_column + col] = value;
}


/* AWQ Implementation */

// Pack two half values.
static inline __device__ __host__ unsigned
__pack_half2(const half x, const half y) {
  unsigned v0 = *((unsigned short *)&x);
  unsigned v1 = *((unsigned short *)&y);
  return (v1 << 16) | v0;
}

__global__ void __launch_bounds__(64) gemm_forward_4bit_cuda_m16n128k32(int G, int split_k_iters, half* __restrict__ A, int* __restrict__ B, half* __restrict__ scaling_factors, int* __restrict__ zeros, int M, int IC, int OC, half* __restrict__ C) 
{
  static constexpr uint32_t ZERO = 0x0;
  float C_warp[32];
  __shared__ half A_shared[16 * (32 + 8)];
  __shared__ half B_shared[32 * (128 + 8)];
  
  __shared__ half scaling_factors_shared[128];
  __shared__ half zeros_shared[128];

  int j_factors1 = ((OC + 128 - 1) / 128);
  int blockIdx_x = 0;
  int blockIdx_y = blockIdx.x % ((M + 16 - 1) / 16 * j_factors1);
  int blockIdx_z = blockIdx.x / ((M + 16 - 1) / 16 * j_factors1);

  half A_shared_warp[8];
  half B_shared_warp[32];
  for (int j_0_4_init = 0; j_0_4_init < 4; ++j_0_4_init) {
    for (int i = 0; i < 8; ++i) {
      C_warp[(j_0_4_init * 8) + i] = 0.0;
    }
  }

  static constexpr int row_stride_warp = 32 * 8 / 32;
  static constexpr int row_stride = 2 * 32 * 8 / 128;
  bool ld_zero_flag = (threadIdx.y * 32 + threadIdx.x) * 8 < 128;
  // TODO: Haotian: blockIdx_y / j_factors1 in A loading to support bsz > 16
  bool ld_A_flag = (blockIdx_y / j_factors1 * 16 + threadIdx.y * row_stride_warp + threadIdx.x * 8 / 32) < M;     // threadIdx.y is warp_id
  // bool wb_C_flag = (threadIdx.x / 4) < M;

  half* A_ptr = A 
                + (((int)blockIdx_y) / j_factors1 * 16 + (((int)threadIdx.y) * row_stride_warp) + ((int)threadIdx.x) / (32 / 8)) * IC
                + (((int)threadIdx.x) % (32 / 8)) * 8;
  
  int* B_ptr = B
            + ((int)threadIdx.y) * (OC / 8) * 2
            + (((int)threadIdx.x) / (128 / 8)) * (OC / 8)
            + (((int)blockIdx_y) % j_factors1) * (128 / 8)
            + (((int)threadIdx.x) % (128 / 8)) * 1;
// Why * 1 in the above line?
                        
  half* A_shared_ptr = A_shared 
                    + ((int)threadIdx.y) * row_stride_warp * (32 + 8) 
                    + (((int)threadIdx.x) / (32 / 8)) * (32 + 8)
                    + (((int)threadIdx.x) % (32 / 8) ) * 8;

  half* B_shared_ptr = B_shared
                    + ((int)threadIdx.y) * (row_stride / 2) * (128 + 8)
                    + (((int)threadIdx.x) / (128 / 8)) * (128 + 8)
                    + (((int)threadIdx.x) % (128 / 8)) * 8;
  
  int* zeros_ptr = zeros
                + (((int)blockIdx_y) % j_factors1) * (128 / 8)
                + ((int)threadIdx.x) % (128 / 8);
  
  half* scaling_factors_ptr = scaling_factors
                            + (((int)blockIdx_y) % j_factors1) * (128) 
                            + (((int)threadIdx.x) % (128 / 8)) * 8;

  half* C_ptr = C 
              + blockIdx_z * M * OC        // blockIdz.x -> split_k dim
              + (((int)blockIdx_y) % j_factors1) * 128
              + ((int)threadIdx.y) * 64
              + (((int)threadIdx.x) % 4) * 2;

  // preload s.f. and zeros
  int k_bound = (IC / 32 + split_k_iters - 1) / split_k_iters;
  if ((k_bound - 1) * split_k_iters * 32 + blockIdx_z * 32 >= IC) k_bound -= 1;
  for (int _k_0_0 = 0; _k_0_0 < k_bound; ++_k_0_0) {
    int k_0_0 = _k_0_0 * split_k_iters + blockIdx_z;
    __syncthreads();
    // TODO: Haotian: blockIdx_y / j_factors1 in A loading to support bsz > 16
    if (ld_A_flag)
    {
      *(uint4*)(A_shared_ptr) = *(uint4*)(A_ptr + (k_0_0 * 32));
    }
    else
    {
      *(uint4*)(A_shared_ptr) = make_uint4(0, 0, 0, 0);
    }

    // for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 2; ++ax0_ax1_fused_0) {
    uint32_t zeros_loaded = *(uint32_t*)(zeros_ptr + k_0_0 * 32 / G * (OC / 8));
    uint4 B_loaded_zero = dequantize_s4_to_fp16x2(zeros_loaded);
    uint4 B_loaded_scale = *(uint4*)(scaling_factors_ptr + k_0_0 * 32 / G * (OC));
    /*
    if (blockIdx_z == 0 && blockIdx_y == 0 && k_0_0 == 0 && threadIdx.x == 0 && threadIdx.y == 0){
      printf("%x %x %x %x %x %x %x %x\n", B_loaded_scale.x, B_loaded_scale.y, B_loaded_scale.z, B_loaded_scale.w, B_loaded_zero.x, B_loaded_zero.y, B_loaded_zero.z, B_loaded_zero.w);
    }
    */
    // uint4 B_loaded_scale = make_uint4(0, 0, 0, 0);
    int* B_ptr_local = B_ptr + k_0_0 * 32 * (OC / 8);

    for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 8; ++ax0_ax1_fused_0) {
// TODO: Shang: double check how to get 8.

      // B: 32 x 136 (128+8) float16
      // each warp: 32 x 4
      // each thr: read 32 bit -> convert to 8xFP16 (a UINT4) -> scale and minus zero -> WB UINT4
      // *(uint4*)(B_shared + ((((ax0_ax1_fused_0 * 544) + (((int)threadIdx.y) * 272)) + ((((int)threadIdx.x) >> 4) * 136)) + ((((int)threadIdx.x) & 15) * 8))) = *(uint4*)(B + ((((((k_0_0 * 163840) + (ax0_ax1_fused_0 * 20480)) + (((int)threadIdx.y) * 10240)) + ((((int)threadIdx.x) >> 4) * 5120)) + (((int)blockIdx_y) * 128)) + ((((int)threadIdx.x) & 15) * 8)));
      // row stride in shared memory: (NWARPS * 32 * 8 / cta_N) 
      uint32_t B_loaded = *(uint32_t*)(B_ptr_local + ax0_ax1_fused_0 * row_stride * (OC / 8));
      uint4 B_loaded_fp16 = dequantize_s4_to_fp16x2(B_loaded);
      //uint4 B_loaded_zero = *(uint4*)(zeros_shared + (threadIdx.x % (cta_N / 8)) * 8);

      // uint4 B_loaded_scale = *(uint4*)(scaling_factors_shared + (threadIdx.x % (cta_N / 8)) * 8);
      // - zero and * scale
      // TODO (Haotian): can save 4 assembly instructions if sormulate as deq = q * scale - zero * scale.
      asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.x) : "r"(B_loaded_fp16.x), "r"(B_loaded_zero.x));
      asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.x) : "r"(B_loaded_fp16.x), "r"(B_loaded_scale.x), "r"(ZERO));
      asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.y) : "r"(B_loaded_fp16.y), "r"(B_loaded_zero.y));
      asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.y) : "r"(B_loaded_fp16.y), "r"(B_loaded_scale.y), "r"(ZERO));
      asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.z) : "r"(B_loaded_fp16.z), "r"(B_loaded_zero.z));
      asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.z) : "r"(B_loaded_fp16.z), "r"(B_loaded_scale.z), "r"(ZERO));
      asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.w) : "r"(B_loaded_fp16.w), "r"(B_loaded_zero.w));
      asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.w) : "r"(B_loaded_fp16.w), "r"(B_loaded_scale.w), "r"(ZERO));
      /*
      if (ax0_ax1_fused_0 == 0 && blockIdx_z == 0 && blockIdx_y == 0 && k_0_0 == 0 && threadIdx.x == 17 && threadIdx.y == 0){
        printf("[x] %X %X %X %X\n", B_loaded_fp16.x, B_loaded_fp16.y, B_loaded_fp16.z, B_loaded_fp16.w);
      }
      */

      // write back
      *(uint4*)(B_shared_ptr + ax0_ax1_fused_0 * row_stride * (128 + 8)) = B_loaded_fp16;
    }
    __syncthreads();

    for (int k_0_1 = 0; k_0_1 < 2; ++k_0_1) {
      {
        unsigned int addr;
        __asm__ __volatile__(
          "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
          : "=r"(addr)
          : "l"((void *)((&(A_shared[(k_0_1 * 16)])) + (((((int)threadIdx.x) & 15) * 40) + ((((int)threadIdx.x) >> 4) * 8))))
        );


        __asm__ __volatile__(
          "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
          "{%0, %1, %2, %3}, [%4];\n"
          : "=r"(((unsigned *)(A_shared_warp + 0))[0]), "=r"(((unsigned *)(A_shared_warp + 0))[1]), "=r"(((unsigned *)(A_shared_warp + 0))[2]), "=r"(((unsigned *)(A_shared_warp + 0))[3])
          : "r"(addr)
        );
      }

      for (int ax1_0 = 0; ax1_0 < 4; ++ax1_0) {
        {
          unsigned int addr;
          __asm__ __volatile__(
            "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
            : "=r"(addr)
            : "l"((void *)((&(B_shared[(((k_0_1 * 2176) + (((int)threadIdx.y) * 64)) + (ax1_0 * 16))])) + (((((int)threadIdx.x) & 15) * 136) + ((((int)threadIdx.x) >> 4) * 8))))
          );
          __asm__ __volatile__(
            "ldmatrix.sync.aligned.m8n8.x4.trans.shared.b16"
            "{%0, %1, %2, %3}, [%4];\n"
            : "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[0]), "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[1]), "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[2]), "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[3])
            : "r"(addr)
          );
        }
      }
      for (int j_0_4 = 0; j_0_4 < 4; ++j_0_4) {
        {
          __asm__ __volatile__(
            "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
            "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
            :  "=f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[3])
            : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + (j_0_4 * 8)))[0]), "r"(((unsigned *)(B_shared_warp + (j_0_4 * 8)))[1]), "f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "f"(((float *)(C_warp + (j_0_4 * 8)))[3]));
        }

        {
          __asm__ __volatile__(
            "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
            "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
            :  "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3])
            : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + ((j_0_4 * 8) + 4)))[0]), "r"(((unsigned *)(B_shared_warp + ((j_0_4 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3]));
        }
      }
    }
  }

// TODO: Shang: Hoist loop invariance.
  for (int ax1_0_1 = 0; ax1_0_1 < 4; ++ax1_0_1) {
    for (int local_id = 0; local_id < 8; ++local_id) {
      int row_offset = (((int)blockIdx_y) / j_factors1) * 16 + ((int)threadIdx.x) / 4 + (local_id % 4) / 2 * 8;
      if (row_offset < M)
      {
        *(C_ptr + ax1_0_1 * 16 + row_offset * OC + (local_id / 4) * 8 + local_id % 2) = __float2half(C_warp[(ax1_0_1 * 8) + local_id]);
      }
    }
  }
}


__global__ void __launch_bounds__(64) gemm_forward_4bit_cuda_m16n64k32(int G, int split_k_iters, half* __restrict__ A, int* __restrict__ B, half* __restrict__ scaling_factors, int* __restrict__ zeros, int M, int IC, int OC, half* __restrict__ C) 
{
  static constexpr uint32_t ZERO = 0x0;
  float C_warp[32];
  __shared__ half A_shared[16 * (32 + 8)];
  __shared__ half B_shared[32 * (64 + 8)];
  
  __shared__ half scaling_factors_shared[64];
  __shared__ half zeros_shared[64];

  int j_factors1 = ((OC + 64 - 1) / 64);

  int blockIdx_x = 0;
  int blockIdx_y = blockIdx.x % ((M + 16 - 1) / 16 * j_factors1);
  int blockIdx_z = blockIdx.x / ((M + 16 - 1) / 16 * j_factors1);

  half A_shared_warp[8];
  half B_shared_warp[16];
  for (int j_0_4_init = 0; j_0_4_init < 2; ++j_0_4_init) {
    for (int i = 0; i < 8; ++i) {
      C_warp[(j_0_4_init * 8) + i] = 0.0;
    }
  }

  static constexpr int row_stride_warp = 32 * 8 / 32;
  static constexpr int row_stride = 2 * 32 * 8 / 64;
  bool ld_zero_flag = (threadIdx.y * 32 + threadIdx.x) * 8 < 64;
  // TODO: Haotian: blockIdx_y / j_factors1 in A loading to support bsz > 16
  bool ld_A_flag = (blockIdx_y / j_factors1 * 16 + threadIdx.y * row_stride_warp + threadIdx.x * 8 / 32) < M;     // threadIdx.y is warp_id
  // bool wb_C_flag = (threadIdx.x / 4) < M;

  half* A_ptr = A 
                + (((int)blockIdx_y) / j_factors1 * 16 + (((int)threadIdx.y) * row_stride_warp) + ((int)threadIdx.x) / (32 / 8)) * IC
                + (((int)threadIdx.x) % (32 / 8)) * 8;
  
  int* B_ptr = B
            + ((int)threadIdx.y) * (OC / 8) * 4
            + (((int)threadIdx.x) / (64 / 8)) * (OC / 8)
            + (((int)blockIdx_y) % j_factors1) * (64 / 8)
            + (((int)threadIdx.x) % (64 / 8)) * 1;
// Why * 1 in the above line?
                        
  half* A_shared_ptr = A_shared 
                    + ((int)threadIdx.y) * row_stride_warp * (32 + 8) 
                    + (((int)threadIdx.x) / (32 / 8)) * (32 + 8)
                    + (((int)threadIdx.x) % (32 / 8) ) * 8;

  half* B_shared_ptr = B_shared
                    + ((int)threadIdx.y) * (row_stride / 2) * (64 + 8)
                    + (((int)threadIdx.x) / (64 / 8)) * (64 + 8)
                    + (((int)threadIdx.x) % (64 / 8)) * 8;
  
  int* zeros_ptr = zeros
                + (((int)blockIdx_y) % j_factors1) * (64 / 8)
                + ((int)threadIdx.x) % (64 / 8);
  
  half* scaling_factors_ptr = scaling_factors
                            + (((int)blockIdx_y) % j_factors1) * (64) 
                            + (((int)threadIdx.x) % (64 / 8)) * 8;

  half* C_ptr = C 
              + blockIdx_z * M * OC        // blockIdz.x -> split_k dim
              + (((int)blockIdx_y) % j_factors1) * 64
              + ((int)threadIdx.y) * 32
              + (((int)threadIdx.x) % 4) * 2;

  // preload s.f. and zeros
  int k_bound = (IC / 32 + split_k_iters - 1) / split_k_iters;
  if ((k_bound - 1) * split_k_iters * 32 + blockIdx_z * 32 >= IC) k_bound -= 1;
  for (int _k_0_0 = 0; _k_0_0 < k_bound; ++_k_0_0) {
    int k_0_0 = _k_0_0 * split_k_iters + blockIdx_z;
    __syncthreads();
    // TODO: Haotian: blockIdx_y / j_factors1 in A loading to support bsz > 16
    if (ld_A_flag)
    {
      *(uint4*)(A_shared_ptr) = *(uint4*)(A_ptr + (k_0_0 * 32));
    }
    else
    {
      *(uint4*)(A_shared_ptr) = make_uint4(0, 0, 0, 0);
    }

    // for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 2; ++ax0_ax1_fused_0) {
    uint32_t zeros_loaded = *(uint32_t*)(zeros_ptr + k_0_0 * 32 / G * (OC / 8));
    uint4 B_loaded_zero = dequantize_s4_to_fp16x2(zeros_loaded);
    uint4 B_loaded_scale = *(uint4*)(scaling_factors_ptr + k_0_0 * 32 / G * (OC));
    /*
    if (blockIdx_z == 0 && blockIdx_y == 0 && k_0_0 == 0 && threadIdx.x == 0 && threadIdx.y == 0){
      printf("%x %x %x %x %x %x %x %x\n", B_loaded_scale.x, B_loaded_scale.y, B_loaded_scale.z, B_loaded_scale.w, B_loaded_zero.x, B_loaded_zero.y, B_loaded_zero.z, B_loaded_zero.w);
    }
    */
    // uint4 B_loaded_scale = make_uint4(0, 0, 0, 0);
    int* B_ptr_local = B_ptr + k_0_0 * 32 * (OC / 8);

    for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 4; ++ax0_ax1_fused_0) {

      // B: 32 x 136 (128+8) float16
      // each warp: 32 x 4
      // each thr: read 32 bit -> convert to 8xFP16 (a UINT4) -> scale and minus zero -> WB UINT4
      // *(uint4*)(B_shared + ((((ax0_ax1_fused_0 * 544) + (((int)threadIdx.y) * 272)) + ((((int)threadIdx.x) >> 4) * 136)) + ((((int)threadIdx.x) & 15) * 8))) = *(uint4*)(B + ((((((k_0_0 * 163840) + (ax0_ax1_fused_0 * 20480)) + (((int)threadIdx.y) * 10240)) + ((((int)threadIdx.x) >> 4) * 5120)) + (((int)blockIdx_y) * 128)) + ((((int)threadIdx.x) & 15) * 8)));
      // row stride in shared memory: (NWARPS * 32 * 8 / cta_N) 
      uint32_t B_loaded = *(uint32_t*)(B_ptr_local + ax0_ax1_fused_0 * row_stride * (OC / 8));
      uint4 B_loaded_fp16 = dequantize_s4_to_fp16x2(B_loaded);
      //uint4 B_loaded_zero = *(uint4*)(zeros_shared + (threadIdx.x % (cta_N / 8)) * 8);

      // uint4 B_loaded_scale = *(uint4*)(scaling_factors_shared + (threadIdx.x % (cta_N / 8)) * 8);
      // - zero and * scale
      // TODO (Haotian): can save 4 assembly instructions if sormulate as deq = q * scale - zero * scale.
      asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.x) : "r"(B_loaded_fp16.x), "r"(B_loaded_zero.x));
      asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.x) : "r"(B_loaded_fp16.x), "r"(B_loaded_scale.x), "r"(ZERO));
      asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.y) : "r"(B_loaded_fp16.y), "r"(B_loaded_zero.y));
      asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.y) : "r"(B_loaded_fp16.y), "r"(B_loaded_scale.y), "r"(ZERO));
      asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.z) : "r"(B_loaded_fp16.z), "r"(B_loaded_zero.z));
      asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.z) : "r"(B_loaded_fp16.z), "r"(B_loaded_scale.z), "r"(ZERO));
      asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.w) : "r"(B_loaded_fp16.w), "r"(B_loaded_zero.w));
      asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.w) : "r"(B_loaded_fp16.w), "r"(B_loaded_scale.w), "r"(ZERO));
      /*
      if (ax0_ax1_fused_0 == 0 && blockIdx_z == 0 && blockIdx_y == 0 && k_0_0 == 0 && threadIdx.x == 17 && threadIdx.y == 0){
        printf("[x] %X %X %X %X\n", B_loaded_fp16.x, B_loaded_fp16.y, B_loaded_fp16.z, B_loaded_fp16.w);
      }
      */

      // write back
      *(uint4*)(B_shared_ptr + ax0_ax1_fused_0 * row_stride * (64 + 8)) = B_loaded_fp16;
    }
    __syncthreads();

    for (int k_0_1 = 0; k_0_1 < 2; ++k_0_1) 
    {
      {
        unsigned int addr;
        __asm__ __volatile__(
          "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
          : "=r"(addr)
          : "l"((void *)((&(A_shared[(k_0_1 * 16)])) + (((((int)threadIdx.x) & 15) * 40) + ((((int)threadIdx.x) >> 4) * 8))))
        );
        __asm__ __volatile__(
          "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
          "{%0, %1, %2, %3}, [%4];\n"
          : "=r"(((unsigned *)(A_shared_warp + 0))[0]), "=r"(((unsigned *)(A_shared_warp + 0))[1]), "=r"(((unsigned *)(A_shared_warp + 0))[2]), "=r"(((unsigned *)(A_shared_warp + 0))[3])
          : "r"(addr)
        );
      }
        

      for (int ax1_0 = 0; ax1_0 < 2; ++ax1_0) 
      {
        {
          unsigned int addr;
          __asm__ __volatile__(
            "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
            : "=r"(addr)
            : "l"((void *)((&(B_shared[(((k_0_1 * 1152) + (((int)threadIdx.y) * 32)) + (ax1_0 * 16))])) + (((((int)threadIdx.x) & 15) * 72) + ((((int)threadIdx.x) >> 4) * 8))))
          );
          __asm__ __volatile__(
            "ldmatrix.sync.aligned.m8n8.x4.trans.shared.b16"
            "{%0, %1, %2, %3}, [%4];\n"
            : "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[0]), "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[1]), "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[2]), "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[3])
            : "r"(addr)
          );
        }
      }
      
      for (int j_0_4 = 0; j_0_4 < 2; ++j_0_4) 
      {

        {
          __asm__ __volatile__(
            "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
            "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
            :  "=f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[3])
            : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + (j_0_4 * 8)))[0]), "r"(((unsigned *)(B_shared_warp + (j_0_4 * 8)))[1]), "f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "f"(((float *)(C_warp + (j_0_4 * 8)))[3]));
        }

        {
          __asm__ __volatile__(
            "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
            "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
            :  "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3])
            : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + ((j_0_4 * 8) + 4)))[0]), "r"(((unsigned *)(B_shared_warp + ((j_0_4 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3]));
        }
      }
    }
  }

// TODO: Shang: Hoist loop invariance.
  for (int ax1_0_1 = 0; ax1_0_1 < 2; ++ax1_0_1) {
    for (int local_id = 0; local_id < 8; ++local_id) {
      int row_offset = (((int)blockIdx_y) / j_factors1) * 16 + ((int)threadIdx.x) / 4 + (local_id % 4) / 2 * 8;
      if (row_offset < M)
      {
        *(C_ptr + ax1_0_1 * 16 + row_offset * OC + (local_id / 4) * 8 + local_id % 2) = __float2half(C_warp[(ax1_0_1 * 8) + local_id]);
      }
    }
  }
}

// in_feats: M, IC [float16]
// kernel: IC, OC // 8 [int32] -> cast to IC, OC [uint4b]
// scaling_factors: IC // G, OC [float16]
// zeros: IC // G, OC // 8 [int32] -> cast to IC // G, OC [uint4b]
// assume that batch_size < 16 for now

// torch::Tensor gemm_forward_cuda_origin(
//   torch::Tensor _in_feats,
//   torch::Tensor _kernel,
//   torch::Tensor _scaling_factors,
//   torch::Tensor _zeros,
//   int split_k_iters)
// {
//   int num_in_feats = _in_feats.size(0);
//   int num_in_channels = _in_feats.size(1);
//   const at::cuda::OptionalCUDAGuard device_guard(device_of(_in_feats));

//   auto options = torch::TensorOptions().dtype(_in_feats.dtype()).device(_in_feats.device());
//   at::Tensor _out_feats = torch::empty({split_k_iters, num_in_feats, _kernel.size(1) * 8}, options);
//   int num_out_feats = _out_feats.size(-2);
//   int num_out_channels = _out_feats.size(-1);

//   auto in_feats = reinterpret_cast<half*>(_in_feats.data_ptr<at::Half>());
//   auto kernel = reinterpret_cast<int*>(_kernel.data_ptr<int>());
//   auto out_feats = reinterpret_cast<half*>(_out_feats.data_ptr<at::Half>());
//   auto scaling_factors = reinterpret_cast<half*>(_scaling_factors.data_ptr<at::Half>());
//   auto zeros = reinterpret_cast<int*>(_zeros.data_ptr<int>());
//   int group_size = num_in_channels / _scaling_factors.size(0);

//   // std::cout << "_kernel: " << _kernel << std::endl;

//   if (num_out_channels % 64 != 0)
//     throw std::invalid_argument("OC is not multiple of cta_N = 64");
//   if (num_out_channels % 8 != 0)
//     throw std::invalid_argument("OC is not multiple of pack_num = 8");
//   if (group_size % 32 != 0)
//     throw std::invalid_argument("Group size should be a multiple of 32");
//   if (num_out_channels % group_size != 0)
//     throw std::invalid_argument("OC is not multiple of Group size");

//   if (num_out_channels % 128 == 0)
//   {
//     int j_factors1 = num_out_channels / 128 / 1;
//     dim3 num_blocks((num_out_feats + 16 - 1) / 16 * j_factors1 * split_k_iters);
//     // threadIdx.x: 32
//     // threadIdx.y: i_factors[2] * j_factors[2]
//     dim3 threads_per_block(32, 2);
    
//     // std::cout << "AAA group_size: " << group_size << std::endl;

//     gemm_forward_4bit_cuda_m16n128k32<<<num_blocks, threads_per_block>>>(
//         group_size, split_k_iters, in_feats, kernel, scaling_factors, zeros, num_in_feats, num_in_channels, num_out_channels, out_feats);
//   }
//   else if (num_out_channels % 64 == 0)
//   {
//     int j_factors1 = num_out_channels / 64 / 1;
//     dim3 num_blocks(1 * (num_out_feats + 16 - 1) / 16 * j_factors1 * split_k_iters);

//     // threadIdx.x: 32
//     // threadIdx.y: i_factors[2] * j_factors[2]
//     dim3 threads_per_block(32, 2);

//     // std::cout << "BBB group_size: " << group_size << std::endl;

//     gemm_forward_4bit_cuda_m16n64k32<<<num_blocks, threads_per_block>>>(
//         group_size, split_k_iters, in_feats, kernel, scaling_factors, zeros, num_in_feats, num_in_channels, num_out_channels, out_feats);
//   }

//   return _out_feats.to(torch::kFloat32).sum(0);
// }


__global__ void my_add(half* data_0, half* data_1, half* data_2, int num_elements)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_elements)
  {
    data_2[idx] = data_0[idx] + data_1[idx];
  }
}

// in_feats: M, IC [float16]
// kernel: IC, OC // 8 [int32] -> cast to IC, OC [uint4b]
// scaling_factors: IC // G, OC [float16]
// zeros: IC // G, OC // 8 [int32] -> cast to IC // G, OC [uint4b]
// assume that batch_size < 16 for now

void gemm_forward_cuda(const struct matmul_params *params, int split_k_iters)
{
	const struct matrix *A = &params->A, *B = &params->B, *C = &params->C;

  int num_in_feats = A->row;
  int num_in_channels = A->column;
  int num_out_feats = C->row;
  int num_out_channels = C->column;

  // auto in_feats = (half*)A->data_ptr;
  // auto kernel = B->int32_data_ptr;
  // auto out_feats = (half*)C->data_ptr;
  // auto scaling_factors = (half*)params->scales;
  // auto zeros = params->int32_zero_point;
  // int group_size = QK;

  // half* in_feats = (half*)A->data_ptr;
  int* kernel = B->int32_data_ptr;
  // half* out_feats = (half*)C->data_ptr;
  half* scaling_factors = params->scales_fp16;
  int* zeros = params->int32_zero_point;
  int group_size = QK;

  half* in_feats;
  // int* kernel;
  half* out_feats;
  // half* scaling_factors;
  // int* zeros;

  // // Allocate device memory
  int A_size = A->row * A->column;
  // printf("A_size: %d\n", A_size);
  // printf("A->row: %d\n", A->row);
  // printf("A->column: %d\n", A->column);
  int C_size = C->row * C->column;
  // printf("C_size: %d\n", C_size);
  // printf("C->row: %d\n", C->row);
  // printf("C->column: %d\n", C->column);
  int sf_size = B->row / group_size * B->column * 8;
  // printf("sf_size: %d\n", sf_size);
  // printf("B->row: %d\n", B->row);
  // printf("B->column: %d\n", B->column);

  hipError_t err;
  err = hipMallocManaged(&in_feats, A_size * sizeof(half));
  err = hipMallocManaged(&out_feats, split_k_iters * C_size * sizeof(half));

  // Launch the kernel
  int blockSize = 256;
  int numBlocks = (A_size + blockSize - 1) / blockSize;

  PROFILE_START("float2half::in_feats");
  float2half<<<numBlocks, blockSize>>>(A->data_ptr, in_feats, A_size);
  // hipDeviceSynchronize();
  // for (int i = 0; i < A_size; i++) {
  //   in_feats[i] = __float2half(A->data_ptr[i]);
  // }
  // err = hipGetLastError();
  // if (err != hipSuccess) {
  //   printf("Error launching float2half kernel for in_feats: %s\n", hipGetErrorString(err));
  // }
  PROFILE_END("float2half::in_feats");

  if (num_out_channels % 64 != 0)
    throw std::invalid_argument("OC is not multiple of cta_N = 64");
  if (num_out_channels % 8 != 0)
    throw std::invalid_argument("OC is not multiple of pack_num = 8");
  if (group_size % 32 != 0)
    throw std::invalid_argument("Group size should be a multiple of 32");
  if (num_out_channels % group_size != 0)
    throw std::invalid_argument("OC is not multiple of Group size");

  PROFILE_START("gemm_forward_4bit_cuda_m16n128k32");
  if (num_out_channels % 128 == 0)
  {
    int j_factors1 = num_out_channels / 128 / 1;
    dim3 num_blocks((num_out_feats + 16 - 1) / 16 * j_factors1 * split_k_iters);
    // threadIdx.x: 32
    // threadIdx.y: i_factors[2] * j_factors[2]
    dim3 threads_per_block(32, 2);
    
    gemm_forward_4bit_cuda_m16n128k32<<<num_blocks, threads_per_block>>>(
        group_size, split_k_iters, in_feats, kernel, scaling_factors, zeros, num_in_feats, num_in_channels, num_out_channels, out_feats);
  }
  else if (num_out_channels % 64 == 0)
  {
    int j_factors1 = num_out_channels / 64 / 1;
    dim3 num_blocks(1 * (num_out_feats + 16 - 1) / 16 * j_factors1 * split_k_iters);

    // threadIdx.x: 32
    // threadIdx.y: i_factors[2] * j_factors[2]
    dim3 threads_per_block(32, 2);

    gemm_forward_4bit_cuda_m16n64k32<<<num_blocks, threads_per_block>>>(
        group_size, split_k_iters, in_feats, kernel, scaling_factors, zeros, num_in_feats, num_in_channels, num_out_channels, out_feats);
  }
  PROFILE_END("gemm_forward_4bit_cuda_m16n128k32");

  numBlocks = (C_size + blockSize - 1) / blockSize;

  PROFILE_START("half2float_merge_k_iters");
  half2float_merge_k_iters<<<numBlocks, blockSize>>>(out_feats, C->data_ptr, C_size, split_k_iters);
  hipDeviceSynchronize();
  // for (int i = 0; i < C_size; i++) {
  //   C->data_ptr[i] = 0;
  //   for (int j = 0; j < split_k_iters; j++) {
  //     C->data_ptr[i] += __half2float(out_feats[i + j * C_size]);
  //   }
  // }
  PROFILE_END("half2float_merge_k_iters");

  // Free memory
  PROFILE_START("hipFree");
  hipFree(in_feats);
  hipFree(out_feats);
  PROFILE_END("hipFree");


  // float weight;
  // for (int i = 0; i < C->row; i++) {
  //   for (int j = 0; j < C->column; j++) {
  //     float acc = 0;

  //     for (int k = 0; k < B->row; k++) {
  //       float s = __half2float(params->scales_fp16[(k / 32) * C->column + j]);
  //       float input = A->data_ptr[i * A->column + k];

  //       if (j % 8 == 0)
  //         weight = ((B->int32_data_ptr[k * B->column + (j / 8)] & 0x0000000F) - 8.0) * s;
  //       else if (j % 8 == 1)
  //         weight = (((B->int32_data_ptr[k * B->column + (j / 8)] & 0x000000F0) >> 4) - 8.0) * s;
  //       else if (j % 8 == 2)
  //         weight = (((B->int32_data_ptr[k * B->column + (j / 8)] & 0x00000F00) >> 8) - 8.0) * s;
  //       else if (j % 8 == 3)
  //         weight = (((B->int32_data_ptr[k * B->column + (j / 8)] & 0x0000F000) >> 12) - 8.0) * s;
  //       else if (j % 8 == 4)
  //         weight = (((B->int32_data_ptr[k * B->column + (j / 8)] & 0x000F0000) >> 16) - 8.0) * s;
  //       else if (j % 8 == 5)
  //         weight = (((B->int32_data_ptr[k * B->column + (j / 8)] & 0x00F00000) >> 20) - 8.0) * s;
  //       else if (j % 8 == 6)
  //         weight = (((B->int32_data_ptr[k * B->column + (j / 8)] & 0x0F000000) >> 24) - 8.0) * s;
  //       else if (j % 8 == 7)
  //         weight = (((B->int32_data_ptr[k * B->column + (j / 8)] & 0xF0000000) >> 28) - 8.0) * s;

  //       acc += input * weight;
  //     }
      
  //     C->data_ptr[i * C->column + j] = acc;
  //   }
  // }
}


namespace matmul{
  // in_feats: M, IC [float16]
  // kernel: IC, OC // 8 [int32] -> cast to IC, OC [uint4b]
  // scaling_factors: IC // G, OC [float16]
  // zeros: IC // G, OC // 8 [int32] -> cast to IC // G, OC [uint4b]
  // assume that batch_size < 16 for now

  void MatmulOperator::gemm_forward_cuda_half(const struct matmul_params *params, int split_k_iters)
  {
    const struct matrix *A = &params->A, *B = &params->B, *C = &params->C;

    int num_in_feats = A->row;
    int num_in_channels = A->column;
    int num_out_feats = C->row;
    int num_out_channels = C->column;

    // half* in_feats = (half*)A->data_ptr;
    int* kernel = B->int32_data_ptr;
    // half* out_feats = (half*)C->data_ptr;
    half* scaling_factors = params->scales_fp16;
    int* zeros = params->int32_zero_point;
    int group_size = QK;

    half* in_feats;
    // int* kernel;
    half* out_feats = C->fp16_data_ptr;
    // half* scaling_factors;
    // int* zeros;

    // // Allocate device memory
    int A_size = A->row * A->column;
    // printf("A_size: %d\n", A_size);
    // printf("A->row: %d\n", A->row);
    // printf("A->column: %d\n", A->column);
    int C_size = C->row * C->column;
    // printf("C_size: %d\n", C_size);
    // printf("C->row: %d\n", C->row);
    // printf("C->column: %d\n", C->column);
    int sf_size = B->row / group_size * B->column * 8;
    // printf("sf_size: %d\n", sf_size);
    // printf("B->row: %d\n", B->row);
    // printf("B->column: %d\n", B->column);

    hipError_t err;
    err = hipMallocManaged(&in_feats, A_size * sizeof(half));
    // err = hipMallocManaged(&out_feats, split_k_iters * C_size * sizeof(half));

    // Launch the kernel
    int blockSize = 256;
    int numBlocks = (A_size + blockSize - 1) / blockSize;

    PROFILE_START("float2half::in_feats");
    float2half<<<numBlocks, blockSize>>>(A->data_ptr, in_feats, A_size);
    PROFILE_END("float2half::in_feats");

    if (num_out_channels % 64 != 0)
      throw std::invalid_argument("OC is not multiple of cta_N = 64");
    if (num_out_channels % 8 != 0)
      throw std::invalid_argument("OC is not multiple of pack_num = 8");
    if (group_size % 32 != 0)
      throw std::invalid_argument("Group size should be a multiple of 32");
    if (num_out_channels % group_size != 0)
      throw std::invalid_argument("OC is not multiple of Group size");

    PROFILE_START("gemm_forward_4bit_cuda_m16n128k32");
    if (num_out_channels % 128 == 0)
    {
      int j_factors1 = num_out_channels / 128 / 1;
      dim3 num_blocks((num_out_feats + 16 - 1) / 16 * j_factors1 * split_k_iters);
      // threadIdx.x: 32
      // threadIdx.y: i_factors[2] * j_factors[2]
      dim3 threads_per_block(32, 2);
      
      gemm_forward_4bit_cuda_m16n128k32<<<num_blocks, threads_per_block>>>(
          group_size, split_k_iters, in_feats, kernel, scaling_factors, zeros, num_in_feats, num_in_channels, num_out_channels, out_feats);
    }
    else if (num_out_channels % 64 == 0)
    {
      int j_factors1 = num_out_channels / 64 / 1;
      dim3 num_blocks(1 * (num_out_feats + 16 - 1) / 16 * j_factors1 * split_k_iters);

      // threadIdx.x: 32
      // threadIdx.y: i_factors[2] * j_factors[2]
      dim3 threads_per_block(32, 2);

      gemm_forward_4bit_cuda_m16n64k32<<<num_blocks, threads_per_block>>>(
          group_size, split_k_iters, in_feats, kernel, scaling_factors, zeros, num_in_feats, num_in_channels, num_out_channels, out_feats);
    }
    PROFILE_END("gemm_forward_4bit_cuda_m16n128k32");

    // numBlocks = (C_size + blockSize - 1) / blockSize;

    PROFILE_START("half2float_merge_k_iters");
    // half2float_merge_k_iters<<<numBlocks, blockSize>>>(out_feats, C->data_ptr, C_size, split_k_iters);
    hipDeviceSynchronize();
    PROFILE_END("half2float_merge_k_iters");

    // Free memory
    PROFILE_START("hipFree");
    hipFree(in_feats);
    // hipFree(out_feats);
    PROFILE_END("hipFree");
  }

  void MatmulOperator::mat_mul_accelerator_int4_fast(const struct matmul_params *params) {
		// const struct matrix *A = &params->A, *B = &params->B, *C = &params->C;

    // // std::cout << "mat_mul_accelerator_int4_fast -- A->row: " << A->row << " A->column: " << A->column 
    //           // << "; B->row: " << B->row << " B->column: " << B->column 
    //           // << "; C->row: " << C->row << " C->column: " << C->column << std::endl;
    
    // // torch::Tensor out_feats = gemm_forward_cuda(
    // //     torch::from_blob(A->data_ptr, {A->row, A->column}, torch::kHalf),
    // //     torch::from_blob(B->data_ptr, {B->row, B->column}, torch::kInt),
    // //     torch::from_blob(params->scales, {B->row / 128, B->column * 8}, torch::kHalf),
    // //     torch::from_blob(params->int32_zero_point, {B->row / 128, B->column}, torch::kInt),
    // //     8);
    
    // auto option_fp = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, 1);
    // auto option_int = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA, 1);
    // torch::Tensor _in_feats = torch::from_blob(A->data_ptr, {A->row, A->column}, option_fp);
    // torch::Tensor _kernel = torch::from_blob(B->data_ptr, {B->row, B->column}, option_int);
    // torch::Tensor _scaling_factors = torch::from_blob(params->scales, {B->row / 128, B->column * 8}, option_fp);
    // torch::Tensor _zeros = torch::from_blob(params->int32_zero_point, {B->row / 128, B->column}, option_int);

    // torch::Tensor out_feats = gemm_forward_cuda(
    //     _in_feats,
    //     _kernel,
    //     _scaling_factors,
    //     _zeros,
    //     8);
    
    // hipMemcpy(C->data_ptr, out_feats.data_ptr(), C->column * C->row * sizeof(float), hipMemcpyDeviceToHost);
  };

  void MatmulOperator::mat_mul_accelerator_int4_fast_no_offset(const struct matmul_params *params) {
		// const struct matrix *A = &params->A, *B = &params->B, *C = &params->C;

    // std::cout << "mat_mul_accelerator_int4_fast_no_offset -- A->row: " << A->row << " A->column: " << A->column 
              // << "; B->row: " << B->row << " B->column: " << B->column 
              // << "; C->row: " << C->row << " C->column: " << C->column << std::endl;
    
    // std::cout << "AAAAA" << std::endl;
    
    // torch::Tensor out_feats = gemm_forward_cuda(
    //     torch::from_blob(A->data_ptr, {A->row, A->column}, torch::kHalf),
    //     torch::from_blob(B->data_ptr, {B->row, B->column}, torch::kInt),
    //     torch::from_blob(params->scales, {B->row / 128, B->column * 8}, torch::kHalf),
    //     torch::from_blob(params->int32_zero_point, {B->row / 128, B->column}, torch::kInt),
    //     8);
    
    // auto option_fp = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, 1);
    // auto option_int = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA, 1);

    // std::cout << "jcuycu" << std::endl;
    //torch::Tensor _in_feats = torch::from_blob(A->data_ptr, {A->row, A->column}, option_fp);
    
    // torch::Tensor _in_feats = torch::from_blob(A->data_ptr, {A->row, A->column}, torch::kFloat32).to(torch::kCUDA).to(torch::kHalf);
    
    // std::cout << "kchy" << std::endl;
    // torch::Tensor _kernel = torch::from_blob(B->int32_data_ptr, {B->row, B->column}, option_int);
    
    // torch::Tensor _kernel = torch::from_blob(B->int32_data_ptr, {B->row, B->column}, torch::kInt32).to(torch::kCUDA);

    // for (int i = 0; i < 128; i++) {
    //   std::cout << B->int32_data_ptr[i] << " ";
    // }
    // std::cout << std::endl;
    

    // std::cout << "qued" << std::endl;
    // torch::Tensor _scaling_factors = torch::from_blob(params->scales, {B->row / 128, B->column * 8}, option_fp);
    
    // torch::Tensor _scaling_factors = torch::from_blob(params->scales, {B->row / QK, B->column * 8}, torch::kFloat32).to(torch::kCUDA).to(torch::kHalf);
    
    // for (int i = 0; i < 128; i++) { 
    //   std::cout << params->scales[i] << " ";
    // }
    // std::cout << std::endl;
    
    // std::cout << "-cud" << std::endl;
    // torch::Tensor _zeros = torch::from_blob(params->int32_zero_point, {B->row / 128, B->column}, option_int);
    
    // torch::Tensor _zeros = torch::from_blob(params->int32_zero_point, {B->row / QK, B->column}, torch::kInt32).to(torch::kCUDA);
    
    // for (int i = 0; i < 128; i++) { 
    //   std::cout << params->int32_zero_point[i] << " ";
    // }
    // std::cout << std::endl;
    
    // std::cout << "msadi" << std::endl;

    gemm_forward_cuda(params, 1);
    
    // hipMemcpy(C->data_ptr, out_feats.data_ptr(), C->column * C->row * sizeof(float), hipMemcpyDeviceToHost);


    // // Testing (Naive implementation)
		// const struct matrix *A = &params->A, *B = &params->B, *C = &params->C;
		// assert(A->column == B->row / QK);
		// assert(C->column == B->column / 8);
		// assert(C->row == A->row);

		// float *d_A;
		// float *d_B;
		// float *d_C;

		// // Allocate memory
		// hipMalloc(&d_A, A->column*A->row*sizeof(float));
		// hipMalloc(&d_B, B->column*B->row*sizeof(float));
		// hipMalloc(&d_C, C->column*C->row*sizeof(float));

		// // Copy data to GPU
		// hipMemcpy(d_A, A->data_ptr, A->column*A->row*sizeof(float), hipMemcpyHostToDevice);
		// hipMemcpy(d_B, B->data_ptr, B->column*B->row*sizeof(float), hipMemcpyHostToDevice);
		// hipMemcpy(d_C, C->data_ptr, C->column*C->row*sizeof(float), hipMemcpyHostToDevice);

    // matrixMul_block_zp<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, A->row, A->column, C->column);
  };

	void MatmulOperator::mat_mul_cuda(const struct matmul_params *params){
		const struct matrix *A = &params->A, *B = &params->B, *C = &params->C;
		assert(A->column == B->row);
		assert(C->column == B->column);
		assert(C->row == A->row);

		float *d_A;
		float *d_B;
		float *d_C;

		// Initailize C
		/*for (int i = 0; i < C->row; i++)
		  for (int j = 0; j < C->column; j++)
		  C->data_ptr[j + C->column * i] = 0;*/

		// Allocate memory
		hipMalloc(&d_A, A->column*A->row*sizeof(float));
		hipMalloc(&d_B, B->column*B->row*sizeof(float));
		hipMalloc(&d_C, C->column*C->row*sizeof(float));

		// Copy data to GPU
		hipMemcpy(d_A, A->data_ptr, A->column*A->row*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_B, B->data_ptr, B->column*B->row*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_C, C->data_ptr, C->column*C->row*sizeof(float), hipMemcpyHostToDevice);

		// Make sure we can break the input matrix into blocks
		assert(A->column % threadDim == 0);
		assert(A->row % threadDim == 0);
		assert(B->column % threadDim == 0);
		const dim3 threadsPerBlock(threadDim, threadDim);
    const dim3 numBlocks((C->column + threadDim - 1) / threadsPerBlock.x, 
                         (C->row + threadDim - 1) / threadsPerBlock.y);  // Using ceil to cover all elements

		// Invoke the cuda imp.

		// struct timeval start, end;
		// gettimeofday(&start, NULL);
		// matrixMul_blockC<<< numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, A->row, A->column, B->column);
		matrixMultiplyShared<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, A->row, A->column, B->column);
		hipDeviceSynchronize();
		// gettimeofday(&end, NULL);
		// int us = interval_to_us(&start, &end);
		// // std::cout << "cuda kernel: " << us / 1000 << " ms" << std::endl;

		// Get the result back
		hipMemcpy(C->data_ptr, d_C, C->column*C->row*sizeof(float), hipMemcpyDeviceToHost);
	}
}  // namespace matmul





////////// HERE

// #include <assert.h>
// #include <sys/time.h>
// #include <stdlib.h>
// #include <cstdlib>
// #include <iostream>

// #include "matmul.h"
// #include "linear.h"

// // #include <torch/torch.h>
// #include <hip/hip_runtime.h>
// // #include <torch/extension.h>
// #include <hip/hip_fp16.h>
// // #include <c10/cuda/CUDAGuard.h>
// #include "gemm_cuda.h"
// #include "dequantize.cuh"

// __global__ void float2half(float* floatArray, half* halfArray, int N) {
//     int index = threadIdx.x + blockIdx.x * blockDim.x;
//     if (index < N) {
//         halfArray[index] = __float2half(floatArray[index]);
//     }
// }

// __global__ void half2float(half* halfArray, float* floatArray, int N) {
//     int index = threadIdx.x + blockIdx.x * blockDim.x;
//     if (index < N) {
//         floatArray[index] = __half2float(halfArray[index]);
//     }
// }

// __global__ void half2float_merge_k_iters(half *halfArray, float *floatArray, int N, int split_k_iters) {
//     int index = threadIdx.x + blockIdx.x * blockDim.x;

//     if (index < N) {
//         float sum = 0;
//         for (int j = 0; j < split_k_iters; j++) {
//             sum += __half2float(halfArray[index + j * N]);
//         }
//         floatArray[index] = sum;
//     }
// }

// const int threadDim = 32;
// const int TILE_SIZE = threadDim;

// __global__ void matrixMul_blockC(float *A, float *B, float *C, int A_row, int A_column, int B_column){
// 	int i = blockIdx.x * blockDim.x + threadIdx.x;
// 	int j = blockIdx.y * blockDim.y + threadIdx.y;

// 	float acc = 0;
//   if(i < B_column && j < A_row) {
//     for (int k = 0; k < A_column; k++)
//       acc += A[j * A_column + k] * B[k * B_column + i];
//     C[j * B_column +i] = acc;
//   }
// }

// __global__ void matrixMultiplyShared(const float *A, const float *B, float *C, int A_row, int A_column, int B_column) {
// 	int row = blockIdx.y * blockDim.y + threadIdx.y;
// 	int col = blockIdx.x * blockDim.x + threadIdx.x;

// 	__shared__ float As[TILE_SIZE][TILE_SIZE];
// 	__shared__ float Bs[TILE_SIZE][TILE_SIZE];

// 	float value = 0;

// 	for (int i = 0; i < A_column / TILE_SIZE; i++){
// 		As[threadIdx.y][threadIdx.x] = A[(blockIdx.y * TILE_SIZE + threadIdx.y) * A_column + TILE_SIZE * i + threadIdx.x];
// 		Bs[threadIdx.y][threadIdx.x] = B[(i * TILE_SIZE + threadIdx.y) * B_column + blockIdx.x * TILE_SIZE + threadIdx.x];

// 		__syncthreads();

// 		for (int k = 0; k < TILE_SIZE; k++)
// 			value += As[threadIdx.y][k] * Bs[k][threadIdx.x];

// 		__syncthreads();
// 	}


// 	C[row * B_column + col] = value;
// }


// /* AWQ Implementation */

// // Pack two half values.
// static inline __device__ __host__ unsigned
// __pack_half2(const half x, const half y) {
//   unsigned v0 = *((unsigned short *)&x);
//   unsigned v1 = *((unsigned short *)&y);
//   return (v1 << 16) | v0;
// }

// __global__ void __launch_bounds__(64) gemm_forward_4bit_cuda_m16n128k32(int G, int split_k_iters, half* __restrict__ A, int* __restrict__ B, half* __restrict__ scaling_factors, int* __restrict__ zeros, int M, int IC, int OC, half* __restrict__ C) 
// {
//   static constexpr uint32_t ZERO = 0x0;
//   float C_warp[32];
//   __shared__ half A_shared[16 * (32 + 8)];
//   __shared__ half B_shared[32 * (128 + 8)];
  
//   __shared__ half scaling_factors_shared[128];
//   __shared__ half zeros_shared[128];

//   int j_factors1 = ((OC + 128 - 1) / 128);
//   int blockIdx_x = 0;
//   int blockIdx_y = blockIdx.x % ((M + 16 - 1) / 16 * j_factors1);
//   int blockIdx_z = blockIdx.x / ((M + 16 - 1) / 16 * j_factors1);

//   half A_shared_warp[8];
//   half B_shared_warp[32];
//   for (int j_0_4_init = 0; j_0_4_init < 4; ++j_0_4_init) {
//     for (int i = 0; i < 8; ++i) {
//       C_warp[(j_0_4_init * 8) + i] = 0.0;
//     }
//   }

//   static constexpr int row_stride_warp = 32 * 8 / 32;
//   static constexpr int row_stride = 2 * 32 * 8 / 128;
//   bool ld_zero_flag = (threadIdx.y * 32 + threadIdx.x) * 8 < 128;
//   // TODO: Haotian: blockIdx_y / j_factors1 in A loading to support bsz > 16
//   bool ld_A_flag = (blockIdx_y / j_factors1 * 16 + threadIdx.y * row_stride_warp + threadIdx.x * 8 / 32) < M;     // threadIdx.y is warp_id
//   // bool wb_C_flag = (threadIdx.x / 4) < M;

//   half* A_ptr = A 
//                 + (((int)blockIdx_y) / j_factors1 * 16 + (((int)threadIdx.y) * row_stride_warp) + ((int)threadIdx.x) / (32 / 8)) * IC
//                 + (((int)threadIdx.x) % (32 / 8)) * 8;
  
//   int* B_ptr = B
//             + ((int)threadIdx.y) * (OC / 8) * 2
//             + (((int)threadIdx.x) / (128 / 8)) * (OC / 8)
//             + (((int)blockIdx_y) % j_factors1) * (128 / 8)
//             + (((int)threadIdx.x) % (128 / 8)) * 1;
// // Why * 1 in the above line?
                        
//   half* A_shared_ptr = A_shared 
//                     + ((int)threadIdx.y) * row_stride_warp * (32 + 8) 
//                     + (((int)threadIdx.x) / (32 / 8)) * (32 + 8)
//                     + (((int)threadIdx.x) % (32 / 8) ) * 8;

//   half* B_shared_ptr = B_shared
//                     + ((int)threadIdx.y) * (row_stride / 2) * (128 + 8)
//                     + (((int)threadIdx.x) / (128 / 8)) * (128 + 8)
//                     + (((int)threadIdx.x) % (128 / 8)) * 8;
  
//   int* zeros_ptr = zeros
//                 + (((int)blockIdx_y) % j_factors1) * (128 / 8)
//                 + ((int)threadIdx.x) % (128 / 8);
  
//   half* scaling_factors_ptr = scaling_factors
//                             + (((int)blockIdx_y) % j_factors1) * (128) 
//                             + (((int)threadIdx.x) % (128 / 8)) * 8;

//   half* C_ptr = C 
//               + blockIdx_z * M * OC        // blockIdz.x -> split_k dim
//               + (((int)blockIdx_y) % j_factors1) * 128
//               + ((int)threadIdx.y) * 64
//               + (((int)threadIdx.x) % 4) * 2;

//   // preload s.f. and zeros
//   int k_bound = (IC / 32 + split_k_iters - 1) / split_k_iters;
//   if ((k_bound - 1) * split_k_iters * 32 + blockIdx_z * 32 >= IC) k_bound -= 1;
//   for (int _k_0_0 = 0; _k_0_0 < k_bound; ++_k_0_0) {
//     int k_0_0 = _k_0_0 * split_k_iters + blockIdx_z;
//     __syncthreads();
//     // TODO: Haotian: blockIdx_y / j_factors1 in A loading to support bsz > 16
//     if (ld_A_flag)
//     {
//       *(uint4*)(A_shared_ptr) = *(uint4*)(A_ptr + (k_0_0 * 32));
//     }
//     else
//     {
//       *(uint4*)(A_shared_ptr) = make_uint4(0, 0, 0, 0);
//     }

//     // for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 2; ++ax0_ax1_fused_0) {
//     uint32_t zeros_loaded = *(uint32_t*)(zeros_ptr + k_0_0 * 32 / G * (OC / 8));
//     uint4 B_loaded_zero = dequantize_s4_to_fp16x2(zeros_loaded);
//     uint4 B_loaded_scale = *(uint4*)(scaling_factors_ptr + k_0_0 * 32 / G * (OC));
//     /*
//     if (blockIdx_z == 0 && blockIdx_y == 0 && k_0_0 == 0 && threadIdx.x == 0 && threadIdx.y == 0){
//       printf("%x %x %x %x %x %x %x %x\n", B_loaded_scale.x, B_loaded_scale.y, B_loaded_scale.z, B_loaded_scale.w, B_loaded_zero.x, B_loaded_zero.y, B_loaded_zero.z, B_loaded_zero.w);
//     }
//     */
//     // uint4 B_loaded_scale = make_uint4(0, 0, 0, 0);
//     int* B_ptr_local = B_ptr + k_0_0 * 32 * (OC / 8);

//     for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 8; ++ax0_ax1_fused_0) {
// // TODO: Shang: double check how to get 8.

//       // B: 32 x 136 (128+8) float16
//       // each warp: 32 x 4
//       // each thr: read 32 bit -> convert to 8xFP16 (a UINT4) -> scale and minus zero -> WB UINT4
//       // *(uint4*)(B_shared + ((((ax0_ax1_fused_0 * 544) + (((int)threadIdx.y) * 272)) + ((((int)threadIdx.x) >> 4) * 136)) + ((((int)threadIdx.x) & 15) * 8))) = *(uint4*)(B + ((((((k_0_0 * 163840) + (ax0_ax1_fused_0 * 20480)) + (((int)threadIdx.y) * 10240)) + ((((int)threadIdx.x) >> 4) * 5120)) + (((int)blockIdx_y) * 128)) + ((((int)threadIdx.x) & 15) * 8)));
//       // row stride in shared memory: (NWARPS * 32 * 8 / cta_N) 
//       uint32_t B_loaded = *(uint32_t*)(B_ptr_local + ax0_ax1_fused_0 * row_stride * (OC / 8));
//       uint4 B_loaded_fp16 = dequantize_s4_to_fp16x2(B_loaded);
//       //uint4 B_loaded_zero = *(uint4*)(zeros_shared + (threadIdx.x % (cta_N / 8)) * 8);

//       // uint4 B_loaded_scale = *(uint4*)(scaling_factors_shared + (threadIdx.x % (cta_N / 8)) * 8);
//       // - zero and * scale
//       // TODO (Haotian): can save 4 assembly instructions if sormulate as deq = q * scale - zero * scale.
//       asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.x) : "r"(B_loaded_fp16.x), "r"(B_loaded_zero.x));
//       asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.x) : "r"(B_loaded_fp16.x), "r"(B_loaded_scale.x), "r"(ZERO));
//       asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.y) : "r"(B_loaded_fp16.y), "r"(B_loaded_zero.y));
//       asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.y) : "r"(B_loaded_fp16.y), "r"(B_loaded_scale.y), "r"(ZERO));
//       asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.z) : "r"(B_loaded_fp16.z), "r"(B_loaded_zero.z));
//       asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.z) : "r"(B_loaded_fp16.z), "r"(B_loaded_scale.z), "r"(ZERO));
//       asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.w) : "r"(B_loaded_fp16.w), "r"(B_loaded_zero.w));
//       asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.w) : "r"(B_loaded_fp16.w), "r"(B_loaded_scale.w), "r"(ZERO));
//       /*
//       if (ax0_ax1_fused_0 == 0 && blockIdx_z == 0 && blockIdx_y == 0 && k_0_0 == 0 && threadIdx.x == 17 && threadIdx.y == 0){
//         printf("[x] %X %X %X %X\n", B_loaded_fp16.x, B_loaded_fp16.y, B_loaded_fp16.z, B_loaded_fp16.w);
//       }
//       */

//       // write back
//       *(uint4*)(B_shared_ptr + ax0_ax1_fused_0 * row_stride * (128 + 8)) = B_loaded_fp16;
//     }
//     __syncthreads();

//     for (int k_0_1 = 0; k_0_1 < 2; ++k_0_1) {
//       {
//         unsigned int addr;
//         __asm__ __volatile__(
//           "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
//           : "=r"(addr)
//           : "l"((void *)((&(A_shared[(k_0_1 * 16)])) + (((((int)threadIdx.x) & 15) * 40) + ((((int)threadIdx.x) >> 4) * 8))))
//         );


//         __asm__ __volatile__(
//           "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
//           "{%0, %1, %2, %3}, [%4];\n"
//           : "=r"(((unsigned *)(A_shared_warp + 0))[0]), "=r"(((unsigned *)(A_shared_warp + 0))[1]), "=r"(((unsigned *)(A_shared_warp + 0))[2]), "=r"(((unsigned *)(A_shared_warp + 0))[3])
//           : "r"(addr)
//         );
//       }

//       for (int ax1_0 = 0; ax1_0 < 4; ++ax1_0) {
//         {
//           unsigned int addr;
//           __asm__ __volatile__(
//             "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
//             : "=r"(addr)
//             : "l"((void *)((&(B_shared[(((k_0_1 * 2176) + (((int)threadIdx.y) * 64)) + (ax1_0 * 16))])) + (((((int)threadIdx.x) & 15) * 136) + ((((int)threadIdx.x) >> 4) * 8))))
//           );
//           __asm__ __volatile__(
//             "ldmatrix.sync.aligned.m8n8.x4.trans.shared.b16"
//             "{%0, %1, %2, %3}, [%4];\n"
//             : "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[0]), "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[1]), "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[2]), "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[3])
//             : "r"(addr)
//           );
//         }
//       }
//       for (int j_0_4 = 0; j_0_4 < 4; ++j_0_4) {
//         {
//           __asm__ __volatile__(
//             "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
//             "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
//             :  "=f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[3])
//             : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + (j_0_4 * 8)))[0]), "r"(((unsigned *)(B_shared_warp + (j_0_4 * 8)))[1]), "f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "f"(((float *)(C_warp + (j_0_4 * 8)))[3]));
//         }

//         {
//           __asm__ __volatile__(
//             "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
//             "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
//             :  "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3])
//             : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + ((j_0_4 * 8) + 4)))[0]), "r"(((unsigned *)(B_shared_warp + ((j_0_4 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3]));
//         }
//       }
//     }
//   }

// // TODO: Shang: Hoist loop invariance.
//   for (int ax1_0_1 = 0; ax1_0_1 < 4; ++ax1_0_1) {
//     for (int local_id = 0; local_id < 8; ++local_id) {
//       int row_offset = (((int)blockIdx_y) / j_factors1) * 16 + ((int)threadIdx.x) / 4 + (local_id % 4) / 2 * 8;
//       if (row_offset < M)
//       {
//         *(C_ptr + ax1_0_1 * 16 + row_offset * OC + (local_id / 4) * 8 + local_id % 2) = __float2half(C_warp[(ax1_0_1 * 8) + local_id]);
//       }
//     }
//   }
// }


// __global__ void __launch_bounds__(64) gemm_forward_4bit_cuda_m16n64k32(int G, int split_k_iters, half* __restrict__ A, int* __restrict__ B, half* __restrict__ scaling_factors, int* __restrict__ zeros, int M, int IC, int OC, half* __restrict__ C) 
// {
//   static constexpr uint32_t ZERO = 0x0;
//   float C_warp[32];
//   __shared__ half A_shared[16 * (32 + 8)];
//   __shared__ half B_shared[32 * (64 + 8)];
  
//   __shared__ half scaling_factors_shared[64];
//   __shared__ half zeros_shared[64];

//   int j_factors1 = ((OC + 64 - 1) / 64);

//   int blockIdx_x = 0;
//   int blockIdx_y = blockIdx.x % ((M + 16 - 1) / 16 * j_factors1);
//   int blockIdx_z = blockIdx.x / ((M + 16 - 1) / 16 * j_factors1);

//   half A_shared_warp[8];
//   half B_shared_warp[16];
//   for (int j_0_4_init = 0; j_0_4_init < 2; ++j_0_4_init) {
//     for (int i = 0; i < 8; ++i) {
//       C_warp[(j_0_4_init * 8) + i] = 0.0;
//     }
//   }

//   static constexpr int row_stride_warp = 32 * 8 / 32;
//   static constexpr int row_stride = 2 * 32 * 8 / 64;
//   bool ld_zero_flag = (threadIdx.y * 32 + threadIdx.x) * 8 < 64;
//   // TODO: Haotian: blockIdx_y / j_factors1 in A loading to support bsz > 16
//   bool ld_A_flag = (blockIdx_y / j_factors1 * 16 + threadIdx.y * row_stride_warp + threadIdx.x * 8 / 32) < M;     // threadIdx.y is warp_id
//   // bool wb_C_flag = (threadIdx.x / 4) < M;

//   half* A_ptr = A 
//                 + (((int)blockIdx_y) / j_factors1 * 16 + (((int)threadIdx.y) * row_stride_warp) + ((int)threadIdx.x) / (32 / 8)) * IC
//                 + (((int)threadIdx.x) % (32 / 8)) * 8;
  
//   int* B_ptr = B
//             + ((int)threadIdx.y) * (OC / 8) * 4
//             + (((int)threadIdx.x) / (64 / 8)) * (OC / 8)
//             + (((int)blockIdx_y) % j_factors1) * (64 / 8)
//             + (((int)threadIdx.x) % (64 / 8)) * 1;
// // Why * 1 in the above line?
                        
//   half* A_shared_ptr = A_shared 
//                     + ((int)threadIdx.y) * row_stride_warp * (32 + 8) 
//                     + (((int)threadIdx.x) / (32 / 8)) * (32 + 8)
//                     + (((int)threadIdx.x) % (32 / 8) ) * 8;

//   half* B_shared_ptr = B_shared
//                     + ((int)threadIdx.y) * (row_stride / 2) * (64 + 8)
//                     + (((int)threadIdx.x) / (64 / 8)) * (64 + 8)
//                     + (((int)threadIdx.x) % (64 / 8)) * 8;
  
//   int* zeros_ptr = zeros
//                 + (((int)blockIdx_y) % j_factors1) * (64 / 8)
//                 + ((int)threadIdx.x) % (64 / 8);
  
//   half* scaling_factors_ptr = scaling_factors
//                             + (((int)blockIdx_y) % j_factors1) * (64) 
//                             + (((int)threadIdx.x) % (64 / 8)) * 8;

//   half* C_ptr = C 
//               + blockIdx_z * M * OC        // blockIdz.x -> split_k dim
//               + (((int)blockIdx_y) % j_factors1) * 64
//               + ((int)threadIdx.y) * 32
//               + (((int)threadIdx.x) % 4) * 2;

//   // preload s.f. and zeros
//   int k_bound = (IC / 32 + split_k_iters - 1) / split_k_iters;
//   if ((k_bound - 1) * split_k_iters * 32 + blockIdx_z * 32 >= IC) k_bound -= 1;
//   for (int _k_0_0 = 0; _k_0_0 < k_bound; ++_k_0_0) {
//     int k_0_0 = _k_0_0 * split_k_iters + blockIdx_z;
//     __syncthreads();
//     // TODO: Haotian: blockIdx_y / j_factors1 in A loading to support bsz > 16
//     if (ld_A_flag)
//     {
//       *(uint4*)(A_shared_ptr) = *(uint4*)(A_ptr + (k_0_0 * 32));
//     }
//     else
//     {
//       *(uint4*)(A_shared_ptr) = make_uint4(0, 0, 0, 0);
//     }

//     // for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 2; ++ax0_ax1_fused_0) {
//     uint32_t zeros_loaded = *(uint32_t*)(zeros_ptr + k_0_0 * 32 / G * (OC / 8));
//     uint4 B_loaded_zero = dequantize_s4_to_fp16x2(zeros_loaded);
//     uint4 B_loaded_scale = *(uint4*)(scaling_factors_ptr + k_0_0 * 32 / G * (OC));
//     /*
//     if (blockIdx_z == 0 && blockIdx_y == 0 && k_0_0 == 0 && threadIdx.x == 0 && threadIdx.y == 0){
//       printf("%x %x %x %x %x %x %x %x\n", B_loaded_scale.x, B_loaded_scale.y, B_loaded_scale.z, B_loaded_scale.w, B_loaded_zero.x, B_loaded_zero.y, B_loaded_zero.z, B_loaded_zero.w);
//     }
//     */
//     // uint4 B_loaded_scale = make_uint4(0, 0, 0, 0);
//     int* B_ptr_local = B_ptr + k_0_0 * 32 * (OC / 8);

//     for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 4; ++ax0_ax1_fused_0) {

//       // B: 32 x 136 (128+8) float16
//       // each warp: 32 x 4
//       // each thr: read 32 bit -> convert to 8xFP16 (a UINT4) -> scale and minus zero -> WB UINT4
//       // *(uint4*)(B_shared + ((((ax0_ax1_fused_0 * 544) + (((int)threadIdx.y) * 272)) + ((((int)threadIdx.x) >> 4) * 136)) + ((((int)threadIdx.x) & 15) * 8))) = *(uint4*)(B + ((((((k_0_0 * 163840) + (ax0_ax1_fused_0 * 20480)) + (((int)threadIdx.y) * 10240)) + ((((int)threadIdx.x) >> 4) * 5120)) + (((int)blockIdx_y) * 128)) + ((((int)threadIdx.x) & 15) * 8)));
//       // row stride in shared memory: (NWARPS * 32 * 8 / cta_N) 
//       uint32_t B_loaded = *(uint32_t*)(B_ptr_local + ax0_ax1_fused_0 * row_stride * (OC / 8));
//       uint4 B_loaded_fp16 = dequantize_s4_to_fp16x2(B_loaded);
//       //uint4 B_loaded_zero = *(uint4*)(zeros_shared + (threadIdx.x % (cta_N / 8)) * 8);

//       // uint4 B_loaded_scale = *(uint4*)(scaling_factors_shared + (threadIdx.x % (cta_N / 8)) * 8);
//       // - zero and * scale
//       // TODO (Haotian): can save 4 assembly instructions if sormulate as deq = q * scale - zero * scale.
//       asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.x) : "r"(B_loaded_fp16.x), "r"(B_loaded_zero.x));
//       asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.x) : "r"(B_loaded_fp16.x), "r"(B_loaded_scale.x), "r"(ZERO));
//       asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.y) : "r"(B_loaded_fp16.y), "r"(B_loaded_zero.y));
//       asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.y) : "r"(B_loaded_fp16.y), "r"(B_loaded_scale.y), "r"(ZERO));
//       asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.z) : "r"(B_loaded_fp16.z), "r"(B_loaded_zero.z));
//       asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.z) : "r"(B_loaded_fp16.z), "r"(B_loaded_scale.z), "r"(ZERO));
//       asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(B_loaded_fp16.w) : "r"(B_loaded_fp16.w), "r"(B_loaded_zero.w));
//       asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(B_loaded_fp16.w) : "r"(B_loaded_fp16.w), "r"(B_loaded_scale.w), "r"(ZERO));
//       /*
//       if (ax0_ax1_fused_0 == 0 && blockIdx_z == 0 && blockIdx_y == 0 && k_0_0 == 0 && threadIdx.x == 17 && threadIdx.y == 0){
//         printf("[x] %X %X %X %X\n", B_loaded_fp16.x, B_loaded_fp16.y, B_loaded_fp16.z, B_loaded_fp16.w);
//       }
//       */

//       // write back
//       *(uint4*)(B_shared_ptr + ax0_ax1_fused_0 * row_stride * (64 + 8)) = B_loaded_fp16;
//     }
//     __syncthreads();

//     for (int k_0_1 = 0; k_0_1 < 2; ++k_0_1) 
//     {
//       {
//         unsigned int addr;
//         __asm__ __volatile__(
//           "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
//           : "=r"(addr)
//           : "l"((void *)((&(A_shared[(k_0_1 * 16)])) + (((((int)threadIdx.x) & 15) * 40) + ((((int)threadIdx.x) >> 4) * 8))))
//         );
//         __asm__ __volatile__(
//           "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
//           "{%0, %1, %2, %3}, [%4];\n"
//           : "=r"(((unsigned *)(A_shared_warp + 0))[0]), "=r"(((unsigned *)(A_shared_warp + 0))[1]), "=r"(((unsigned *)(A_shared_warp + 0))[2]), "=r"(((unsigned *)(A_shared_warp + 0))[3])
//           : "r"(addr)
//         );
//       }
        

//       for (int ax1_0 = 0; ax1_0 < 2; ++ax1_0) 
//       {
//         {
//           unsigned int addr;
//           __asm__ __volatile__(
//             "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
//             : "=r"(addr)
//             : "l"((void *)((&(B_shared[(((k_0_1 * 1152) + (((int)threadIdx.y) * 32)) + (ax1_0 * 16))])) + (((((int)threadIdx.x) & 15) * 72) + ((((int)threadIdx.x) >> 4) * 8))))
//           );
//           __asm__ __volatile__(
//             "ldmatrix.sync.aligned.m8n8.x4.trans.shared.b16"
//             "{%0, %1, %2, %3}, [%4];\n"
//             : "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[0]), "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[1]), "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[2]), "=r"(((unsigned *)(B_shared_warp + (ax1_0 * 8)))[3])
//             : "r"(addr)
//           );
//         }
//       }
      
//       for (int j_0_4 = 0; j_0_4 < 2; ++j_0_4) 
//       {

//         {
//           __asm__ __volatile__(
//             "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
//             "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
//             :  "=f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "=f"(((float *)(C_warp + (j_0_4 * 8)))[3])
//             : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + (j_0_4 * 8)))[0]), "r"(((unsigned *)(B_shared_warp + (j_0_4 * 8)))[1]), "f"(((float *)(C_warp + (j_0_4 * 8)))[0]), "f"(((float *)(C_warp + (j_0_4 * 8)))[1]), "f"(((float *)(C_warp + (j_0_4 * 8)))[2]), "f"(((float *)(C_warp + (j_0_4 * 8)))[3]));
//         }

//         {
//           __asm__ __volatile__(
//             "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
//             "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
//             :  "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "=f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3])
//             : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_shared_warp + ((j_0_4 * 8) + 4)))[0]), "r"(((unsigned *)(B_shared_warp + ((j_0_4 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[0]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[2]), "f"(((float *)(C_warp + ((j_0_4 * 8) + 4)))[3]));
//         }
//       }
//     }
//   }

// // TODO: Shang: Hoist loop invariance.
//   for (int ax1_0_1 = 0; ax1_0_1 < 2; ++ax1_0_1) {
//     for (int local_id = 0; local_id < 8; ++local_id) {
//       int row_offset = (((int)blockIdx_y) / j_factors1) * 16 + ((int)threadIdx.x) / 4 + (local_id % 4) / 2 * 8;
//       if (row_offset < M)
//       {
//         *(C_ptr + ax1_0_1 * 16 + row_offset * OC + (local_id / 4) * 8 + local_id % 2) = __float2half(C_warp[(ax1_0_1 * 8) + local_id]);
//       }
//     }
//   }
// }

// // in_feats: M, IC [float16]
// // kernel: IC, OC // 8 [int32] -> cast to IC, OC [uint4b]
// // scaling_factors: IC // G, OC [float16]
// // zeros: IC // G, OC // 8 [int32] -> cast to IC // G, OC [uint4b]
// // assume that batch_size < 16 for now

// // torch::Tensor gemm_forward_cuda_origin(
// //   torch::Tensor _in_feats,
// //   torch::Tensor _kernel,
// //   torch::Tensor _scaling_factors,
// //   torch::Tensor _zeros,
// //   int split_k_iters)
// // {
// //   int num_in_feats = _in_feats.size(0);
// //   int num_in_channels = _in_feats.size(1);
// //   const at::cuda::OptionalCUDAGuard device_guard(device_of(_in_feats));

// //   auto options = torch::TensorOptions().dtype(_in_feats.dtype()).device(_in_feats.device());
// //   at::Tensor _out_feats = torch::empty({split_k_iters, num_in_feats, _kernel.size(1) * 8}, options);
// //   int num_out_feats = _out_feats.size(-2);
// //   int num_out_channels = _out_feats.size(-1);

// //   auto in_feats = reinterpret_cast<half*>(_in_feats.data_ptr<at::Half>());
// //   auto kernel = reinterpret_cast<int*>(_kernel.data_ptr<int>());
// //   auto out_feats = reinterpret_cast<half*>(_out_feats.data_ptr<at::Half>());
// //   auto scaling_factors = reinterpret_cast<half*>(_scaling_factors.data_ptr<at::Half>());
// //   auto zeros = reinterpret_cast<int*>(_zeros.data_ptr<int>());
// //   int group_size = num_in_channels / _scaling_factors.size(0);

// //   // std::cout << "_kernel: " << _kernel << std::endl;

// //   if (num_out_channels % 64 != 0)
// //     throw std::invalid_argument("OC is not multiple of cta_N = 64");
// //   if (num_out_channels % 8 != 0)
// //     throw std::invalid_argument("OC is not multiple of pack_num = 8");
// //   if (group_size % 32 != 0)
// //     throw std::invalid_argument("Group size should be a multiple of 32");
// //   if (num_out_channels % group_size != 0)
// //     throw std::invalid_argument("OC is not multiple of Group size");

// //   if (num_out_channels % 128 == 0)
// //   {
// //     int j_factors1 = num_out_channels / 128 / 1;
// //     dim3 num_blocks((num_out_feats + 16 - 1) / 16 * j_factors1 * split_k_iters);
// //     // threadIdx.x: 32
// //     // threadIdx.y: i_factors[2] * j_factors[2]
// //     dim3 threads_per_block(32, 2);
    
// //     // std::cout << "AAA group_size: " << group_size << std::endl;

// //     gemm_forward_4bit_cuda_m16n128k32<<<num_blocks, threads_per_block>>>(
// //         group_size, split_k_iters, in_feats, kernel, scaling_factors, zeros, num_in_feats, num_in_channels, num_out_channels, out_feats);
// //   }
// //   else if (num_out_channels % 64 == 0)
// //   {
// //     int j_factors1 = num_out_channels / 64 / 1;
// //     dim3 num_blocks(1 * (num_out_feats + 16 - 1) / 16 * j_factors1 * split_k_iters);

// //     // threadIdx.x: 32
// //     // threadIdx.y: i_factors[2] * j_factors[2]
// //     dim3 threads_per_block(32, 2);

// //     // std::cout << "BBB group_size: " << group_size << std::endl;

// //     gemm_forward_4bit_cuda_m16n64k32<<<num_blocks, threads_per_block>>>(
// //         group_size, split_k_iters, in_feats, kernel, scaling_factors, zeros, num_in_feats, num_in_channels, num_out_channels, out_feats);
// //   }

// //   return _out_feats.to(torch::kFloat32).sum(0);
// // }


// __global__ void my_add(half* data_0, half* data_1, half* data_2, int num_elements)
// {
//   int idx = blockIdx.x * blockDim.x + threadIdx.x;
//   if (idx < num_elements)
//   {
//     data_2[idx] = data_0[idx] + data_1[idx];
//   }
// }

// // in_feats: M, IC [float16]
// // kernel: IC, OC // 8 [int32] -> cast to IC, OC [uint4b]
// // scaling_factors: IC // G, OC [float16]
// // zeros: IC // G, OC // 8 [int32] -> cast to IC // G, OC [uint4b]
// // assume that batch_size < 16 for now

// void gemm_forward_cuda(const struct matmul_params *params, int split_k_iters)
// {
// 	const struct matrix *A = &params->A, *B = &params->B, *C = &params->C;

//   int num_in_feats = A->row;
//   int num_in_channels = A->column;
//   // const at::cuda::OptionalCUDAGuard device_guard(device_of(_in_feats));

//   // auto options = torch::TensorOptions().dtype(_in_feats.dtype()).device(_in_feats.device());
  
//   // at::Tensor _out_feats = torch::empty({split_k_iters, num_in_feats, _kernel.size(1) * 8}, options);
//   int num_out_feats = C->row;
//   int num_out_channels = C->column;

//   // auto in_feats = reinterpret_cast<half*>(_in_feats.data_ptr<at::Half>());
//   // auto kernel = reinterpret_cast<int*>(_kernel.data_ptr<int>());
//   // auto out_feats = reinterpret_cast<half*>(_out_feats.data_ptr<at::Half>());
//   // auto scaling_factors = reinterpret_cast<half*>(_scaling_factors.data_ptr<at::Half>());
//   // auto zeros = reinterpret_cast<int*>(_zeros.data_ptr<int>());
//   // int group_size = num_in_channels / _scaling_factors.size(0);

//   // auto in_feats = (half*)A->data_ptr;
//   // auto kernel = B->int32_data_ptr;
//   // auto out_feats = (half*)C->data_ptr;
//   // auto scaling_factors = (half*)params->scales;
//   // auto zeros = params->int32_zero_point;
//   // int group_size = QK;

//   half* in_feats = A->data_fp16_ptr;
//   int* kernel = B->int32_data_ptr;
//   half* out_feats = C->data_fp16_ptr;
//   half* scaling_factors = params->scales_fp16;
//   int* zeros = params->int32_zero_point;
//   int group_size = QK;

//   // half* in_feats;
//   // // int* kernel;
//   // half* out_feats;
//   // // half* scaling_factors;
//   // // int* zeros;

//   // // Allocate device memory
//   int A_size = A->row * A->column;
//   // printf("A_size: %d\n", A_size);
//   // printf("A->row: %d\n", A->row);
//   // printf("A->column: %d\n", A->column);
//   int C_size = C->row * C->column;
//   // printf("C_size: %d\n", C_size);
//   // printf("C->row: %d\n", C->row);
//   // printf("C->column: %d\n", C->column);
//   int sf_size = B->row / group_size * B->column * 8;
//   // printf("sf_size: %d\n", sf_size);
//   // printf("B->row: %d\n", B->row);
//   // printf("B->column: %d\n", B->column);

//   // hipError_t err;
//   // printf("1111111\n");
//   // err = hipMallocManaged(&in_feats, A_size * sizeof(half));
//   // printf("2222222\n");
//   // if (err != hipSuccess) {
//   //   printf("Error allocating memory for in_feats: %s\n", hipGetErrorString(err));
//   // }
//   // // allocate_aligned_memory_gpu(in_feats, A_size * sizeof(half));
//   // err = hipGetLastError();
//   // if (err != hipSuccess) {
//   //   printf("Error allocating memory for in_feats: %s\n", hipGetErrorString(err));
//   // }

//   // err = hipMallocManaged(&kernel, B->row * B->column * sizeof(int));
//   // if (err != hipSuccess) {
//   //   printf("Error allocating memory for kernel: %s\n", hipGetErrorString(err));
//   // }
//   // // allocate_aligned_memory_gpu(kernel, B->row * B->column * sizeof(int));
//   // err = hipGetLastError();
//   // if (err != hipSuccess) {
//   //   printf("Error allocating memory for kernel: %s\n", hipGetErrorString(err));
//   // }

//   // err = hipMallocManaged(&out_feats, split_k_iters * C_size * sizeof(half));
//   // if (err != hipSuccess) {
//   //   printf("Error allocating memory for out_feats: %s\n", hipGetErrorString(err));
//   // }
//   // // allocate_aligned_memory_gpu(out_feats, C_size * sizeof(half));
//   // err = hipGetLastError();
//   // if (err != hipSuccess) {
//   //     printf("Error allocating memory for out_feats: %s\n", hipGetErrorString(err));
//   //     return;
//   // }

//   // err = hipMallocManaged(&scaling_factors, sf_size * sizeof(half));
//   // if (err != hipSuccess) {
//   //   printf("Error allocating memory for scaling_factors: %s\n", hipGetErrorString(err));
//   // }
//   // // allocate_aligned_memory_gpu(scaling_factors, sf_size * sizeof(half));
//   // err = hipGetLastError();
//   // if (err != hipSuccess) {
//   //     printf("Error allocating memory for scaling_factors: %s\n", hipGetErrorString(err));
//   //     return;
//   // }

//   // err = hipMallocManaged(&zeros, B->row * B->column / group_size * sizeof(int));
//   // if (err != hipSuccess) {
//   //   printf("Error allocating memory for zeros: %s\n", hipGetErrorString(err));
//   // }
//   // // allocate_aligned_memory_gpu(zeros, B->row * B->column / group_size * sizeof(int));
//   // err = hipGetLastError();
//   // if (err != hipSuccess) {
//   //     printf("Error allocating memory for zeros: %s\n", hipGetErrorString(err));
//   //     return;
//   // }
    
//   // printf("3333333\n");

//   // // Launch the kernel
//   // int blockSize = 256;
//   // int numBlocks = (A_size + blockSize - 1) / blockSize;
//   // PROFILE_START("float2half::in_feats");
//   // float2half<<<numBlocks, blockSize>>>(A->data_ptr, in_feats, A_size);
//   // // for (int i = 0; i < A_size; i++) {
//   // //   in_feats[i] = __float2half(A->data_ptr[i]);
//   // // }
//   // // err = hipGetLastError();
//   // // if (err != hipSuccess) {
//   // //   printf("Error launching float2half kernel for in_feats: %s\n", hipGetErrorString(err));
//   // // }

//   // // for (int i = 0; i < B->row * B->column; i++) {
//   // //   kernel[i] = B->int32_data_ptr[i];
//   // // }

//   // // for (int i = 0; i < B->row * B->column / group_size; i++) {
//   // //   zeros[i] = params->int32_zero_point[i];
//   // // }

//   // // // printf("4444444\n");
//   // // // numBlocks = (C_size + blockSize - 1) / blockSize;
//   // // // float2half<<<numBlocks, blockSize>>>(C->data_ptr, out_feats, C_size);
//   // // for (int i = 0; i < C_size; i++) {
//   // //   out_feats[i] = __float2half(C->data_ptr[i]);
//   // // }

//   // // numBlocks = (sf_size + blockSize - 1) / blockSize;
//   // // float2half<<<numBlocks, blockSize>>>(params->scales, scaling_factors, sf_size);
//   // // for (int i = 0; i < sf_size; i++) {
//   // //   scaling_factors[i] = __float2half(params->scales[i]);
//   // // }

//   // // err = hipGetLastError();
//   // // if (err != hipSuccess) {
//   // //   printf("Error launching float2half kernel for scaling_factors: %s\n", hipGetErrorString(err));
//   // // }

//   // hipDeviceSynchronize();
//   // PROFILE_END("float2half::in_feats");

//   // err = hipGetLastError();
//   // if (err != hipSuccess) {
//   //   printf("Error launching hipDeviceSynchronize kernel: %s\n", hipGetErrorString(err));
//   // }

//   // printf("A->data_ptr: ");
//   // for (int i = 0; i < 32; i++) {
//   //   printf("%f ", A->data_ptr[i]);
//   // }
//   // printf("\n");

//   // printf("in_feats: ");
//   // for (int i = 0; i < 32; i++) {
//   //   printf("%f ", __half2float(in_feats[i]));
//   // }
//   // printf("\n");

//   // printf("5555555\n");
//   // // Synchronize CPU and GPU before accessing managed memory
//   // hipDeviceSynchronize();

//   // std::cout << "_kernel: " << _kernel << std::endl;

//   if (num_out_channels % 64 != 0)
//     throw std::invalid_argument("OC is not multiple of cta_N = 64");
//   if (num_out_channels % 8 != 0)
//     throw std::invalid_argument("OC is not multiple of pack_num = 8");
//   if (group_size % 32 != 0)
//     throw std::invalid_argument("Group size should be a multiple of 32");
//   if (num_out_channels % group_size != 0)
//     throw std::invalid_argument("OC is not multiple of Group size");

//   PROFILE_START("gemm_forward_4bit_cuda_m16n128k32");
//   // printf("6666666\n");
//   if (num_out_channels % 128 == 0)
//   {
//     int j_factors1 = num_out_channels / 128 / 1;
//     dim3 num_blocks((num_out_feats + 16 - 1) / 16 * j_factors1 * split_k_iters);
//     // threadIdx.x: 32
//     // threadIdx.y: i_factors[2] * j_factors[2]
//     dim3 threads_per_block(32, 2);
    
//     // std::cout << "AAA group_size: " << group_size << std::endl;

//     // printf("7777777\n");
//     gemm_forward_4bit_cuda_m16n128k32<<<num_blocks, threads_per_block>>>(
//         group_size, split_k_iters, in_feats, kernel, scaling_factors, zeros, num_in_feats, num_in_channels, num_out_channels, out_feats);
//     // err = hipGetLastError();
//     // if (err != hipSuccess) {
//     //     printf("Error during kernel launch 1: %s\n", hipGetErrorString(err));
//     // }
//     // printf("8888888\n");
//   }
//   else if (num_out_channels % 64 == 0)
//   {
//     int j_factors1 = num_out_channels / 64 / 1;
//     dim3 num_blocks(1 * (num_out_feats + 16 - 1) / 16 * j_factors1 * split_k_iters);

//     // threadIdx.x: 32
//     // threadIdx.y: i_factors[2] * j_factors[2]
//     dim3 threads_per_block(32, 2);

//     // std::cout << "BBB group_size: " << group_size << std::endl;

//     // printf("9999999\n");
//     gemm_forward_4bit_cuda_m16n64k32<<<num_blocks, threads_per_block>>>(
//         group_size, split_k_iters, in_feats, kernel, scaling_factors, zeros, num_in_feats, num_in_channels, num_out_channels, out_feats);
//     // err = hipGetLastError();
//     // if (err != hipSuccess) {
//     //     printf("Error during kernel launch 2: %s\n", hipGetErrorString(err));
//     // }
//     // printf("1010101010\n");
//   }

//   // return _out_feats.to(torch::kFloat32).sum(0);

//   // Synchronize CPU and GPU before accessing managed memory
//   hipDeviceSynchronize();
//   PROFILE_END("gemm_forward_4bit_cuda_m16n128k32");

//   // err = hipGetLastError();
//   // if (err != hipSuccess) {
//   //   printf("Error launching hipDeviceSynchronize 2222 kernel: %s\n", hipGetErrorString(err));
//   // }

//   // printf("aaaaaaaaaaa\n");
//   // numBlocks = (A_size + blockSize - 1) / blockSize;
//   // half2float<<<numBlocks, blockSize>>>(in_feats, A->data_ptr, A_size);

//   // printf("bbbbbbbbbbb\n");
//   // numBlocks = (C_size + blockSize - 1) / blockSize;
//   // half2float<<<numBlocks, blockSize>>>(out_feats, C->data_ptr, C_size);
//   // for (int i = 0; i < C_size; i++) {
//   //   C->data_ptr[i] = __half2float(out_feats[i]) + __half2float(out_feats[i + C_size]) + __half2float(out_feats[i + 2 * C_size]) + __half2float(out_feats[i + 3 * C_size]) +
//   //     __half2float(out_feats[i + 4 * C_size]) + __half2float(out_feats[i + 5 * C_size]) + __half2float(out_feats[i + 6 * C_size]) + __half2float(out_feats[i + 7 * C_size]);
//   // }

//   // for (int i = 0; i < C_size; i++) {
//   //   C->data_ptr[i] = 0;
//   //   for (int j = 0; j < split_k_iters; j++) {
//   //     C->data_ptr[i] += __half2float(out_feats[i + j * C_size]);
//   //   }
//   // }

//   // PROFILE_START("half2float_merge_k_iters");
//   // half2float_merge_k_iters<<<numBlocks, blockSize>>>(out_feats, C->data_ptr, C_size, split_k_iters);

//   // // numBlocks = (sf_size + blockSize - 1) / blockSize;
//   // // half2float<<<numBlocks, blockSize>>>(scaling_factors, params->scales, sf_size);
//   // // printf("ccccccccccc\n");
//   // // Synchronize CPU and GPU before accessing managed memory
//   // hipDeviceSynchronize();
//   // PROFILE_END("half2float_merge_k_iters");

//   // // Free memory
//   // // printf("ddddddddddd\n");
//   // PROFILE_START("hipFree");
//   // hipFree(in_feats);
//   // // cukdaFree(kernel);
//   // hipFree(out_feats);
//   // // hipFree(scaling_factors);
//   // // hipFree(zeros);
//   // // printf("fffffffffff\n");
//   // PROFILE_END("hipFree");


//   // float weight;
//   // for (int i = 0; i < C->row; i++) {
//   //   for (int j = 0; j < C->column; j++) {
//   //     float acc = 0;

//   //     for (int k = 0; k < B->row; k++) {
//   //       float s = __half2float(params->scales_fp16[(k / 32) * C->column + j]);
//   //       float input = A->data_ptr[i * A->column + k];

//   //       if (j % 8 == 0)
//   //         weight = ((B->int32_data_ptr[k * B->column + (j / 8)] & 0x0000000F) - 8.0) * s;
//   //       else if (j % 8 == 1)
//   //         weight = (((B->int32_data_ptr[k * B->column + (j / 8)] & 0x000000F0) >> 4) - 8.0) * s;
//   //       else if (j % 8 == 2)
//   //         weight = (((B->int32_data_ptr[k * B->column + (j / 8)] & 0x00000F00) >> 8) - 8.0) * s;
//   //       else if (j % 8 == 3)
//   //         weight = (((B->int32_data_ptr[k * B->column + (j / 8)] & 0x0000F000) >> 12) - 8.0) * s;
//   //       else if (j % 8 == 4)
//   //         weight = (((B->int32_data_ptr[k * B->column + (j / 8)] & 0x000F0000) >> 16) - 8.0) * s;
//   //       else if (j % 8 == 5)
//   //         weight = (((B->int32_data_ptr[k * B->column + (j / 8)] & 0x00F00000) >> 20) - 8.0) * s;
//   //       else if (j % 8 == 6)
//   //         weight = (((B->int32_data_ptr[k * B->column + (j / 8)] & 0x0F000000) >> 24) - 8.0) * s;
//   //       else if (j % 8 == 7)
//   //         weight = (((B->int32_data_ptr[k * B->column + (j / 8)] & 0xF0000000) >> 28) - 8.0) * s;

//   //       acc += input * weight;
//   //     }
      
//   //     C->data_ptr[i * C->column + j] = acc;
//   //   }
//   // }

//   // my_add<<<1, 1>>>(&data_0, &data_1, &data_2, 1);
//   // printf("data_2: %f\n", __half2float(data_2));
//   // printf("data_2: %f\n", __half2float(data_2));
// }


// namespace matmul{
//   void MatmulOperator::mat_mul_accelerator_int4_fast(const struct matmul_params *params) {
// 		// const struct matrix *A = &params->A, *B = &params->B, *C = &params->C;

//     // // std::cout << "mat_mul_accelerator_int4_fast -- A->row: " << A->row << " A->column: " << A->column 
//     //           // << "; B->row: " << B->row << " B->column: " << B->column 
//     //           // << "; C->row: " << C->row << " C->column: " << C->column << std::endl;
    
//     // // torch::Tensor out_feats = gemm_forward_cuda(
//     // //     torch::from_blob(A->data_ptr, {A->row, A->column}, torch::kHalf),
//     // //     torch::from_blob(B->data_ptr, {B->row, B->column}, torch::kInt),
//     // //     torch::from_blob(params->scales, {B->row / 128, B->column * 8}, torch::kHalf),
//     // //     torch::from_blob(params->int32_zero_point, {B->row / 128, B->column}, torch::kInt),
//     // //     8);
    
//     // auto option_fp = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, 1);
//     // auto option_int = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA, 1);
//     // torch::Tensor _in_feats = torch::from_blob(A->data_ptr, {A->row, A->column}, option_fp);
//     // torch::Tensor _kernel = torch::from_blob(B->data_ptr, {B->row, B->column}, option_int);
//     // torch::Tensor _scaling_factors = torch::from_blob(params->scales, {B->row / 128, B->column * 8}, option_fp);
//     // torch::Tensor _zeros = torch::from_blob(params->int32_zero_point, {B->row / 128, B->column}, option_int);

//     // torch::Tensor out_feats = gemm_forward_cuda(
//     //     _in_feats,
//     //     _kernel,
//     //     _scaling_factors,
//     //     _zeros,
//     //     8);
    
//     // hipMemcpy(C->data_ptr, out_feats.data_ptr(), C->column * C->row * sizeof(float), hipMemcpyDeviceToHost);
//   };

//   void MatmulOperator::mat_mul_accelerator_int4_fast_no_offset(const struct matmul_params *params) {
// 		// const struct matrix *A = &params->A, *B = &params->B, *C = &params->C;

//     // std::cout << "mat_mul_accelerator_int4_fast_no_offset -- A->row: " << A->row << " A->column: " << A->column 
//               // << "; B->row: " << B->row << " B->column: " << B->column 
//               // << "; C->row: " << C->row << " C->column: " << C->column << std::endl;
    
//     // std::cout << "AAAAA" << std::endl;
    
//     // torch::Tensor out_feats = gemm_forward_cuda(
//     //     torch::from_blob(A->data_ptr, {A->row, A->column}, torch::kHalf),
//     //     torch::from_blob(B->data_ptr, {B->row, B->column}, torch::kInt),
//     //     torch::from_blob(params->scales, {B->row / 128, B->column * 8}, torch::kHalf),
//     //     torch::from_blob(params->int32_zero_point, {B->row / 128, B->column}, torch::kInt),
//     //     8);
    
//     // auto option_fp = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, 1);
//     // auto option_int = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA, 1);

//     // std::cout << "jcuycu" << std::endl;
//     //torch::Tensor _in_feats = torch::from_blob(A->data_ptr, {A->row, A->column}, option_fp);
    
//     // torch::Tensor _in_feats = torch::from_blob(A->data_ptr, {A->row, A->column}, torch::kFloat32).to(torch::kCUDA).to(torch::kHalf);
    
//     // std::cout << "kchy" << std::endl;
//     // torch::Tensor _kernel = torch::from_blob(B->int32_data_ptr, {B->row, B->column}, option_int);
    
//     // torch::Tensor _kernel = torch::from_blob(B->int32_data_ptr, {B->row, B->column}, torch::kInt32).to(torch::kCUDA);

//     // for (int i = 0; i < 128; i++) {
//     //   std::cout << B->int32_data_ptr[i] << " ";
//     // }
//     // std::cout << std::endl;
    

//     // std::cout << "qued" << std::endl;
//     // torch::Tensor _scaling_factors = torch::from_blob(params->scales, {B->row / 128, B->column * 8}, option_fp);
    
//     // torch::Tensor _scaling_factors = torch::from_blob(params->scales, {B->row / QK, B->column * 8}, torch::kFloat32).to(torch::kCUDA).to(torch::kHalf);
    
//     // for (int i = 0; i < 128; i++) { 
//     //   std::cout << params->scales[i] << " ";
//     // }
//     // std::cout << std::endl;
    
//     // std::cout << "-cud" << std::endl;
//     // torch::Tensor _zeros = torch::from_blob(params->int32_zero_point, {B->row / 128, B->column}, option_int);
    
//     // torch::Tensor _zeros = torch::from_blob(params->int32_zero_point, {B->row / QK, B->column}, torch::kInt32).to(torch::kCUDA);
    
//     // for (int i = 0; i < 128; i++) { 
//     //   std::cout << params->int32_zero_point[i] << " ";
//     // }
//     // std::cout << std::endl;
    
//     // std::cout << "msadi" << std::endl;

//     gemm_forward_cuda(params, 1);
    
//     // hipMemcpy(C->data_ptr, out_feats.data_ptr(), C->column * C->row * sizeof(float), hipMemcpyDeviceToHost);


//     // // Testing (Naive implementation)
// 		// const struct matrix *A = &params->A, *B = &params->B, *C = &params->C;
// 		// assert(A->column == B->row / QK);
// 		// assert(C->column == B->column / 8);
// 		// assert(C->row == A->row);

// 		// float *d_A;
// 		// float *d_B;
// 		// float *d_C;

// 		// // Allocate memory
// 		// hipMalloc(&d_A, A->column*A->row*sizeof(float));
// 		// hipMalloc(&d_B, B->column*B->row*sizeof(float));
// 		// hipMalloc(&d_C, C->column*C->row*sizeof(float));

// 		// // Copy data to GPU
// 		// hipMemcpy(d_A, A->data_ptr, A->column*A->row*sizeof(float), hipMemcpyHostToDevice);
// 		// hipMemcpy(d_B, B->data_ptr, B->column*B->row*sizeof(float), hipMemcpyHostToDevice);
// 		// hipMemcpy(d_C, C->data_ptr, C->column*C->row*sizeof(float), hipMemcpyHostToDevice);

//     // matrixMul_block_zp<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, A->row, A->column, C->column);
//   };

// 	void MatmulOperator::mat_mul_cuda(const struct matmul_params *params){
// 		const struct matrix *A = &params->A, *B = &params->B, *C = &params->C;
// 		assert(A->column == B->row);
// 		assert(C->column == B->column);
// 		assert(C->row == A->row);

// 		float *d_A;
// 		float *d_B;
// 		float *d_C;

// 		// Initailize C
// 		/*for (int i = 0; i < C->row; i++)
// 		  for (int j = 0; j < C->column; j++)
// 		  C->data_ptr[j + C->column * i] = 0;*/

// 		// Allocate memory
// 		hipMalloc(&d_A, A->column*A->row*sizeof(float));
// 		hipMalloc(&d_B, B->column*B->row*sizeof(float));
// 		hipMalloc(&d_C, C->column*C->row*sizeof(float));

// 		// Copy data to GPU
// 		hipMemcpy(d_A, A->data_ptr, A->column*A->row*sizeof(float), hipMemcpyHostToDevice);
// 		hipMemcpy(d_B, B->data_ptr, B->column*B->row*sizeof(float), hipMemcpyHostToDevice);
// 		hipMemcpy(d_C, C->data_ptr, C->column*C->row*sizeof(float), hipMemcpyHostToDevice);

// 		// Make sure we can break the input matrix into blocks
// 		assert(A->column % threadDim == 0);
// 		assert(A->row % threadDim == 0);
// 		assert(B->column % threadDim == 0);
// 		const dim3 threadsPerBlock(threadDim, threadDim);
//     const dim3 numBlocks((C->column + threadDim - 1) / threadsPerBlock.x, 
//                          (C->row + threadDim - 1) / threadsPerBlock.y);  // Using ceil to cover all elements

// 		// Invoke the cuda imp.

// 		// struct timeval start, end;
// 		// gettimeofday(&start, NULL);
// 		// matrixMul_blockC<<< numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, A->row, A->column, B->column);
// 		matrixMultiplyShared<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, A->row, A->column, B->column);
// 		hipDeviceSynchronize();
// 		// gettimeofday(&end, NULL);
// 		// int us = interval_to_us(&start, &end);
// 		// // std::cout << "cuda kernel: " << us / 1000 << " ms" << std::endl;

// 		// Get the result back
// 		hipMemcpy(C->data_ptr, d_C, C->column*C->row*sizeof(float), hipMemcpyDeviceToHost);
// 	}
// }  // namespace matmul
