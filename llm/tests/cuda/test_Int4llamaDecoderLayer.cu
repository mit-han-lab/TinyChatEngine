#include "Int4llamaDecoderLayer.h"
#include "operators.h"
#include "utils.h"

int NUM_THREAD = 8;

static void Int4LLaMAFreeMemory() {
    // Int4llamaDecoderLayer
    Int4llamaDecoderLayer llamaDecoderLayer;
    llamaDecoderLayer.free_cuda_memory();

    // Int4llamaAttention
    Int4llamaAttention llamaAttention;
    llamaAttention.free_cuda_memory();
}

void test_Int4llamaDecoderLayer() {
    const struct model_config llama7B = llama_7B;
    const int sqlen = 9, b = 1, embed_dim = llama7B.embed_dim, num_heads = llama7B.num_heads;

    Int4llamaDecoderLayer layer = Int4llamaDecoderLayer("INT4/models/LLaMA_7B_2_chat/decoder/layer0", llama7B, 0);

    half* buffer_1;
    hipMallocManaged(&buffer_1, sizeof(half) * embed_dim * sqlen * b);
    Matrix3D<half> hidden_states(buffer_1, b, sqlen, embed_dim);
    read_to_array_half("assets/llama/tests/layer0/sqlen9/hidden_states_half.bin", hidden_states.m_data, hidden_states.length());
    half* buffer_2;
    hipMallocManaged(&buffer_2, sizeof(half) * sqlen * sqlen);
    Matrix3D<half> attention_mask(buffer_2, 1, sqlen, sqlen);
    read_to_array_half("assets/llama/tests/layer0/sqlen9/attention_mask_half.bin", attention_mask.m_data, attention_mask.length());

    struct Int4llamaDecoderLayer_input input(hidden_states, attention_mask);
    struct Int4llamaDecoderLayer_output output = layer.forward("INT4/models/LLaMA_7B_2_chat/decoder/layer0", input, 0);
    hipDeviceSynchronize();

    half* buffer_3;
    hipMallocManaged(&buffer_3, sizeof(half) * embed_dim * sqlen * b);
    Matrix3D<half> outputGT(buffer_3, b, sqlen, embed_dim);
    read_to_array_half("assets/llama/tests/layer0/sqlen9/output_hidden_states_half.bin", outputGT.m_data, outputGT.length());

    half* buffer_4;
    hipMallocManaged(&buffer_4, sizeof(half) * embed_dim * sqlen * b);
    Matrix3D<half> key_statesGT(buffer_4, num_heads, sqlen, embed_dim / num_heads);
    read_to_array_half("assets/llama/tests/layer0/sqlen9/present_key_half.bin", key_statesGT.m_data, key_statesGT.length());

    half* buffer_5;
    hipMallocManaged(&buffer_5, sizeof(half) * embed_dim * sqlen * b);
    Matrix3D<half> value_statesGT(buffer_5, num_heads, sqlen, embed_dim / num_heads);
    read_to_array_half("assets/llama/tests/layer0/sqlen9/present_value_half.bin", value_statesGT.m_data, value_statesGT.length());

    bool success = check_two_equal_half_half(value_statesGT.m_data, output.past_key_value.second.m_data, value_statesGT.length());
    success &= check_two_equal_half_half(key_statesGT.m_data, output.past_key_value.first.m_data, key_statesGT.length());
    success &= check_two_equal_half_half(outputGT.m_data, output.hidden_states.m_data, outputGT.length());
    if (!success)
        std::cout << "Test of " << __func__ << ": Fail!" << std::endl;
    else
        std::cout << "-------- Test of " << __func__ << ": Passed! -------- " << std::endl;

    // Free memory
    free_aligned_memory_gpu(buffer_1);
    free_aligned_memory_gpu(buffer_2);
    free_aligned_memory_gpu(buffer_3);
    free_aligned_memory_gpu(buffer_4);
    free_aligned_memory_gpu(buffer_5);
    Int4LLaMAFreeMemory();
}

void test_Int4llamaDecoderLayer_gen() {
    const struct model_config llama7B = llama_7B;
    const int sqlen = 1, b = 1, past_sqlen = 9, embed_dim = llama7B.embed_dim, num_heads = llama7B.num_heads,
              head_dim = embed_dim / num_heads;
    const int tgz = (sqlen + past_sqlen);

    Int4llamaDecoderLayer layer = Int4llamaDecoderLayer("INT4/models/LLaMA_7B_2_chat/decoder/layer0", llama7B, 0);

    half* buffer_1;
    hipMallocManaged(&buffer_1, sizeof(half) * embed_dim * sqlen * b);
    Matrix3D<half> hidden_states(buffer_1, b, sqlen, embed_dim);
    read_to_array_half("assets/llama/tests/layer0/sqlen1/hidden_states_half.bin", hidden_states.m_data, hidden_states.length());
    half* buffer_2;
    hipMallocManaged(&buffer_2, sizeof(half) * sqlen * tgz);
    Matrix3D<half> attention_mask(buffer_2, 1, sqlen, tgz);
    read_to_array_half("assets/llama/tests/layer0/sqlen1/attention_mask_half.bin", attention_mask.m_data, attention_mask.length());
    half* buffer_3;
    hipMallocManaged(&buffer_3, sizeof(half) * num_heads * past_sqlen * embed_dim);
    Matrix3D<half> past_key(buffer_3, num_heads, past_sqlen, head_dim);
    read_to_array_half("assets/llama/tests/atten/sqlen9/past_key_half.bin", past_key.m_data, past_key.length());
    half* buffer_4;
    hipMallocManaged(&buffer_4, sizeof(half) * num_heads * past_sqlen * embed_dim);
    Matrix3D<half> past_value(buffer_4, num_heads, past_sqlen, head_dim);
    read_to_array_half("assets/llama/tests/atten/sqlen9/past_value_half.bin", past_value.m_data, past_value.length());

    struct Int4llamaDecoderLayer_input input(hidden_states, attention_mask, past_key, past_value);
    struct Int4llamaDecoderLayer_output output = layer.forward("INT4/models/LLaMA_7B_2_chat/decoder/layer0", input, 0);
    hipDeviceSynchronize();

    half* buffer_5;
    hipMallocManaged(&buffer_5, sizeof(half) * embed_dim * sqlen * b);
    Matrix3D<half> outputGT(buffer_5, b, sqlen, embed_dim);
    read_to_array_half("assets/llama/tests/layer0/sqlen1/output_hidden_states_half.bin", outputGT.m_data, outputGT.length());

    half* buffer_6;
    hipMallocManaged(&buffer_6, sizeof(half) * embed_dim * sqlen * b);
    Matrix3D<half> key_statesGT(buffer_6, num_heads, sqlen, embed_dim / num_heads);
    read_to_array_half("assets/llama/tests/layer0/sqlen1/present_key_half.bin", key_statesGT.m_data, key_statesGT.length());

    half* buffer_7;
    hipMallocManaged(&buffer_7, sizeof(half) * embed_dim * sqlen * b);
    Matrix3D<half> value_statesGT(buffer_7, num_heads, sqlen, embed_dim / num_heads);
    read_to_array_half("assets/llama/tests/layer0/sqlen1/present_value_half.bin", value_statesGT.m_data, value_statesGT.length());

    bool success = check_two_equal_half_half(value_statesGT.m_data, output.past_key_value.second.m_data, value_statesGT.length());
    success &= check_two_equal_half_half(key_statesGT.m_data, output.past_key_value.first.m_data, key_statesGT.length());
    success &= check_two_equal_half_half(outputGT.m_data, output.hidden_states.m_data, outputGT.length());
    if (!success)
        std::cout << "Test of " << __func__ << ": Fail!" << std::endl;
    else
        std::cout << "-------- Test of " << __func__ << ": Passed! -------- " << std::endl;

    // Free memory
    free_aligned_memory_gpu(buffer_1);
    free_aligned_memory_gpu(buffer_2);
    free_aligned_memory_gpu(buffer_3);
    free_aligned_memory_gpu(buffer_4);
    free_aligned_memory_gpu(buffer_5);
    free_aligned_memory_gpu(buffer_6);
    free_aligned_memory_gpu(buffer_7);
    Int4LLaMAFreeMemory();
}

int main() {
    test_Int4llamaDecoderLayer();
    test_Int4llamaDecoderLayer_gen();
}
