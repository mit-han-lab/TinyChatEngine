#include <cstring>

#include "Int4llamaDecoder.h"
#include "operators.h"
#include "utils.h"

int NUM_THREAD = 8;

static void Int4LLaMAFreeMemory() {
    // Int4llamaDecoder
    Int4llamaDecoder llamaDecoder;
    llamaDecoder.free_cuda_memory();

    // Int4llamaDecoderLayer
    Int4llamaDecoderLayer llamaDecoderLayer;
    llamaDecoderLayer.free_cuda_memory();

    // Int4llamaAttention
    Int4llamaAttention llamaAttention;
    llamaAttention.free_cuda_memory();
}

void test_Decoder() {
    const struct model_config llama7B = llama_7B;
    const int sqlen = 9, b = 1, embed_dim = llama7B.embed_dim, num_heads = llama7B.num_heads, num_layers = llama7B.num_layers;

    int* buffer_1;
    hipMallocManaged(&buffer_1, sizeof(int) * sqlen);
    Matrix3D<int> input_ids(buffer_1, b, 1, sqlen);
    input_ids.load("assets/llama/tests/decoder/1st_input_ids.bin");
    struct Int4llamaDecoder_input input_1st = {input_ids};

    Int4llamaDecoder decoder = Int4llamaDecoder("INT4/models/LLaMA_7B_2_chat/decoder/", llama7B);
    struct Int4llamaDecoder_output output_1st = decoder.forward("INT4/models/LLaMA_7B_2_chat/decoder/", input_1st);
    hipDeviceSynchronize();

    half* buffer_2;
    hipMallocManaged(&buffer_2, sizeof(half) * b * sqlen * embed_dim);
    Matrix3D<half> last_hidden_state1_GT(buffer_2, b, sqlen, embed_dim);
    read_to_array_half("assets/llama/tests/decoder/1st_last_hidden_state_half.bin", last_hidden_state1_GT.m_data, last_hidden_state1_GT.length());

    bool success = check_two_equal_half_half(last_hidden_state1_GT.m_data, output_1st.last_hidden_state.m_data,
                                   last_hidden_state1_GT.length());

    half* buffer_3;
    hipMallocManaged(&buffer_3, sizeof(half) * b * sqlen * embed_dim);
    Matrix3D<half> temp_key_value(buffer_3, num_heads, sqlen, embed_dim / num_heads);
    for (int i = 0; i < num_layers; i++) {
        std::string path = "assets/llama/tests/decoder/1st/past_key_value/key" + std::to_string(i) + "_half.bin";
        read_to_array_half(path.c_str(), temp_key_value.m_data, temp_key_value.length());
        success &= check_two_equal_half_half(temp_key_value.m_data, output_1st.past_keys[i].m_data, temp_key_value.length());

        path = "assets/llama/tests/decoder/1st/past_key_value/value" + std::to_string(i) + "_half.bin";
        read_to_array_half(path.c_str(), temp_key_value.m_data, temp_key_value.length());
        success &= check_two_equal_half_half(temp_key_value.m_data, output_1st.past_values[i].m_data, temp_key_value.length());
    }

    // generating phase: 2nd run
    int* buffer_4;
    hipMallocManaged(&buffer_4, sizeof(int) * sqlen);
    Matrix3D<int> input_ids_2nd(buffer_4, b, 1, 1);
    input_ids_2nd.load("assets/llama/tests/decoder/2nd/input_ids.bin");

    struct Int4llamaDecoder_input input_2nd = {input_ids_2nd, output_1st.past_keys, output_1st.past_values};
    struct Int4llamaDecoder_output output_2nd = decoder.forward("INT4/models/LLaMA_7B_2_chat/decoder/", input_2nd);
    hipDeviceSynchronize();

    half* buffer_5;
    hipMallocManaged(&buffer_5, sizeof(half) * b * 1 * embed_dim);
    Matrix3D<half> last_hidden_state2_GT(buffer_5, b, 1, embed_dim);
    read_to_array_half("assets/llama/tests/decoder/2nd/last_hidden_state_half.bin", last_hidden_state2_GT.m_data, last_hidden_state2_GT.length());

    success &= check_two_equal_half_half(last_hidden_state2_GT.m_data, output_2nd.last_hidden_state.m_data,
                               last_hidden_state2_GT.length());

    half* buffer_6;
    hipMallocManaged(&buffer_6, sizeof(half) * b * (sqlen + 1) * embed_dim);
    Matrix3D<half> temp_key_value_2nd(buffer_6, num_heads, (sqlen + 1), embed_dim / num_heads);
    for (int i = 0; i < num_layers; i++) {
        std::string path = "assets/llama/tests/decoder/2nd/past_key_value/key" + std::to_string(i) + "_half.bin";
        read_to_array_half(path.c_str(), temp_key_value_2nd.m_data, temp_key_value_2nd.length());
        success &= check_two_equal_half_half(temp_key_value_2nd.m_data, output_2nd.past_keys[i].m_data, temp_key_value_2nd.length());

        path = "assets/llama/tests/decoder/2nd/past_key_value/value" + std::to_string(i) + "_half.bin";
        read_to_array_half(path.c_str(), temp_key_value_2nd.m_data, temp_key_value_2nd.length());
        success &= check_two_equal_half_half(temp_key_value_2nd.m_data, output_2nd.past_values[i].m_data, temp_key_value_2nd.length());
    }

    if (!success)
        std::cout << "-------- Test of " << __func__ << ": Fail! -------- " << std::endl;
    else
        std::cout << "-------- Test of " << __func__ << ": Passed! -------- " << std::endl;

    // Free memory
    free_aligned_memory_gpu(buffer_1);
    free_aligned_memory_gpu(buffer_2);
    free_aligned_memory_gpu(buffer_3);
    free_aligned_memory_gpu(buffer_4);
    free_aligned_memory_gpu(buffer_5);
    free_aligned_memory_gpu(buffer_6);
    Int4LLaMAFreeMemory();
}

int main() { test_Decoder(); }
