#include "Int4llamaAttention.h"
#include "operators.h"
#include "utils.h"
// #include "utils.cuh"
#include "utils_memalloc.cuh"

void test_Int4llamaAttention() {
    const struct model_config llama7B = llama_7B;
    const int sqlen = 9, b = 1, embed_dim = llama7B.embed_dim, num_heads = llama7B.num_heads;

    MemoryAllocator mem_buf;

    Int4llamaAttention attn = Int4llamaAttention("models/LLaMA_7B/decoder/layer0/self_attn", llama7B);

    float* buffer_1;
    hipMallocManaged(&buffer_1, sizeof(float) * embed_dim * sqlen * b);
    Matrix3D<float> hidden_states_float(buffer_1, b, sqlen, embed_dim);
    read_to_array("assets/llama/tests/atten/sqlen9/hidden_states.bin", hidden_states_float.m_data, b * sqlen * embed_dim);

    half* buffer_1_half;
    hipMallocManaged(&buffer_1_half, sizeof(half) * embed_dim * sqlen * b);
    Matrix3D<half> hidden_states(buffer_1_half, b, sqlen, embed_dim);
    int threadsPerBlock = 1024;
    int blocksPerGrid_1 =(b * sqlen * embed_dim + threadsPerBlock - 1) / threadsPerBlock;
    float2half<<<blocksPerGrid_1, threadsPerBlock>>>(buffer_1, buffer_1_half, b * sqlen * embed_dim);
    hipDeviceSynchronize();


    float* buffer_2;
    hipMallocManaged(&buffer_2, sizeof(float) * sqlen * sqlen);
    Matrix3D<float> attention_mask_float(buffer_2, 1, sqlen, sqlen);
    read_to_array("assets/llama/tests/atten/sqlen9/attention_mask.bin", attention_mask_float.m_data, attention_mask_float.length());

    half* buffer_2_half;
    hipMallocManaged(&buffer_2_half, sizeof(half) * sqlen * sqlen);
    Matrix3D<half> attention_mask(buffer_2_half, 1, sqlen, sqlen);
    int blocksPerGrid_2 =(sqlen * sqlen + threadsPerBlock - 1) / threadsPerBlock;
    float2half<<<blocksPerGrid_2, threadsPerBlock>>>(buffer_2, buffer_2_half, sqlen * sqlen);
    hipDeviceSynchronize();


    attn.initialized_memory(llama7B);
    struct Int4llamaAttention_input input(hidden_states, attention_mask, 0);
    struct Int4llamaAttention_output output = attn.forward(input);
    hipDeviceSynchronize();

    float* buffer_3;
    hipMallocManaged(&buffer_3, sizeof(float) * embed_dim * sqlen * b);
    Matrix3D<float> attn_outputGT(buffer_3, b, sqlen, embed_dim);
    read_to_array("assets/llama/tests/atten/sqlen9/attn_output.bin", attn_outputGT.m_data, b * sqlen * embed_dim);

    float* buffer_4;
    hipMallocManaged(&buffer_4, sizeof(float) * embed_dim * sqlen * b);
    Matrix3D<float> key_statesGT(buffer_4, num_heads, sqlen, embed_dim / num_heads);
    read_to_array("assets/llama/tests/atten/sqlen9/past_key.bin", key_statesGT.m_data, b * sqlen * embed_dim);

    float* buffer_5;
    hipMallocManaged(&buffer_5, sizeof(float) * embed_dim * sqlen * b);
    Matrix3D<float> value_statesGT(buffer_5, num_heads, sqlen, embed_dim / num_heads);
    read_to_array("assets/llama/tests/atten/sqlen9/past_value.bin", value_statesGT.m_data, b * sqlen * embed_dim);

    bool success = check_two_equal_float_half(value_statesGT.m_data, output.past_key_value.second.m_data, value_statesGT.length());
    success &= check_two_equal_float_half(key_statesGT.m_data, output.past_key_value.first.m_data, key_statesGT.length());
    success &= check_two_equal_float_half(attn_outputGT.m_data, output.attn_output.m_data, attn_outputGT.length());
    if (!success)
        std::cout << "Test of " << __func__ << ": Fail!" << std::endl;
    else
        std::cout << "-------- Test of " << __func__ << ": Passed! -------- " << std::endl;

    // Free memory
    hipFree(buffer_1);
    hipFree(buffer_1_half);
    hipFree(buffer_2);
    hipFree(buffer_2_half);
    hipFree(buffer_3);
    hipFree(buffer_4);
    hipFree(buffer_5);
}

void test_Int4llamaAttention_gen() {
    const struct model_config llama7B = llama_7B;
    const int sqlen = 1, b = 1, past_sqlen = 9, embed_dim = llama7B.embed_dim, num_heads = llama7B.num_heads,
              head_dim = embed_dim / num_heads;

    MemoryAllocator mem_buf;

    Int4llamaAttention attn = Int4llamaAttention("models/LLaMA_7B/decoder/layer0/self_attn", llama7B);

    float* buffer_1;
    hipMallocManaged(&buffer_1, sizeof(float) * embed_dim * sqlen * b);
    Matrix3D<float> hidden_states_float(buffer_1, b, sqlen, embed_dim);
    hidden_states_float.load("assets/llama/tests/atten/sqlen1/hidden_states.bin");

    half* buffer_1_half;
    hipMallocManaged(&buffer_1_half, sizeof(half) * embed_dim * sqlen * b);
    Matrix3D<half> hidden_states(buffer_1_half, b, sqlen, embed_dim);
    int threadsPerBlock = 1024;
    int blocksPerGrid_1 =(b * sqlen * embed_dim + threadsPerBlock - 1) / threadsPerBlock;
    float2half<<<blocksPerGrid_1, threadsPerBlock>>>(buffer_1, buffer_1_half, b * sqlen * embed_dim);
    hipDeviceSynchronize();


    float* buffer_2;
    hipMallocManaged(&buffer_2, sizeof(float) * sqlen * (sqlen + past_sqlen));
    Matrix3D<float> attention_mask_float(buffer_2, b, sqlen, sqlen + past_sqlen);
    attention_mask_float.load("assets/llama/tests/atten/sqlen1/attention_mask.bin");

    half* buffer_2_half;
    hipMallocManaged(&buffer_2_half, sizeof(half) * sqlen * (sqlen + past_sqlen));
    Matrix3D<half> attention_mask(buffer_2_half, b, sqlen, sqlen + past_sqlen);
    int blocksPerGrid_2 =(sqlen * (sqlen + past_sqlen) + threadsPerBlock - 1) / threadsPerBlock;
    float2half<<<blocksPerGrid_2, threadsPerBlock>>>(buffer_2, buffer_2_half, sqlen * (sqlen + past_sqlen));
    hipDeviceSynchronize();


    float* buffer_3;
    hipMallocManaged(&buffer_3, sizeof(float) * num_heads * past_sqlen * embed_dim);
    Matrix3D<float> past_key_float(buffer_3, num_heads, past_sqlen, head_dim);
    past_key_float.load("assets/llama/tests/atten/sqlen9/past_key.bin");

    half* buffer_3_half;
    hipMallocManaged(&buffer_3_half, sizeof(half) * num_heads * past_sqlen * embed_dim);
    Matrix3D<half> past_key(buffer_3_half, num_heads, past_sqlen, head_dim);
    int blocksPerGrid_3 =(num_heads * past_sqlen * embed_dim + threadsPerBlock - 1) / threadsPerBlock;
    float2half<<<blocksPerGrid_3, threadsPerBlock>>>(buffer_3, buffer_3_half, num_heads * past_sqlen * embed_dim);
    hipDeviceSynchronize();


    float* buffer_4;
    hipMallocManaged(&buffer_4, sizeof(float) * num_heads * past_sqlen * embed_dim);
    Matrix3D<float> past_value_float(buffer_4, num_heads, past_sqlen, head_dim);
    past_value_float.load("assets/llama/tests/atten/sqlen9/past_value.bin");

    half* buffer_4_half;
    hipMallocManaged(&buffer_4_half, sizeof(half) * num_heads * past_sqlen * embed_dim);
    Matrix3D<half> past_value(buffer_4_half, num_heads, past_sqlen, head_dim);
    int blocksPerGrid_4 =(num_heads * past_sqlen * embed_dim + threadsPerBlock - 1) / threadsPerBlock;
    float2half<<<blocksPerGrid_4, threadsPerBlock>>>(buffer_4, buffer_4_half, num_heads * past_sqlen * embed_dim);
    hipDeviceSynchronize();


    attn.initialized_memory(llama7B);
    struct Int4llamaAttention_input input(hidden_states, attention_mask, past_key, past_value, true, 0);
    struct Int4llamaAttention_output output = attn.forward(input);
    hipDeviceSynchronize();

    float* buffer_5;
    hipMallocManaged(&buffer_5, sizeof(float) * embed_dim * sqlen * b);
    Matrix3D<float> attn_outputGT(buffer_5, b, sqlen, embed_dim);
    attn_outputGT.load("assets/llama/tests/atten/sqlen1/attn_output.bin");

    float* buffer_6;
    hipMallocManaged(&buffer_6, sizeof(float) * (sqlen + past_sqlen) * embed_dim);
    Matrix3D<float> key_statesGT(buffer_6, num_heads, sqlen + past_sqlen,
                                 embed_dim / num_heads);
    key_statesGT.load("assets/llama/tests/atten/sqlen1/past_key.bin");

    float* buffer_7;
    hipMallocManaged(&buffer_7, sizeof(float) * (sqlen + past_sqlen) * embed_dim);
    Matrix3D<float> value_statesGT(buffer_7, num_heads,
                                   sqlen + past_sqlen, embed_dim / num_heads);
    value_statesGT.load("assets/llama/tests/atten/sqlen1/past_value.bin");

    bool success = check_two_equal_float_half(value_statesGT.m_data, output.past_key_value.second.m_data, value_statesGT.length());
    success &= check_two_equal_float_half(key_statesGT.m_data, output.past_key_value.first.m_data, key_statesGT.length());
    success &= check_two_equal_float_half(attn_outputGT.m_data, output.attn_output.m_data, attn_outputGT.length());
    if (!success)
        std::cout << "Test of " << __func__ << ": Fail!" << std::endl;
    else
        std::cout << "-------- Test of " << __func__ << ": Passed! -------- " << std::endl;

    // Free memory
    hipFree(buffer_1);
    hipFree(buffer_1_half);
    hipFree(buffer_2);
    hipFree(buffer_2_half);
    hipFree(buffer_3);
    hipFree(buffer_3_half);
    hipFree(buffer_4);
    hipFree(buffer_4_half);
    hipFree(buffer_5);
    hipFree(buffer_6);
    hipFree(buffer_7);
}

int main() {
    test_Int4llamaAttention();
    test_Int4llamaAttention_gen();
}
