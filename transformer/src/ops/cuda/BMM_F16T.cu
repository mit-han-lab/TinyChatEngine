#include "hip/hip_runtime.h"
#include "operators.cuh"
#include "utils.h"
#include "lib/matmul.h"

__global__ void mat_mul_transposed_cuda(const struct matmul_params* params, const float alpha) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < params->C.row && j < params->C.column) {
        const struct matrix *A = &params->A, *B = &params->B, *C = &params->C;
        float* data_A = A->data_ptr, *data_B = B->data_ptr, *data_C = C->data_ptr;

        float acc = 0;
        for (int k = 0; k < A->column; k++) {
            acc += data_A[i * A->column + k] * data_B[j * B->column + k];
        }
        
        data_C[i * C->column + j] = alpha * acc;
    }
}

void load_BMM_F16T(BMM_F16T &op, std::string prefix) { read_to_array((prefix + "/alpha.bin").c_str(), &op.alpha, 1); }

BMM_F16T::BMM_F16T(float _alpha) { this->alpha = _alpha; }

void BMM_F16T::forward(const Matrix3D_cuda<float> &a, const Matrix3D_cuda<float> &weight, Matrix3D<float> &c) {
    const Matrix3D_cuda<float> b = weight;
    const int m = a.m_dim_y, n = b.m_dim_y, k = a.m_dim_z, b_size = b.m_dim_x;
    const long long ops = (long long)b_size * 2 * (long long)m * (long long)n * (long long)k;
    PROFILE_START_FLOPS(profile_name, ops);

    // a: m x k   b: n x k   c: m x n
    assert(a.m_dim_x == b.m_dim_x);  // batch dim
    assert(a.m_dim_z == b.m_dim_z);  // k
    assert(a.m_dim_y == c.m_dim_y);  // m
    assert(b.m_dim_y == c.m_dim_z);  // n

    struct matmul_params params;
    params.A.row = a.m_dim_y;
    params.A.column = a.m_dim_z;
    params.A.data_ptr = a.m_data;
    params.B.row = b.m_dim_y;
    params.B.column = b.m_dim_z;
    params.B.data_ptr = b.m_data;
    params.C.row = c.m_dim_y;
    params.C.column = c.m_dim_z;
    params.C.data_ptr = c.m_data;
    // params.opt_params.blk_size = BLK_SIZE;
    // params.opt_params.num_thread = NUM_THREAD;
    params.alpha = alpha;

    matmul::MatmulOperator op = matmul::MatmulOperator();

    for (int bz = 0; bz < a.m_dim_x; bz++) {
        // if (params.A.column % 8 == 0) // TODO: debug this
        //     op.mat_mul_transposed_fastover_column((const struct matmul_params
        //     *)&params);
        // else

        // op.mat_mul_transposed(&params);  // TODO: optimize this
        dim3 block(16, 16);  // You might want to optimize this size.
        dim3 grid((params.C.row + block.x - 1) / block.x, (params.C.column + block.y - 1) / block.y);
        // printf("bz: %d\n", bz);
        mat_mul_transposed_cuda<<<grid, block>>>(&params, this->alpha);
        hipDeviceSynchronize();

        // TODO: apply SIMD here
        // for (int i = 0; i < m * n; i++) {
        //     params.C.data_ptr[i] *= this->alpha;
        // }
        params.A.data_ptr += m * k;
        params.B.data_ptr += k * n;
        params.C.data_ptr += m * n;
    }

    PROFILE_END(profile_name);
}
