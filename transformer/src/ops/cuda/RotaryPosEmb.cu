#include "hip/hip_runtime.h"
#include "operators.cuh"
#include <cmath>

// __global__ void RotaryPosEmb_half_forward(Matrix3D<float> &query, Matrix3D<float> &key, Matrix3D<float> &cos, Matrix3D<float> &sin, int start_idx, int len) {
//   __shared__ float query_buf[4096], key_buf[4096];

//   int num_heads = query.m_dim_x;
//   int head_embed = cos.m_dim_z;
//   // int max_sqlen = cos.m_dim_y;

//   int half = head_embed / 2;
//   int total_threads = blockDim.x * gridDim.x;
//   int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  
//   // Convert the 1D CUDA thread indices into 3D indices
//   int b = global_thread_id / (head_embed * len);
//   int i = (global_thread_id / head_embed) % len;
//   int j = global_thread_id % head_embed;

//   if (b < num_heads && i < len && j < head_embed) {
//     if (j < half) {
//       // query_buf[j] = -1 * query[(b * len + i) * head_embed + j + half];
//       // key_buf[j] = -1 * key[(b * len + i) * head_embed + j + half];
//       query_buf[j] = -1 * query(b, i, j + half);
//       key_buf[j] = -1 * key(b, i, j + half);
//     }
//     else {
//       // query_buf[j] = query[(b * len + i) * head_embed + j - half];
//       // key_buf[j] = key[(b * len + i) * head_embed + j - half];
//       query_buf[j] = query(b, i, j - half);
//       key_buf[j] = key(b, i, j - half);
//     }

//     query(b, i, j) = ((query(b, i, j) * cos(0, i + start_idx, j)) +
//                       (query_buf[j] * sin(0, i + start_idx, j)));
//     key(b, i, j) = ((key(b, i, j) * cos(0, i + start_idx, j)) +
//                     (key_buf[j] * sin(0, i + start_idx, j)));
//   }
// }

__global__ void RotaryPosEmb_half_forward(Matrix3D<float> query, Matrix3D<float> key, Matrix3D<float> cos, Matrix3D<float> sin, int start_idx, int len) {
  // TODO: maybe we can use shared memory here
  float query_buf[4096], key_buf[4096];

  int num_heads = query.m_dim_x;
  int head_embed = cos.m_dim_z;
  // int max_sqlen = cos.m_dim_y;
  int half = head_embed / 2;
  
  // Convert the 1D CUDA thread indices into 3D indices
  int b = blockIdx.x;
  int i = threadIdx.x;

  if(b < num_heads && i < len) {
    for(int j = 0; j < half; j++) {
      // query_buf[j] = -1 * query[(b * len + i) * head_embed + j + half];
      // key_buf[j] = -1 * key[(b * len + i) * head_embed + j + half];
      query_buf[j] = -1 * query(b, i, j + half);
      key_buf[j] = -1 * key(b, i, j + half);
    }
    for(int j = half; j < head_embed; j++) {
      // query_buf[j] = query[(b * len + i) * head_embed + j - half];
      // key_buf[j] = key[(b * len + i) * head_embed + j - half];
      query_buf[j] = query(b, i, j - half);
      key_buf[j] = key(b, i, j - half);
    }

    for(int j = 0; j < head_embed; j++) {
      query(b, i, j) = ((query(b, i, j) * cos(0, i + start_idx, j)) +
                        (query_buf[j] * sin(0, i + start_idx, j)));
      key(b, i, j) = ((key(b, i, j) * cos(0, i + start_idx, j)) +
                      (key_buf[j] * sin(0, i + start_idx, j)));
    }
  }
}



// // TODO: optimize this with multithreading
// __global__ void RotaryPosEmb_kernel(Matrix3D<float> &query, Matrix3D<float> &key, Matrix3D<float> &cos, Matrix3D<float> &sin, 
//            int start_idx, int num_heads, int len, int head_embed) {
//   __shared__ float query_buf[4096], key_buf[4096];

//   int half = head_embed / 2;
//   int total_threads = blockDim.x * gridDim.x;
//   int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  
//   // Convert the 1D CUDA thread indices into 3D indices
//   int b = global_thread_id / (head_embed * len);
//   int i = (global_thread_id / head_embed) % len;
//   int j = global_thread_id % head_embed;

//   if (b < num_heads && i < len && j < head_embed) {
//     if (j < half) {
//       // query_buf[j] = -1 * query[(b * len + i) * head_embed + j + half];
//       // key_buf[j] = -1 * key[(b * len + i) * head_embed + j + half];
//       query_buf[j] = -1 * query(b, i, j + half);
//       key_buf[j] = -1 * key(b, i, j + half);
//     }
//     else {
//       // query_buf[j] = query[(b * len + i) * head_embed + j - half];
//       // key_buf[j] = key[(b * len + i) * head_embed + j - half];
//       query_buf[j] = query(b, i, j - half);
//       key_buf[j] = key(b, i, j - half);
//     }

//     query(b, i, j) = ((query(b, i, j) * cos(0, i + start_idx, j)) +
//                       (query_buf[j] * sin(0, i + start_idx, j)));
//     key(b, i, j) = ((key(b, i, j) * cos(0, i + start_idx, j)) +
//                     (key_buf[j] * sin(0, i + start_idx, j)));
//   }
// }

// void RotaryPosEmb_half::forward(Matrix3D<float> &query, Matrix3D<float> &key,
//                            int start_idx, int len) {
//   PROFILE_START(profile_name);
//   int num_heads = query.m_dim_x;
//   int head_embed = cos.m_dim_z;
//   int max_sqlen = cos.m_dim_y;

//   assert(query.m_dim_z == cos.m_dim_z);
//   assert(key.m_dim_z == cos.m_dim_z);
//   assert(max_sqlen > len + start_idx);

//   // int half = head_embed / 2;
//   // for (int b = 0; b < num_heads; b++) {
//   //   for (int i = 0; i < len; i++) {
//   //     // first half
//   //     for (int j = 0; j < half; j++) {
//   //       q_buf[j] = -1 * query(b, i, j + half);
//   //       k_buf[j] = -1 * key(b, i, j + half);
//   //     }
//   //     // second half
//   //     for (int j = half; j < head_embed; j++) {
//   //       q_buf[j] = query(b, i, j - half);
//   //       k_buf[j] = key(b, i, j - half);
//   //     }

//   //     for (int j = 0; j < head_embed; j++) {
//   //       query(b, i, j) = ((query(b, i, j) * cos(0, i + start_idx, j)) +
//   //                         (q_buf[j] * sin(0, i + start_idx, j)));
//   //       key(b, i, j) = ((key(b, i, j) * cos(0, i + start_idx, j)) +
//   //                       (k_buf[j] * sin(0, i + start_idx, j)));
//   //     }
//   //   }
//   // }

//   dim3 block(1024);
//   dim3 grid((num_heads * head_embed * len + block.x - 1) / block.x);
//   RotaryPosEmb_kernel<<<grid, block>>>(query, key, cos, sin, start_idx, num_heads, len, head_embed);

//   PROFILE_END(profile_name);
// }
