#include "hip/hip_runtime.h"
#include <cmath>
#include <iomanip>

#include "operators.h"
#include "utils.h"

// __global__ void LlamaRMSNorm_float_kernel(const Matrix3D<float> x, const Matrix3D<float> weight, Matrix3D<float> output, float eps) {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     int j = blockIdx.y * blockDim.y + threadIdx.y;
    
//     if (i < x.m_dim_x && j < x.m_dim_y) {
//         float var = 0.0f;

//         for (int k = 0; k < x.m_dim_z; k++) {  // hidden states
//             var += x(i, j, k) * x(i, j, k);
//         }

//         var /= static_cast<float>(x.m_dim_z);
//         float variance = 1.0 / sqrtf(var + eps);

//         for (int k = 0; k < x.m_dim_z; k++) {
//             float value = static_cast<float>(x(i, j, k));
//             float fp_out = (value * variance) * weight(0, 0, k);
            
//             output(i, j, k) = fp_out;
//         }
//     }
// }

__global__ void LlamaRMSNorm_cuda_kernel(const Matrix3D<half> x, const Matrix3D<float> weight, Matrix3D<half> output, float eps) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // half epf_half = __float2half(1e-5f);
    
    if (i < x.m_dim_x && j < x.m_dim_y) {
        // //// fp16 version
        // half var = 0;

        // for (int k = 0; k < x.m_dim_z; k++) {  // hidden states
        //     // var = __hadd(var, __hmul(x(i, j, k), x(i, j, k)));
        //     var = __hfma(x(i, j, k), x(i, j, k), var);
        // }

        // var = __hdiv(var, __int2half_rn(x.m_dim_z));
        // half variance = hrcp(hsqrt(__hadd(var, eps)));
        // // half variance = __float2half(1.0f / sqrtf(__half2float(var) + eps));

        // for (int k = 0; k < x.m_dim_z; k++) {
        //     half value = x(i, j, k);
        //     // half half_out = __hmul(__hmul(value, variance), __float2half(weight(0, 0, k)));  // TODO: convert weight to half
        //     half half_out = __float2half(__half2float(__hmul(value, variance)) * weight(0, 0, k));  // TODO: convert weight to half
            
        //     output(i, j, k) = half_out;
        // }


        //// fp32 version
        float var = 0.0f;

        for (int k = 0; k < x.m_dim_z; k++) {  // hidden states
            float value = __half2float(x(i, j, k));
            var += value * value;
        }

        var /= static_cast<float>(x.m_dim_z);
        float variance = rsqrtf(var + eps);

        for (int k = 0; k < x.m_dim_z; k++) {
            float value = __half2float(x(i, j, k));
            float fp_out = (value * variance) * weight(0, 0, k);
            
            // output(i, j, k) = fp_out;
            output(i, j, k) = __float2half(fp_out);
        }


        // //// fp16 version
        // half var = 0;

        // for (int k = 0; k < x.m_dim_z; k++) {  // hidden states
        //     half value = x(i, j, k);
        //     var = __hfma(value, value, var);
        // }

        // var = __hdiv(var, __int2half_rn(x.m_dim_z));
        // half variance = hrcp(hsqrt(__hadd(var, epf_half)));

        // for (int k = 0; k < x.m_dim_z; k++) {
        //     half value = x(i, j, k);
        //     half half_out = __hmul(value, variance) * __float2half(weight(0, 0, k));
            
        //     // output(i, j, k) = fp_out;
        //     output(i, j, k) = half_out;
        // }
    }
}

// __global__ void LlamaRMSNorm_cuda_kernel(const Matrix3D<half> x, const Matrix3D<float> weight, Matrix3D<half> output, float eps) {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     int j = blockIdx.y * blockDim.y + threadIdx.y;
//     int k = blockIdx.z * blockDim.z + threadIdx.z;

//     // index into shared memory (assuming it's size is at least blockDim.z)
//     int shared_index = threadIdx.z;
//     float shared_memory[11008];

//     if (i < x.m_dim_x && j < x.m_dim_y && k < x.m_dim_z) {
//         float value = __half2float(x(i, j, k));
//         shared_memory[shared_index] = value * value;  // square of the value
//     }

//     __syncthreads();  // make sure all threads have written to shared memory

//     // now perform reduction in shared memory
//     for (unsigned int s = blockDim.z / 2; s > 0; s >>= 1) {
//         if (threadIdx.z < s) {
//             shared_memory[shared_index] += shared_memory[shared_index + s];
//         }
//         __syncthreads();  // make sure all additions are done before next step
//     }

//     if (threadIdx.z == 0) {  // only thread 0 writes the result
//         float var = shared_memory[0] / static_cast<float>(x.m_dim_z);
//         float variance = 1.0 / sqrtf(var + eps);

//         // now go through z dimension again to do normalization
//         for (int kk = 0; kk < x.m_dim_z; kk++) {
//             float value = __half2float(x(i, j, kk));
//             float fp_out = (value * variance) * weight(0, 0, kk);
//             output(i, j, kk) = __float2half(fp_out);
//         }
//     }
// }

void LlamaRMSNorm_cuda::forward(const Matrix3D<half> &x, Matrix3D<half> &output) {
    dim3 block(32, 32);
    dim3 grid((x.m_dim_x + block.x - 1) / block.x, (x.m_dim_y + block.y - 1) / block.y);

    LlamaRMSNorm_cuda_kernel<<<grid, block>>>(x, weight, output, eps);
}
