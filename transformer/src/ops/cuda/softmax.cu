#include "hip/hip_runtime.h"
#include <cmath>
#include "operators.cuh"

__device__ static float atomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    
    return __int_as_float(old);
}

// __global__ void softmax_half(Matrix3D<float> input, Matrix3D<float> output) {
//     // Calculate indices i, j in the input array
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     int j = blockIdx.y * blockDim.y + threadIdx.y;

//     if (i < input.m_dim_x && j < input.m_dim_y) {
//         // // Shared memory for the maximum value and sum
//         // __shared__ float max_value;
//         // __shared__ float sum;

//         // if (threadIdx.x == 0 && threadIdx.y == 0) {
//         //     max_value = input(0, 0, 0);
//         //     sum = 0.0f;
//         // }
//         // __syncthreads();

//         float max_value = input.m_data[0];
//         float sum = 0.0f;
//         __syncthreads();

//         // Find the maximum value in the input array
//         for (int k = 0; k < input.m_dim_z; k++) {
//             float value = input(i, j, k);
//             atomicMax(&max_value, value);
//         }
//         __syncthreads();

//         // Compute the sum
//         for (int k = 0; k < input.m_dim_z; k++) {
//             float value = expf(input(i, j, k) - max_value);
//             atomicAdd(&sum, value);
//         }
//         __syncthreads();

//         // Compute the final softmax values
//         for (int k = 0; k < input.m_dim_z; k++) {
//             output(i, j, k) = expf(input(i, j, k) - max_value) / sum;
//         }
//     }
// }


__global__ void softmax_half(Matrix3D<float> input, Matrix3D<float> output) {
    // Calculate indices i, j in the input array
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < input.m_dim_x && j < input.m_dim_y) {
        float max_value = -INFINITY;
        float sum = 0;

        // Find the maximum value in the input array
        for (int k = 0; k < input.m_dim_z; k++) {
            float value = input(i, j, k);
            // atomicMax(&max_value, value);
            max_value = fmaxf(max_value, value);
        }
        // __syncthreads();

        // Compute the sum
        for (int k = 0; k < input.m_dim_z; k++) {
            float value = input(i, j, k);
            // atomicAdd(&sum, value);
            sum += __expf(value - max_value);
        }
        // __syncthreads();

        // Compute the final softmax values
        for (int k = 0; k < input.m_dim_z; k++) {
            float value = input(i, j, k);
            output(i, j, k) = __expf(value - max_value) / sum;
        }
    }
}
