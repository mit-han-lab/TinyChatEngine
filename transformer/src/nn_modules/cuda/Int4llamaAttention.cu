#include "hip/hip_runtime.h"
#include <string.h>
#include <cmath>
#include <cfloat>

#include "Int4llamaAttention.h"
#include "operators.h"
#include "utils.h"

static float16_t *attn_weights_arr = nullptr;
static float16_t *attn_output_half_arr = nullptr;
static float16_t *query_states_unshape_arr = nullptr;
static float16_t *attn_output_arr = nullptr;
static float16_t *attn_output_transpose_arr = nullptr;
static float16_t *key_states_unshape_arr = nullptr;
static float16_t *key_states_arr = nullptr;
static float16_t *value_states_unshape_arr = nullptr;
static float16_t *value_states_arr = nullptr;
static float16_t *query_states_arr = nullptr;
static float16_t *value_states_transpose_arr = nullptr;
static float16_t *key_states_arr_cache = nullptr;
static float16_t *value_states_arr_cache = nullptr;
static int *cache_num = nullptr;

// static float16_t* split_8_buffer;

void Int4llamaAttention::initialized_memory(const struct model_config config) {
    allocate_aligned_memory_gpu(attn_weights_arr, config.num_heads * config.max_sqlen * config.max_sqlen * sizeof(float16_t));
    allocate_aligned_memory_gpu(attn_output_half_arr, config.max_sqlen * config.embed_dim * sizeof(float16_t));
    allocate_aligned_memory_gpu(attn_output_arr, config.max_sqlen * config.embed_dim * sizeof(float16_t));
    allocate_aligned_memory_gpu(attn_output_transpose_arr, config.max_sqlen * config.embed_dim * sizeof(float16_t));
    allocate_aligned_memory_gpu(key_states_unshape_arr, config.max_sqlen * config.embed_dim * sizeof(float16_t));
    allocate_aligned_memory_gpu(key_states_arr, config.max_sqlen * config.embed_dim * sizeof(float16_t));
    allocate_aligned_memory_gpu(value_states_unshape_arr, config.max_sqlen * config.embed_dim * sizeof(float16_t));
    allocate_aligned_memory_gpu(value_states_arr, config.max_sqlen * config.embed_dim * sizeof(float16_t));
    allocate_aligned_memory_gpu(query_states_arr, config.max_sqlen * config.embed_dim * sizeof(float16_t));
    allocate_aligned_memory_gpu(value_states_transpose_arr, config.max_sqlen * config.embed_dim * sizeof(float16_t));
    allocate_aligned_memory_gpu(query_states_unshape_arr, config.max_sqlen * config.embed_dim * sizeof(float16_t));

    allocate_aligned_memory(cache_num, config.num_layers * sizeof(int));
    for (int i = 0; i < config.num_layers; i++) cache_num[i] = 0;

    allocate_aligned_memory_gpu(key_states_arr_cache, config.num_layers * 2 * config.max_sqlen * config.embed_dim * sizeof(float16_t));
    allocate_aligned_memory_gpu(value_states_arr_cache, config.num_layers * 2 * config.max_sqlen * config.embed_dim * sizeof(float16_t));

    // allocate_aligned_memory_gpu(split_8_buffer, config.max_sqlen * config.embed_dim * sizeof(float16_t) * 8);
    allocate_aligned_memory_gpu(split_8_buffer, config.max_sqlen * config.vocsize * sizeof(float16_t) * 8);
    // printf("config.max_sqlen, config.embed_dim: %d, %d\n", config.max_sqlen, config.embed_dim);
}

template <typename T>
__global__ void shape_cuda(Matrix3D<T> unshape, Matrix3D<T> shaped, int num_heads, int sqlen, int head_dim) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int k = threadIdx.z + blockIdx.z * blockDim.z;
    
    if (i < num_heads && j < sqlen && k < head_dim) {
        shaped(i, j, k) = unshape(0, j, i * head_dim + k);
    }
}

template <typename T>
__global__ void unshape_cuda(Matrix3D<T> shaped, Matrix3D<T> unshape, int num_heads, int sqlen, int head_dim) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int k = threadIdx.z + blockIdx.z * blockDim.z;
    
    if (i < num_heads && j < sqlen && k < head_dim) {
        unshape(0, j, i * head_dim + k) = shaped(i, j, k);
    }
}

Int4llamaAttention::Int4llamaAttention(std::string param_path, const struct model_config config) {
    allocate_aligned_memory_gpu(q_weight, (config.embed_dim * config.embed_dim * sizeof(int)) / 8);
    allocate_aligned_memory_gpu(k_weight, (config.embed_dim * config.embed_dim * sizeof(int)) / 8);
    allocate_aligned_memory_gpu(v_weight, (config.embed_dim * config.embed_dim * sizeof(int)) / 8);
    allocate_aligned_memory_gpu(o_weight, (config.embed_dim * config.embed_dim * sizeof(int)) / 8);

    this->q_proj = Linear_half_int4(Matrix3D<int>(q_weight, 1, config.embed_dim / 8, config.embed_dim),
                                  param_path + "/q_proj");
    this->k_proj = Linear_half_int4(Matrix3D<int>(k_weight, 1, config.embed_dim / 8, config.embed_dim),
                                  param_path + "/k_proj");
    this->v_proj = Linear_half_int4(Matrix3D<int>(v_weight, 1, config.embed_dim / 8, config.embed_dim),
                                  param_path + "/v_proj");
    this->o_proj = Linear_half_int4(Matrix3D<int>(o_weight, 1, config.embed_dim / 8, config.embed_dim),
                                  param_path + "/o_proj");

    allocate_aligned_memory_gpu(cos_buf, config.max_sqlen * (config.embed_dim / config.num_heads) * sizeof(half));
    allocate_aligned_memory_gpu(sin_buf, config.max_sqlen * (config.embed_dim / config.num_heads) * sizeof(half));
    Matrix3D<half> cos(cos_buf, 1, config.max_sqlen, (config.embed_dim / config.num_heads));
    Matrix3D<half> sin(sin_buf, 1, config.max_sqlen, (config.embed_dim / config.num_heads));

    this->rotary_pos_emb = RotaryPosEmb_cuda(cos, sin, param_path + "/rotary_emb");

    half qk_bmm_alpha;
    read_to_array_half((param_path + "/qk_bmm/alpha_half.bin").c_str(), &qk_bmm_alpha, 1);
    this->qk_bmm = BMM_F16T(qk_bmm_alpha);
    this->pv_bmm = BMM_F16T(__float2half(1.0f));

    this->embed_dim = config.embed_dim;
    this->num_heads = config.num_heads;
    assert(config.embed_dim % config.num_heads == 0);
    this->head_dim = config.embed_dim / config.num_heads;
    this->max_sqlen = config.max_sqlen;
}

template <typename T>
__global__ void transpose_1_2idx_cuda(Matrix3D<T> input, Matrix3D<T> output) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < input.m_dim_x && j < input.m_dim_y && k < input.m_dim_z) {
        output.m_data[i * output.m_dim_y * output.m_dim_z + k * output.m_dim_z + j] =
            input.m_data[i * input.m_dim_y * input.m_dim_z + j * input.m_dim_z + k];
    }
}

__global__ void transpose_1_2idx_float(Matrix3D<float> input, Matrix3D<float> output) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < input.m_dim_x && j < input.m_dim_y && k < input.m_dim_z) {
        output.m_data[i * output.m_dim_y * output.m_dim_z + k * output.m_dim_z + j] =
            input.m_data[i * input.m_dim_y * input.m_dim_z + j * input.m_dim_z + k];
    }
}

__global__ void transpose_1_2idx_half(Matrix3D<float16_t> input, Matrix3D<float16_t> output) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < input.m_dim_x && j < input.m_dim_y && k < input.m_dim_z) {
        output.m_data[i * output.m_dim_y * output.m_dim_z + k * output.m_dim_z + j] =
            input.m_data[i * input.m_dim_y * input.m_dim_z + j * input.m_dim_z + k];
    }
}

__global__ void check_inf_float(Matrix3D<float> a) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (i < a.length()) {
        if (isinf(a.m_data[i])) {
            a.m_data[i] = -FLT_MAX;
        }
    }
}

__global__ void check_inf_half(Matrix3D<float16_t> a) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (i < a.length()) {
        if (__hisinf(a.m_data[i]) == 1 || __hisinf(a.m_data[i]) == -1 || __hisnan(a.m_data[i])) {
            // a.m_data[i] = __float2half(-FLT_MAX);  // TODO: maybe could be optimized
            a.m_data[i] = __float2half(-65504.0f);  // TODO: maybe could be optimized
        }
    }
}

struct Int4llamaAttention_output Int4llamaAttention::forward(const struct Int4llamaAttention_input &input) {
    PROFILE_START(profile_name);

    // printf("aaaaaaaaaa\n");

    // hipEvent_t start_Attention, stop_Attention, start, stop;
    // hipEventCreate(&start_Attention);
    // hipEventCreate(&stop_Attention);
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // float milliseconds = 0;

    // hipEventRecord(start_Attention);

    struct Int4llamaAttention_output output;
    const int sqlen = input.hidden_states.m_dim_y, b = input.hidden_states.m_dim_x;
    assert(b == 1);

    // Query
    // hipEventRecord(start);
    // PROFILE_START(profile_name + "::q_proj.forward");
    Matrix3D<float16_t> query_states_unshape(query_states_unshape_arr, b, sqlen, embed_dim);
    this->q_proj.forward(input.hidden_states, query_states_unshape, split_8_buffer);
    // PROFILE_END(profile_name + "::q_proj.forward");
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("q_proj.forward: %.2f ms\n", milliseconds * this->num_heads);

    // hipEventRecord(start);
    // PROFILE_START(profile_name + "::shape_cuda_query");
    Matrix3D<float16_t> query_states(query_states_arr, this->num_heads, sqlen, this->head_dim);
    dim3 threadsPerBlock(8, 4, 32);
    dim3 numBlocks((this->num_heads + threadsPerBlock.x - 1) / threadsPerBlock.x,
                (sqlen + threadsPerBlock.y - 1) / threadsPerBlock.y,
                (this->head_dim + threadsPerBlock.z - 1) / threadsPerBlock.z);
    shape_cuda<<<numBlocks, threadsPerBlock>>>(query_states_unshape, query_states, this->num_heads, sqlen, this->head_dim);
    // PROFILE_END(profile_name + "::shape_cuda_query");
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("shape_cuda of q_proj: %.2f ms\n", milliseconds * this->num_heads);

    // hipEventRecord(start);
    // PROFILE_START(profile_name + "::set_cache_num");
    // printf("bbbbbbbbbb\n");
    float16_t *ret_value_states, *ret_key_states;
    if (cache_num[input.layer_idx] == 1) {
        ret_value_states = &value_states_arr_cache[(input.layer_idx * 2 + 1) * this->max_sqlen * this->embed_dim];
        ret_key_states = &key_states_arr_cache[(input.layer_idx * 2 + 1) * this->max_sqlen * this->embed_dim];
        cache_num[input.layer_idx] = 0;
    } else {
        ret_value_states = &value_states_arr_cache[input.layer_idx * 2 * this->max_sqlen * this->embed_dim];
        ret_key_states = &key_states_arr_cache[input.layer_idx * 2 * this->max_sqlen * this->embed_dim];
        cache_num[input.layer_idx] = 1;
    }
    // PROFILE_END(profile_name + "::set_cache_num");
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("set_cache_num: %.2f ms\n", milliseconds * this->num_heads);
    // printf("cccccccccc\n");

    // Key
    // hipEventRecord(start);
    // PROFILE_START(profile_name + "::k_proj.forward");
    Matrix3D<float16_t> key_states_unshape(key_states_unshape_arr, b, sqlen, embed_dim);
    this->k_proj.forward(input.hidden_states, key_states_unshape, split_8_buffer);
    // PROFILE_END(profile_name + "::k_proj.forward");
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("k_proj.forward: %.2f ms\n", milliseconds * this->num_heads);

    // hipEventRecord(start);
    // PROFILE_START(profile_name + "::shape_cuda_key");
    Matrix3D<float16_t> key_states(key_states_arr, this->num_heads, sqlen, this->head_dim);
    shape_cuda<<<numBlocks, threadsPerBlock>>>(key_states_unshape, key_states, this->num_heads, sqlen, this->head_dim);
    // PROFILE_END(profile_name + "::shape_cuda_key");
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("shape_cuda of k_proj: %.2f ms\n", milliseconds * this->num_heads);

    // Value
    // hipEventRecord(start);
    // PROFILE_START(profile_name + "::v_proj.forward");
    Matrix3D<float16_t> value_states_unshape(value_states_unshape_arr, b, sqlen, embed_dim);
    this->v_proj.forward(input.hidden_states, value_states_unshape, split_8_buffer);
    // PROFILE_END(profile_name + "::v_proj.forward");
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("v_proj.forward: %.2f ms\n", milliseconds * this->num_heads);

    // hipEventRecord(start);
    // PROFILE_START(profile_name + "::shape_cuda_value");
    Matrix3D<float16_t> value_states(value_states_arr, this->num_heads, sqlen, this->head_dim);
    shape_cuda<<<numBlocks, threadsPerBlock>>>(value_states_unshape, value_states, this->num_heads, sqlen, this->head_dim);
    // PROFILE_END(profile_name + "::shape_cuda_value");
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("shape_cuda of v_proj: %.2f ms\n", milliseconds * this->num_heads);

    int start_idx = 0;
    if (input.has_past_key_value) start_idx = input.past_key.m_dim_y;
    // printf("dddddddddd\n");

    // hipEventRecord(start);
    // PROFILE_START(profile_name + "::RotaryPosEmb_cuda_forward");
    dim3 grid(num_heads, 1, 1);
    dim3 block(sqlen, 1, 1);
    RotaryPosEmb_cuda_forward<<<grid, block>>>(query_states, key_states, this->rotary_pos_emb.cos, this->rotary_pos_emb.sin, start_idx, sqlen);
    // PROFILE_END(profile_name + "::RotaryPosEmb_cuda_forward");
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("RotaryPosEmb_cuda_forward: %.2f ms\n", milliseconds * this->num_heads);

    // hipEventRecord(start);
    // PROFILE_START(profile_name + "::cat_past_keys_values");
    int tgz = sqlen;
    if (input.has_past_key_value) {
        assert(input.past_key.m_dim_z == this->head_dim);
        tgz += input.past_key.m_dim_y;
        float16_t *val_ptr = ret_value_states, *key_ptr = ret_key_states;
        int past_block = input.past_key.m_dim_y * input.past_key.m_dim_z;
        int sq_block = sqlen * this->head_dim;
        for (int i = 0; i < input.past_key.m_dim_x; i++) {
            hipMemcpyAsync(val_ptr, &input.past_value.m_data[past_block * i], past_block * sizeof(float16_t), hipMemcpyDeviceToDevice);
            val_ptr += past_block;
            hipMemcpyAsync(val_ptr, &value_states.m_data[sq_block * i], sq_block * sizeof(float16_t), hipMemcpyDeviceToDevice);
            val_ptr += sq_block;
            hipMemcpyAsync(key_ptr, &input.past_key.m_data[past_block * i], past_block * sizeof(float16_t), hipMemcpyDeviceToDevice);
            key_ptr += past_block;
            hipMemcpyAsync(key_ptr, &key_states.m_data[sq_block * i], sq_block * sizeof(float16_t), hipMemcpyDeviceToDevice);
            key_ptr += sq_block;
        }
    } else {
        hipMemcpyAsync(ret_value_states, value_states_arr, (this->num_heads * tgz * this->head_dim) * sizeof(float16_t), hipMemcpyDeviceToDevice);
        hipMemcpyAsync(ret_key_states, key_states_arr, (this->num_heads * tgz * this->head_dim) * sizeof(float16_t), hipMemcpyDeviceToDevice);
    }

    // printf("eeeeeeeeee\n");

    Matrix3D<float16_t> final_value_states(ret_value_states, this->num_heads, tgz, this->head_dim);
    Matrix3D<float16_t> final_key_states(ret_key_states, this->num_heads, tgz, this->head_dim);
    // PROFILE_END(profile_name + "::cat_past_keys_values");
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("cat_past_keys_values: %.2f ms\n", milliseconds * this->num_heads);

    // hipEventRecord(start);
    // PROFILE_START(profile_name + "::qk_bmm");
    Matrix3D<float16_t> attn_weights(attn_weights_arr, this->num_heads, sqlen, tgz);
    this->qk_bmm.forward(query_states, final_key_states, attn_weights);
    // PROFILE_END(profile_name + "::qk_bmm");
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("qk_bmm.forward: %.2f ms\n", milliseconds * this->num_heads);

    // hipEventRecord(start);
    // PROFILE_START(profile_name + "::batch_Add_cuda");
    dim3 threadsPerBlock2(16, 4, 16);
    dim3 numBlocks2((this->num_heads + threadsPerBlock2.x - 1) / threadsPerBlock2.x,
                (sqlen + threadsPerBlock2.y - 1) / threadsPerBlock2.y,
                (tgz + threadsPerBlock2.z - 1) / threadsPerBlock2.z);
    // if (attn_weights.m_dim_z % 2 == 0) {
    //     batch_Add_cuda_half2<<<numBlocks2, threadsPerBlock2>>>(attn_weights, input.attention_mask, attn_weights);
    // } else {
    //     batch_Add_cuda<<<numBlocks2, threadsPerBlock2>>>(attn_weights, input.attention_mask, attn_weights);
    // }
    batch_Add_cuda<<<numBlocks2, threadsPerBlock2>>>(attn_weights, input.attention_mask, attn_weights);
    // PROFILE_END(profile_name + "::batch_Add_cuda");
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("batch_Add_cuda: %.2f ms\n", milliseconds * this->num_heads);
    // printf("fffffffff\n");

    // hipEventRecord(start);
    // PROFILE_START(profile_name + "::check_inf_half");
    int threadsPerBlock_1D = 1024;
    int blocksPerGrid_1D =(attn_weights.length() + threadsPerBlock_1D - 1) / threadsPerBlock_1D;
    check_inf_half<<<blocksPerGrid_1D, threadsPerBlock_1D>>>(attn_weights);
    // PROFILE_END(profile_name + "::check_inf_half");
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("check_inf_half: %.2f ms\n", milliseconds * this->num_heads);

    // hipEventRecord(start);
    // PROFILE_START(profile_name + "::softmax_cuda");
    Matrix3D<float16_t> attn_probs(attn_weights_arr, this->num_heads, sqlen, tgz);
    dim3 threadsPerBlock3(64, 16);
    dim3 numBlocks3((this->num_heads + threadsPerBlock3.x - 1) / threadsPerBlock3.x, (sqlen + threadsPerBlock3.y - 1) / threadsPerBlock3.y);
    softmax_cuda<<<numBlocks3, threadsPerBlock3>>>(attn_weights, attn_probs);
    // PROFILE_END(profile_name + "::softmax_cuda");
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("softmax_cuda: %.2f ms\n", milliseconds * this->num_heads);
    // printf("gggggggggg\n");


    /* Legacy Implementation of PV_BMM*/
    // hipEventRecord(start);
    // PROFILE_START(profile_name + "::transpose_1_2idx_cuda");
    Matrix3D<float16_t> value_states_transpose(value_states_transpose_arr, this->num_heads, this->head_dim, tgz);
    dim3 threadsPerBlock4(8, 4, 32);
    dim3 numBlocks4((this->num_heads + threadsPerBlock4.x - 1) / threadsPerBlock4.x,
                (tgz + threadsPerBlock4.y - 1) / threadsPerBlock4.y,
                (this->head_dim + threadsPerBlock4.z - 1) / threadsPerBlock4.z);
    transpose_1_2idx_cuda<<<numBlocks4, threadsPerBlock4>>>(final_value_states, value_states_transpose);
    // PROFILE_END(profile_name + "::transpose_1_2idx_cuda");
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("transpose_1_2idx_cuda: %.2f ms\n", milliseconds * this->num_heads);

    // hipEventRecord(start);
    // PROFILE_START(profile_name + "::pv_bmm");
    Matrix3D<float16_t> attn_output(attn_output_arr, this->num_heads, sqlen, this->head_dim);
    this->pv_bmm.forward(attn_probs, value_states_transpose, attn_output);
    // PROFILE_END(profile_name + "::pv_bmm");
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("pv_bmm.forward: %.2f ms\n", milliseconds * this->num_heads);

    // /* Optimized Implementation of PV_BMM*/
    // // hipEventRecord(start);
    // // PROFILE_START(profile_name + "::pv_bmm");
    // Matrix3D<float16_t> attn_output(attn_output_arr, this->num_heads, sqlen, this->head_dim);
    // this->pv_bmm.forward_weight_untransposed(attn_probs, final_value_states, attn_output);
    // // PROFILE_END(profile_name + "::pv_bmm");
    // // hipEventRecord(stop);
    // // hipEventSynchronize(stop);
    // // hipEventElapsedTime(&milliseconds, start, stop);
    // // printf("pv_bmm.forward: %.2f ms\n", milliseconds * this->num_heads);
    // printf("hhhhhhhhhh\n");


    // hipEventRecord(start);
    // PROFILE_START(profile_name + "::unshape_cuda");
    Matrix3D<float16_t> attn_output_transpose(attn_output_transpose_arr, 1, sqlen, this->num_heads * this->head_dim);
    unshape_cuda<<<numBlocks, threadsPerBlock>>>(attn_output, attn_output_transpose, this->num_heads, sqlen, this->head_dim);
    // PROFILE_END(profile_name + "::unshape_cuda");
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("unshape_cuda: %.2f ms\n", milliseconds * this->num_heads);

    // hipEventRecord(start);
    // PROFILE_START(profile_name + "::o_proj.forward");
    Matrix3D<float16_t> attn_output_half(attn_output_half_arr, 1, sqlen, this->num_heads * this->head_dim);
    this->o_proj.forward(attn_output_transpose, attn_output_half, split_8_buffer);
    // PROFILE_END(profile_name + "::o_proj.forward");
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("o_proj.forward: %.2f ms\n", milliseconds * this->num_heads);

    // output assignment
    output.attn_output = attn_output_half;
    output.past_key_value = {final_key_states, final_value_states};
    
    // hipEventRecord(stop_Attention);
    // hipEventSynchronize(stop_Attention);
    // hipEventElapsedTime(&milliseconds, start_Attention, stop_Attention);
    // printf("The elapsed time of Int4llamaAttention is %.2f ms\n", milliseconds * this->num_heads);

    PROFILE_END(profile_name);

    // printf("iiiiiiiiii\n");

    // hipEventDestroy(start_Attention);
    // hipEventDestroy(stop_Attention);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);

    return output;
}

void Int4llamaAttention::free_cuda_memory() {
    free_aligned_memory_gpu(attn_weights_arr);
    free_aligned_memory_gpu(attn_output_half_arr);
    free_aligned_memory_gpu(query_states_unshape_arr);
    free_aligned_memory_gpu(attn_output_arr);
    free_aligned_memory_gpu(attn_output_transpose_arr);
    free_aligned_memory_gpu(key_states_unshape_arr);
    free_aligned_memory_gpu(key_states_arr);
    free_aligned_memory_gpu(value_states_unshape_arr);
    free_aligned_memory_gpu(value_states_arr);
    free_aligned_memory_gpu(query_states_arr);
    free_aligned_memory_gpu(value_states_transpose_arr);
    free_aligned_memory_gpu(key_states_arr_cache);
    free_aligned_memory_gpu(value_states_arr_cache);
    free_aligned_memory_gpu(cos_buf);
    free_aligned_memory_gpu(sin_buf);
    free_aligned_memory_gpu(q_weight);
    free_aligned_memory_gpu(k_weight);
    free_aligned_memory_gpu(v_weight);
    free_aligned_memory_gpu(o_weight);

    if(cache_num) {
        free(cache_num);
        cache_num = nullptr;
    }
}
