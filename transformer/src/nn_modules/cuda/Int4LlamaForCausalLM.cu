#include <chrono>

#include "Int4LlamaForCausalLM.h"
#include "operators.h"
#include "utils.h"

Int4LlamaForCausalLM::Int4LlamaForCausalLM(std::string param_path, const struct model_config config) {
    allocate_aligned_memory_gpu(logits_output_half, config.max_sqlen * config.vocsize * sizeof(float16_t));
    allocate_aligned_memory_gpu(logits_output, config.max_sqlen * config.vocsize * sizeof(float));
    allocate_aligned_memory_gpu(lm_head_weight, (config.embed_dim * config.vocsize * sizeof(int)) / 8);

    allocate_aligned_memory_gpu(split_8_buffer, config.max_sqlen * config.vocsize * sizeof(float16_t) * 8);

    this->decoder = Int4llamaDecoder(param_path + "/decoder", config);
    this->lm_head = Linear_half_int4(Matrix3D<int>(lm_head_weight, 1, config.vocsize / 8, config.embed_dim),
                                   param_path + "/lm_head");
}

struct Int4LlamaForCausalLM_output Int4LlamaForCausalLM::forward(const struct Int4LlamaForCausalLM_input &input) {
    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // float milliseconds = 0;

    PROFILE_START(profile_name);
    int sqlen = input.input_ids.m_dim_z;

    struct Int4llamaDecoder_output decoder_output;

    // hipEventRecord(start);

    if (input.has_past_keys_values) {
        struct Int4llamaDecoder_input decoder_input = {input.input_ids, input.past_keys, input.past_values};
        decoder_output = this->decoder.forward(decoder_input);

    } else {
        struct Int4llamaDecoder_input decoder_input = {input.input_ids};
        decoder_output = this->decoder.forward(decoder_input);
    }

    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("The elapsed time of Int4llamaDecoder is %.2f ms\n", milliseconds);


    // hipEventRecord(start);

    Matrix3D<float16_t> logits_half(logits_output_half, 1, sqlen, this->decoder.voc_size);
    this->lm_head.forward(decoder_output.last_hidden_state, logits_half, split_8_buffer);

    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("The elapsed time of lm_head.forward is %.2f ms\n", milliseconds);


    // hipEventRecord(start);

    Matrix3D<float> logits(logits_output, 1, sqlen, this->decoder.voc_size);
    int threadsPerBlock_1D = 1024;
    int blocksPerGrid =(sqlen * this->decoder.voc_size + threadsPerBlock_1D - 1) / threadsPerBlock_1D;
    half2float<<<blocksPerGrid, threadsPerBlock_1D>>>(logits_output_half, logits_output, sqlen * this->decoder.voc_size);

    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("The elapsed time of half2float is %.2f ms\n", milliseconds);


    // hipEventRecord(start);

    hipDeviceSynchronize();

    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("The elapsed time of hipDeviceSynchronize is %.2f ms\n", milliseconds);

    // hipEventDestroy(start);
    // hipEventDestroy(stop);

    struct Int4LlamaForCausalLM_output LMoutput = {logits, decoder_output.past_keys, decoder_output.past_values};
    PROFILE_END(profile_name);

    return LMoutput;
}
