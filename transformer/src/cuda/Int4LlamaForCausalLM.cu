#include "Int4LlamaForCausalLM.h"

#include <chrono>

#include "operators.h"
#include "utils.h"

Int4LlamaForCausalLM::Int4LlamaForCausalLM(std::string param_path, const struct model_config config) {
    allocate_aligned_memory_gpu(logits_output, config.max_sqlen * config.vocsize * sizeof(float));
    allocate_aligned_memory_gpu(lm_head_weight, (config.embed_dim * config.vocsize * sizeof(int)) / 8);
    //allocate_aligned_memory_gpu(lm_head_weight, config.embed_dim * config.vocsize * sizeof(float));

    this->decoder = Int4llamaDecoder(param_path + "/decoder", config);
    this->lm_head = Linear_half_int4_ref(Matrix3D<int>(lm_head_weight, 1, config.vocsize / 8, config.embed_dim),
                                   param_path + "/lm_head");
    // this->lm_head =
    //     Linear_FP(Matrix3D<float>(lm_head_weight, 1, config.vocsize, config.embed_dim), param_path + "/lm_head.bin");
}

struct Int4LlamaForCausalLM_output Int4LlamaForCausalLM::forward(const struct Int4LlamaForCausalLM_input &input) {
    PROFILE_START(profile_name);
    int sqlen = input.input_ids.m_dim_z;

    struct Int4llamaDecoder_output decoder_output;

    if (input.has_past_keys_values) {
        struct Int4llamaDecoder_input decoder_input = {input.input_ids, input.past_keys, input.past_values};
        decoder_output = this->decoder.forward(decoder_input);

    } else {
        struct Int4llamaDecoder_input decoder_input = {input.input_ids};
        decoder_output = this->decoder.forward(decoder_input);
    }

    Matrix3D<float> logits(logits_output, 1, sqlen, this->decoder.voc_size);
    this->lm_head.forward(decoder_output.last_hidden_state, logits);
    hipDeviceSynchronize();

    struct Int4LlamaForCausalLM_output LMoutput = {logits, decoder_output.past_keys, decoder_output.past_values};
    PROFILE_END(profile_name);

    // hipFree(lm_head_weight);

    return LMoutput;
}