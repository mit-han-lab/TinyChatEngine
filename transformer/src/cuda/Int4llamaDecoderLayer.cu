#include "hip/hip_runtime.h"
#include "Int4llamaDecoderLayer.h"

#include "utils.h"

// Shared memory space across all layers
static float *hidden_states_float_arr;
static float *final_layer_norm_arr;
static float *gate_proj_arr;
static float *up_proj_arr;
static float *down_proj_arr;
static float *temp;
static float *hidden_states_arr;

template <typename T>
static void add(Matrix3D<T> a, Matrix3D<T> b, Matrix3D<T> c) {
    PROFILE_START("Int4llamaDecoderLayer::add");
    assert(c.length() == a.length() && a.length() == b.length());

    for (int i = 0; i < a.length(); i++) {
        c.m_data[i] = a.m_data[i] + b.m_data[i];
    }
    PROFILE_END("Int4llamaDecoderLayer::add");
}

__global__ void add_half(Matrix3D<float> a, Matrix3D<float> b, Matrix3D<float> c) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (i < a.length()) {
        c.m_data[i] = a.m_data[i] + b.m_data[i];
    }
}

static void SiLuMul(Matrix3D<float> a, Matrix3D<float> b) {
    PROFILE_START("Int4llamaDecoderLayer::MulSiLu");
    for (int i = 0; i < a.length(); i++) {
        float v = a.m_data[i];
        float silu_v = v * (1.0 / (1.0 + exp(-1 * v)));
        a.m_data[i] = silu_v * b.m_data[i];
    }
    PROFILE_END("Int4llamaDecoderLayer::MulSiLu");
}

__global__ void SiLuMul_half(Matrix3D<float> a, Matrix3D<float> b) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < a.length()) {
        float v = a.m_data[i];
        float silu_v = v * (1.0 / (1.0 + __expf(-1.0 * v)));
        a.m_data[i] = silu_v * b.m_data[i];
    }
}


Int4llamaDecoderLayer::Int4llamaDecoderLayer(std::string param_path, const struct model_config config, int layer_idx) {
    if (layer_idx == 0) {
        allocate_aligned_memory_gpu(hidden_states_float_arr, config.max_sqlen * config.embed_dim * sizeof(float));
        allocate_aligned_memory_gpu(final_layer_norm_arr, config.max_sqlen * config.embed_dim * sizeof(float));
        allocate_aligned_memory_gpu(gate_proj_arr, config.max_sqlen * config.hidden_dim * sizeof(float));
        allocate_aligned_memory_gpu(up_proj_arr, config.max_sqlen * config.hidden_dim * sizeof(float));
        allocate_aligned_memory_gpu(down_proj_arr, config.max_sqlen * config.embed_dim * sizeof(float));
        allocate_aligned_memory_gpu(hidden_states_arr, config.max_sqlen * config.embed_dim * sizeof(float));
        Int4llamaAttention::initialized_memory(config);
    }

    float *input_layernorm_weight_ptr;
    allocate_aligned_memory_gpu(input_layernorm_weight_ptr, config.embed_dim * sizeof(float));
    Matrix3D<float> input_layernorm_weight(input_layernorm_weight_ptr, 1, 1, config.embed_dim);
    input_layernorm_weight.load((param_path + "/input_layernorm/weight.bin").c_str());
    //// Original code
    // this->input_layernorm = LlamaRMSNorm(input_layernorm_weight);
    //// CUDA 1
    this->input_layernorm = LlamaRMSNorm_half(input_layernorm_weight);

    float *post_attention_layernorm_ptr;
    allocate_aligned_memory_gpu(post_attention_layernorm_ptr, config.embed_dim * sizeof(float));
    Matrix3D<float> post_attention_layernorm_weight(post_attention_layernorm_ptr, 1, 1, config.embed_dim);
    post_attention_layernorm_weight.load((param_path + "/post_attention_layernorm/weight.bin").c_str());
    //// Original code
    // this->post_attention_layernorm = LlamaRMSNorm(post_attention_layernorm_weight);
    //// CUDA 1
    this->post_attention_layernorm = LlamaRMSNorm_half(post_attention_layernorm_weight);

    this->embed_dim = config.embed_dim;
    this->num_attention_heads = config.num_heads;
    this->hidden_dim = config.hidden_dim;
    this->layer_idx = layer_idx;

    this->attn = Int4llamaAttention(param_path + "/self_attn", config);

    allocate_aligned_memory_gpu(gate_proj_weight, (config.embed_dim * config.hidden_dim * sizeof(int)) / 8);
    allocate_aligned_memory_gpu(down_proj_weight, (config.hidden_dim * config.embed_dim * sizeof(int)) / 8);
    allocate_aligned_memory_gpu(up_proj_weight, (config.embed_dim * config.hidden_dim * sizeof(int)) / 8);
    this->gate_proj = Linear_half_int4_ref(Matrix3D<int>(gate_proj_weight, 1, config.hidden_dim / 8, config.embed_dim),
                                     (param_path + "/gate_proj"));
    this->down_proj = Linear_half_int4_ref(Matrix3D<int>(down_proj_weight, 1, config.embed_dim / 8, config.hidden_dim),
                                     (param_path + "/down_proj"));
    this->up_proj = Linear_half_int4_ref(Matrix3D<int>(up_proj_weight, 1, config.hidden_dim / 8, config.embed_dim),
                                   (param_path + "/up_proj"));
}


struct Int4llamaDecoderLayer_output Int4llamaDecoderLayer::forward(const struct Int4llamaDecoderLayer_input &input) {
    PROFILE_START(profile_name);
    Matrix3D<float> hidden_states(hidden_states_arr, input.hidden_states.m_dim_x, input.hidden_states.m_dim_y,
                                  input.hidden_states.m_dim_z);
    this->input_layernorm.forward(input.hidden_states, hidden_states);
    // hipDeviceSynchronize();

    struct Int4llamaAttention_input attn_param(hidden_states, input.attention_mask, input.past_key, input.past_value,
                                               input.has_past_key_value, this->layer_idx);
    struct Int4llamaAttention_output attn_output = this->attn.forward(attn_param);

    Matrix3D<float> residual_add(hidden_states_float_arr, input.hidden_states.m_dim_x, input.hidden_states.m_dim_y,
                                 input.hidden_states.m_dim_z);
    //// Original code
    // add(input.hidden_states, attn_output.attn_output, residual_add);
    //// CUDA 1
    int threadsPerBlock = 1024;
    int blocksPerGrid =(input.hidden_states.length() + threadsPerBlock - 1) / threadsPerBlock;
    add_half<<<blocksPerGrid, threadsPerBlock>>>(input.hidden_states, attn_output.attn_output, residual_add);
    // hipDeviceSynchronize();
    // hipError_t err;
    // err = hipGetLastError();
    // if (err != hipSuccess) {
    // printf("Error launching add_half 1 kernel: %s\n", hipGetErrorString(err));
    // }

    Matrix3D<float> post_attention_layernorm(final_layer_norm_arr, input.hidden_states.m_dim_x,
                                             input.hidden_states.m_dim_y, input.hidden_states.m_dim_z);
    this->post_attention_layernorm.forward(residual_add, post_attention_layernorm);
    // hipDeviceSynchronize();
    // err = hipGetLastError();
    // if (err != hipSuccess) {
    // printf("Error launching post_attention_layernorm kernel: %s\n", hipGetErrorString(err));
    // }

    Matrix3D<float> gate_proj(gate_proj_arr, input.hidden_states.m_dim_x, input.hidden_states.m_dim_y,
                              this->hidden_dim);
    this->gate_proj.forward(post_attention_layernorm, gate_proj);

    Matrix3D<float> up_proj(up_proj_arr, input.hidden_states.m_dim_x, input.hidden_states.m_dim_y, this->hidden_dim);
    this->up_proj.forward(post_attention_layernorm, up_proj);

    //// Original code
    // SiLuMul(gate_proj, up_proj);
    //// CUDA 1
    // int threadsPerBlock = 256;
    int blocksPerGrid2 =(gate_proj.length() + threadsPerBlock - 1) / threadsPerBlock;
    SiLuMul_half<<<blocksPerGrid2, threadsPerBlock>>>(gate_proj, up_proj);
    // hipDeviceSynchronize();

    Matrix3D<float> down_proj(down_proj_arr, input.hidden_states.m_dim_x, input.hidden_states.m_dim_y, this->embed_dim);
    this->down_proj.forward(gate_proj, down_proj);

    //// Original code
    // add(residual_add, down_proj, residual_add);
    //// CUDA 1
    int blocksPerGrid3 =(residual_add.length() + threadsPerBlock - 1) / threadsPerBlock;
    add_half<<<blocksPerGrid3, threadsPerBlock>>>(residual_add, down_proj, residual_add);
    // hipDeviceSynchronize();

    struct Int4llamaDecoderLayer_output output(residual_add, attn_output.attn_probs_reshaped,
                                               attn_output.past_key_value);
    PROFILE_END(profile_name);

    // hipFree(gate_proj_weight);
    // hipFree(down_proj_weight);
    // hipFree(up_proj_weight);

    return output;
}
