#include "hip/hip_runtime.h"
#include "utils.h"

#include <stdio.h>

#include <cassert>
#include <cerrno>  // for errno
#include <cmath>
#include <cstdlib>
#include <cstring>  // for strerror
#include <iostream>

#define CHECK_CUDA(call) \
    do { \
        hipError_t err = call; \
        if(err != hipSuccess) { \
            throw std::runtime_error(std::string("CUDA error calling \"") + #call + "\", code is " + std::to_string(err)); \
        } \
    } while(0)

bool check_two_equal_cpu_gpu(half_float::half* array, half* array2, int size, float error) {
    float sq_diff = 0;
    float max_sqdiff = 0;
    struct max_error_info error_info;

    for (int i = 0; i < size; i++) {
        float diff = static_cast<float>(array[i]) - __half2float(array2[i]);
        // printf("diff: %f\n, array[i]: %f\n, array2[i]: %f\n", diff, static_cast<float>(array[i]), __half2float(array2[i]));

        sq_diff += diff * diff;
        if (diff * diff > max_sqdiff) {
            max_sqdiff = diff * diff;
            error_info.idx = i;
            error_info.a1 = static_cast<float>(array[i]);
            error_info.a2 = __half2float(array2[i]);
        }
    }
    if ((sq_diff / size) > error) {
        std::cout << "MSE:" << sq_diff / size << ", MAX SQ diff:" << max_sqdiff;
        std::cout << "@:" << error_info.idx << ",a1:" << error_info.a1 << ",a2:" << error_info.a2 << std::endl;
        return false;
    }
    return true;
}

bool check_two_equal_float_half(float* array, half* array2, int size) {
    float sq_diff = 0;
    float max_sqdiff = 0;
    for (int i = 0; i < size; i++) {
        float diff = (float)array[i] - __half2float(array2[i]);
        sq_diff += diff * diff;
        if (diff * diff > max_sqdiff) max_sqdiff = diff * diff;
        if (sqrt(max_sqdiff) > MAX_SQ_ERROR_MAX) {
            std::cout << "i:" << i << ",max_sqdiff:" << sqrt(max_sqdiff) << ", array[i]:";
            std::cout << static_cast<float>(array[i]) << ", array2[i]:" << __half2float(array2[i]) << std::endl;
            return false;
        }
    }
    if ((sq_diff / size) > ERROR_MAX) {
        std::cout << "MSE:" << sq_diff / size << ", MAX SQ diff:" << max_sqdiff << std::endl;
        return false;
    }
    return true;
}

template <typename T>
void allocate_aligned_memory_gpu(T*& ptr, size_t size) {
    // Allocate unified memory
    CHECK_CUDA(hipMallocManaged((void**)&ptr, size));
}

__global__ void float2half(float* floatArray, half* halfArray, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) {
        halfArray[index] = __float2half(floatArray[index]);
    }
}

__global__ void half2float(half* halfArray, float* floatArray, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) {
        floatArray[index] = __half2float(halfArray[index]);
    }
}

__global__ void half2float_merge_k_iters(half *halfArray, float *floatArray, int N, int split_k_iters) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < N) {
        float sum = 0;
        for (int j = 0; j < split_k_iters; j++) {
            sum += __half2float(halfArray[index + j * N]);
        }
        floatArray[index] = sum;
    }
}

// Explicitly instantiate the generic template function for other types (if needed)
template void allocate_aligned_memory_gpu(float*& ptr, size_t size);
template void allocate_aligned_memory_gpu(int*& ptr, size_t size);
template void allocate_aligned_memory_gpu(int8_t*& ptr, size_t size);
template void allocate_aligned_memory_gpu(uint8_t*& ptr, size_t size);
template void allocate_aligned_memory_gpu(half*& ptr, size_t size);
template void allocate_aligned_memory_gpu(half_float::half*& ptr, size_t size);
