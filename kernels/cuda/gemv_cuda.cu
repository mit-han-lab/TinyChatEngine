#include "hip/hip_runtime.h"

// Adapted from https://github.com/mit-han-lab/llm-awq
/*

@article{lin2023awq,
  title={AWQ: Activation-aware Weight Quantization for LLM Compression and Acceleration},
  author={Lin, Ji and Tang, Jiaming and Tang, Haotian and Yang, Shang and Dang, Xingyu and Han, Song},
  journal={arXiv},
  year={2023}
}

*/

#include <assert.h>
#include <stdlib.h>
#include <cstdlib>
#include <iostream>
#include <stdio.h>

#include "../matmul.h"
#include "ops/linear.h"

// #include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#define VECTORIZE_FACTOR 8
#define Q_VECTORIZE_FACTOR 8
#define PACK_FACTOR 8
#define WARP_SIZE 32


// Reduce sum within the warp using the tree reduction algorithm.
__device__ __forceinline__ float warp_reduce_sum(float sum) {
  #pragma unroll
  for(int i = 4; i >= 0; i--){
    sum += __shfl_down_sync(0xffffffff, sum, 1<<i);
  }
  /*
  // Equivalent to the following tree reduction implementation:
  sum += __shfl_down_sync(0xffffffff, sum, 16);
  sum += __shfl_down_sync(0xffffffff, sum, 8);
  sum += __shfl_down_sync(0xffffffff, sum, 4);
  sum += __shfl_down_sync(0xffffffff, sum, 2);
  sum += __shfl_down_sync(0xffffffff, sum, 1);
  */
  return sum;
}

__device__ __forceinline__ int make_divisible(int c, int divisor){
  return (c + divisor - 1) / divisor;
}


/*
Computes GEMV (group_size = 64).

Args:
  inputs: vector of shape [batch_size, IC];
  weight: matrix of shape [OC, IC / 8];
  output: vector of shape [OC];
  zeros: matrix of shape [OC, IC / group_size / 8];
  scaling_factors: matrix of shape [OC, IC / group_size];

Notes:
  One cannot infer group_size from the shape of scaling factors.
  the second dimension is rounded up to a multiple of PACK_FACTOR.
*/
__global__ void gemv_kernel_g64(
  const float4* _inputs, const uint32_t* weight, const uint32_t* zeros, const half* scaling_factors, half* _outputs, 
  const int IC, const int OC){
    const int group_size = 64;
    float psum = 0;
    const int batch_idx = blockIdx.z;
    const int oc_idx = blockIdx.y * blockDim.y + threadIdx.y; 
    const float4* inputs = _inputs + batch_idx * IC / PACK_FACTOR;
    half* outputs = _outputs + batch_idx * OC;
    // This is essentially zeros_w.
    const int num_groups_packed = make_divisible(make_divisible(IC / group_size, PACK_FACTOR), 2) * 2;
    const int weight_w = IC / PACK_FACTOR;
    // TODO (Haotian): zeros_w is incorrect, after fixing we got misaligned address
    const int zeros_w = make_divisible(make_divisible(IC / group_size, PACK_FACTOR), 2) * 2;
    // consistent with input shape
    const int sf_w = make_divisible(make_divisible(IC / group_size, PACK_FACTOR), 2) * 2 * PACK_FACTOR;
    // if(blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) printf("%d %d %d %d %d\n", IC, group_size, PACK_FACTOR, zeros_w, sf_w);
    // tile size: 4 OC x 1024 IC per iter
    for(int packed_group_idx = 0; packed_group_idx < num_groups_packed / 2; packed_group_idx++){
      // 1024 numbers in one iteration across warp. Need 1024 / group_size zeros.
      uint64_t packed_zeros = *reinterpret_cast<const uint64_t*>(zeros + oc_idx * zeros_w + packed_group_idx * 2);
      uint32_t packed_weights[4];
      // use float4 to load weights, each thread load 32 int4 numbers (1 x float4)
      *((float4*)(packed_weights)) = *((float4*)(weight + oc_idx * weight_w + packed_group_idx * (WARP_SIZE * 4) + threadIdx.x * 4));
      // load scaling factors
      // g64: two threads -> 64 numbers -> 1 group; 1 warp = 16 groups.
      float scaling_factor = __half2float(scaling_factors[oc_idx * sf_w + packed_group_idx * 16 + (threadIdx.x / 2)]);
      float current_zeros = (float)((packed_zeros >> (threadIdx.x / 2 * 4)) & 0xF);
      int inputs_ptr_delta = packed_group_idx * WARP_SIZE * 4 + threadIdx.x * 4; 
      const float4* inputs_ptr = inputs + inputs_ptr_delta;
      // multiply 32 weights with 32 inputs
      #pragma unroll
      for (int ic_0 = 0; ic_0 < 4; ic_0++){
        // iterate over different uint32_t packed_weights in this loop
        uint32_t current_packed_weight = packed_weights[ic_0];
        half packed_inputs[PACK_FACTOR];
        // each thread load 8 inputs, starting index is packed_group_idx * 128 * 8 (because each iter loads 128*8)
        if (inputs_ptr_delta + ic_0 < IC / PACK_FACTOR) {
          *((float4*)packed_inputs) = *(inputs_ptr + ic_0);
          #pragma unroll
          for (int ic_1 = 0; ic_1 < PACK_FACTOR; ic_1++){
            // iterate over 8 numbers packed within each uint32_t number
            float current_single_weight_fp = (float)(current_packed_weight & 0xF);
            float dequantized_weight = scaling_factor * (current_single_weight_fp - current_zeros);
            //if(blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0 && ic_0 == 0 && ic_1 == 0 && packed_group_idx == 0) printf("%f %f %f %f %X %X\n", dequantized_weight, current_single_weight_fp, scaling_factor, current_zeros, current_packed_weight, packed_zeros);
            psum += dequantized_weight * __half2float(packed_inputs[ic_1]);
            current_packed_weight = current_packed_weight >> 4;
          }
        }
      }
    }
    psum = warp_reduce_sum(psum);
    if (threadIdx.x == 0) {
     outputs[oc_idx] = __float2half(psum); 
    }
}


/*
Computes GEMV (group_size = 128).

Args:
  inputs: vector of shape [batch_size, IC];
  weight: matrix of shape [OC, IC / 8];
  output: vector of shape [OC];
  zeros: matrix of shape [OC, IC / group_size / 8];
  scaling_factors: matrix of shape [OC, IC / group_size];

Notes:
  One cannot infer group_size from the shape of scaling factors.
  the second dimension is rounded up to a multiple of PACK_FACTOR.
*/
__global__ void gemv_kernel_g128(
  const float4* _inputs, const uint32_t* weight, const uint32_t* zeros, const half* scaling_factors, half* _outputs, 
  const int IC, const int OC){
    const int group_size = 128;
    float psum = 0;
    const int batch_idx = blockIdx.z;
    const int oc_idx = blockIdx.y * blockDim.y + threadIdx.y; 
    const float4* inputs = _inputs + batch_idx * IC / PACK_FACTOR;
    half* outputs = _outputs + batch_idx * OC;
    const int num_groups_packed = make_divisible(IC / group_size, PACK_FACTOR);
    const int weight_w = IC / PACK_FACTOR;
    // TODO (Haotian): zeros_w is incorrect, after fixing we got misaligned address
    const int zeros_w = make_divisible(IC / group_size, PACK_FACTOR);
    // consistent with input shape
    const int sf_w = make_divisible(IC / group_size, PACK_FACTOR) * PACK_FACTOR;
    //if(blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) printf("%d %d %d %d\n", IC, group_size, PACK_FACTOR, zeros_w);
    // tile size: 4 OC x 1024 IC per iter
    for(int packed_group_idx = 0; packed_group_idx < num_groups_packed; packed_group_idx++){
      // 1024 numbers in one iteration across warp. Need 1024 / group_size zeros.
      uint32_t packed_zeros = *(zeros + oc_idx * zeros_w + packed_group_idx);
      uint32_t packed_weights[4];
      // use float4 to load weights, each thread load 32 int4 numbers (1 x float4)
      *((float4*)(packed_weights)) = *((float4*)(weight + oc_idx * weight_w + packed_group_idx * (WARP_SIZE * 4) + threadIdx.x * 4));
      // load scaling factors
      // g128: four threads -> 128 numbers -> 1 group; 1 warp = 8 groups.
      float scaling_factor = __half2float(scaling_factors[oc_idx * sf_w + packed_group_idx * 8 + (threadIdx.x / 4)]);
      float current_zeros = (float)((packed_zeros >> (threadIdx.x / 4 * 4)) & 0xF);
      int inputs_ptr_delta = packed_group_idx * WARP_SIZE * 4 + threadIdx.x * 4; 
      const float4* inputs_ptr = inputs + inputs_ptr_delta;
      // multiply 32 weights with 32 inputs
      #pragma unroll
      for (int ic_0 = 0; ic_0 < 4; ic_0++){
        // iterate over different uint32_t packed_weights in this loop
        uint32_t current_packed_weight = packed_weights[ic_0];
        half packed_inputs[PACK_FACTOR];
        // each thread load 8 inputs, starting index is packed_group_idx * 128 * 8 (because each iter loads 128*8)
        if (inputs_ptr_delta + ic_0 < IC / PACK_FACTOR) {
          *((float4*)packed_inputs) = *(inputs_ptr + ic_0);
          #pragma unroll
          for (int ic_1 = 0; ic_1 < PACK_FACTOR; ic_1++){
            // iterate over 8 numbers packed within each uint32_t number
            float current_single_weight_fp = (float)(current_packed_weight & 0xF);
            float dequantized_weight = scaling_factor * (current_single_weight_fp - current_zeros);
            //if(blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0 && ic_0 == 0 && ic_1 == 0 && packed_group_idx == 0) printf("%f %f %f %f %X %X\n", dequantized_weight, current_single_weight_fp, scaling_factor, current_zeros, current_packed_weight, packed_zeros);
            psum += dequantized_weight * __half2float(packed_inputs[ic_1]);
            current_packed_weight = current_packed_weight >> 4;
          }
        }
      }
    }
    psum = warp_reduce_sum(psum);
    if (threadIdx.x == 0) {
     outputs[oc_idx] = __float2half(psum); 
    }
}


namespace matmul{
  
  /*
  Computes GEMV.

  Args:
    _in_feats: tensor of shape [B, IC];
    _kernel: int tensor of shape [OC, IC // 8];
    _zeros: int tensor of shape [OC, IC // G // 8];
    _scaling_factors: tensor of shape [OC, IC // G];
    blockDim_x: size of thread block, dimension x, where blockDim_x * workload_per_thread = IC;
    blockDim_y: size of thread block, dimension y, where blockDim_y * gridDim_y = OC;

  Returns:
    out_feats: tensor of shape [B, OC];
  */
  void MatmulOperator::gemv_forward_cuda(const struct matmul_params *params)
  {
    const struct matrix *A = &params->A, *B = &params->B, *C = &params->C;

    int num_in_feats = A->row;
    int num_in_channels = A->column;
    int num_out_feats = C->row;
    int num_out_channels = C->column;
    int group_size = QK;

    auto in_feats = reinterpret_cast<float4*>(A->half_data_ptr);
    auto kernel = reinterpret_cast<uint32_t*>(B->int32_data_ptr);
    auto zeros = reinterpret_cast<uint32_t*>(params->int32_zero_point);
    auto scaling_factors = reinterpret_cast<half*>(params->half_scales);
    auto out_feats = reinterpret_cast<half*>(C->half_data_ptr);

    int blockDim_z = num_out_feats;
    dim3 num_blocks(1, num_out_channels / 4, num_out_feats);
    dim3 num_threads(32, 4);

    PROFILE_START("gemv_forward_cuda");

    if (group_size == 64)
    {
      gemv_kernel_g64<<<num_blocks, num_threads>>>(
        // pointers
        in_feats, kernel, zeros, scaling_factors, out_feats,
        // constants
        num_in_channels, num_out_channels
      );
    }
    else if (group_size == 128)
    {
      gemv_kernel_g128<<<num_blocks, num_threads>>>(
        // pointers
        in_feats, kernel, zeros, scaling_factors, out_feats,
        // constants
        num_in_channels, num_out_channels
      );
    }
    else
    {
      printf("Unsupported group size: %d\n", group_size);
      exit(1);
    }

    PROFILE_END("gemv_forward_cuda");
  }

  void MatmulOperator::mat_mul_accelerator_int4_fast(const struct matmul_params *params) {
    // TODO: remove this
  };

  void MatmulOperator::mat_mul_accelerator_int4_fast_no_offset(const struct matmul_params *params) {
    // TODO: remove this
  };

}  // namespace matmul
